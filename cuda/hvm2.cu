#include "hip/hip_runtime.h"
#include <stdarg.h>
#include <stdbool.h>
#include <stdint.h>
#include <stdio.h>
#include <string.h>
#include <unistd.h>

typedef uint8_t u8;
typedef uint16_t u16;
typedef uint32_t u32;
typedef unsigned long long int u64;

// Configuration
// -------------

// This code is initially optimized for RTX 4090
const u32 BLOCK_LOG2    = 9;                         // log2 of block size
const u32 BLOCK_SIZE    = 1 << BLOCK_LOG2;           // threads per block
const u32 UNIT_LOG2     = 2;                         // log2 of unit size
const u32 UNIT_SIZE     = 1 << UNIT_LOG2;            // threads per unit
const u32 GROUP_LOG2    = BLOCK_LOG2 - UNIT_LOG2;    // log2 of group size
const u32 GROUP_SIZE    = 1 << GROUP_LOG2;           // units per group
const u32 NODE_LOG2     = 28;                        // log2 of node size
const u32 NODE_SIZE     = 1 << NODE_LOG2;            // max total nodes (2GB addressable)
const u32 HEAD_LOG2     = GROUP_LOG2 * 2;            // log2 of head size
const u32 HEAD_SIZE     = 1 << HEAD_LOG2;            // max head pointers
const u32 MAX_THREADS   = BLOCK_SIZE * BLOCK_SIZE;   // total number of active threads
const u32 MAX_UNITS     = GROUP_SIZE * GROUP_SIZE;   // total number of active units
const u32 MAX_NEW_REDEX = 4;                         // max new redexes per rewrite
const u32 SMEM_SIZE     = 4;                         // u32's shared by unit
const u32 RBAG_SIZE     = 32;                        // redexes per unit = 32
const u32 LHDS_SIZE     = 32;                        // max local heads
const u32 BAGS_SIZE     = MAX_UNITS * RBAG_SIZE;     // redexes per GPU

// Types
// -----

typedef u8  Tag; // pointer tag: 4-bit
typedef u32 Val; // pointer val: 28-bit

// Core terms
const Tag NIL = 0x0; // empty node
const Tag REF = 0x1; // reference to a definition (closed net)
const Tag ERA = 0x2; // unboxed eraser
const Tag VRR = 0x3; // variable pointing to root
const Tag VR1 = 0x4; // variable pointing to aux1 port of node
const Tag VR2 = 0x5; // variable pointing to aux2 port of node
const Tag RDR = 0x6; // redirection to root
const Tag RD1 = 0x7; // redirection to aux1 port of node
const Tag RD2 = 0x8; // redirection to aux2 port of node
const Tag NUM = 0x9; // unboxed number
const Tag CON = 0xA; // points to main port of con node
const Tag DUP = 0xB; // points to main port of dup node
const Tag TRI = 0xC; // points to main port of tri node
const Tag CTR = 0xF; // last constructor

// Special values
const u32 NEO = 0xFFFFFFFD; // recently allocated value
const u32 TMP = 0xFFFFFFFE; // node has been moved to redex bag
const u32 TKN = 0xFFFFFFFF; // value taken by another thread, will be replaced soon

// Worker types
const u32 A1 = 0; // focuses on the A node, P1 port
const u32 A2 = 1; // focuses on the A node, P2 port
const u32 B1 = 2; // focuses on the B node, P1 port
const u32 B2 = 3; // focuses on the B node, P2 port

// Ports (P1 or P2)
typedef u8 Port;
const u32 P1 = 0;
const u32 P2 = 1;

// Pointers = 4-bit tag + 28-bit val
typedef u32 Ptr;

// Nodes are pairs of pointers
typedef struct {
  Ptr ports[2];
} Node;

// Wires are pairs of pointers
typedef u64 Wire;

// Maximum number of defs in a book
const u32 MAX_DEFS = 1 << 24; // FIXME: make a proper HashMap

typedef struct {
  Ptr   root;
  u32   alen;
  Wire* acts;
  u32   nlen;
  Node* node;
} Term;

// A book
typedef struct {
  Term** defs;
} Book;

// An interaction net 
typedef struct {
  Ptr   root; // root wire
  Wire* bags; // redex bags (active pairs)
  Node* node; // memory buffer with all nodes
  Wire* head; // head expansion buffer
  u32   done; // number of completed threads
  u64   rwts; // number of rewrites performed
} Net;

// A worker local data
typedef struct {
  u32   tid;  // thread id (local)
  u32   uid;  // unit id (global)
  u32   quad; // worker quad (A1|A2|B1|B2)
  u32   port; // worker port (P1|P2)
  u32   aloc; // where to alloc next node
  u64   rwts; // local rewrites performed
  u32*  sm32; // shared 32-bit buffer
  u64*  sm64; // shared 64-bit buffer
  u64*  rlen; // local redex bag length
  Wire* rbag; // local redex bag
} Worker;

// Debug
// -----

__device__ __host__ void stop(const char* tag) {
  printf(tag);
  printf("\n");
}

__device__ __host__ bool dbug(u32* K, const char* tag) {
  *K += 1;
  if (*K > 5000000) {
    stop(tag);
    return false;
  }
  return true;
}

// Runtime
// -------

// Integer ceil division
__host__ __device__ u32 div(u32 a, u32 b) {
  return (a + b - 1) / b;
}

// Pseudorandom Number Generator
__host__ __device__ u32 rng(u32 a) {
  return a * 214013 + 2531011;
}

// Creates a new pointer
__host__ __device__ inline Ptr mkptr(Tag tag, Val val) {
  return ((u32)tag << 28) | (val & 0xFFFFFFF);
}

// Gets the tag of a pointer
__host__ __device__ inline Tag tag(Ptr ptr) {
  return (Tag)(ptr >> 28);
}

// Gets the value of a pointer
__host__ __device__ inline Val val(Ptr ptr) {
  return ptr & 0xFFFFFFF;
}

// Is this pointer a variable?
__host__ __device__ inline bool is_var(Ptr ptr) {
  return tag(ptr) >= VRR && tag(ptr) <= VR2;
}

// Is this pointer a redirection?
__host__ __device__ inline bool is_red(Ptr ptr) {
  return tag(ptr) >= RDR && tag(ptr) <= RD2;
}

// Is this pointer a constructor?
__host__ __device__ inline bool is_ctr(Ptr ptr) {
  return tag(ptr) >= CON && tag(ptr) < CTR;
}

// Is this pointer an eraser?
__host__ __device__ inline bool is_era(Ptr ptr) {
  return tag(ptr) == ERA;
}

// Is this pointer a number?
__host__ __device__ inline bool is_num(Ptr ptr) {
  return tag(ptr) == NUM;
}

// Is this pointer a reference?
__host__ __device__ inline bool is_ref(Ptr ptr) {
  return tag(ptr) == REF;
}

// Is this pointer a main port?
__host__ __device__ inline bool is_pri(Ptr ptr) {
  return is_era(ptr)
      || is_ctr(ptr)
      || is_num(ptr)
      || is_ref(ptr);
}

// Is this pointer carrying a location (that needs adjustment)?
__host__ __device__ inline bool has_loc(Ptr ptr) {
  return is_ctr(ptr)
      || is_var(ptr) && tag(ptr) != VRR
      || is_red(ptr) && tag(ptr) != RDR;
}

// Gets the target ref of a var or redirection pointer
__host__ __device__ inline Ptr* target(Net* net, Ptr ptr) {
  if (tag(ptr) == VRR || tag(ptr) == RDR) {
    return &net->root;
  } else if (tag(ptr) == VR1 || tag(ptr) == RD1) {
    return &net->node[val(ptr)].ports[P1];
  } else if (tag(ptr) == VR2 || tag(ptr) == RD2) {
    return &net->node[val(ptr)].ports[P2];
  } else {
    return NULL;
  }
}

// Traverses to the other side of a wire
__host__ __device__ Ptr enter(Net* net, Ptr ptr) {
  Ptr* ref = target(net, ptr);
  while (tag(*ref) >= RDR && tag(*ref) <= RD2) {
    ptr = *ref;
    ref = target(net, ptr);
  }
  return ptr;
}

// Transforms a variable into a redirection
__host__ __device__ inline Ptr redir(Ptr ptr) {
  return mkptr(tag(ptr) + (is_var(ptr) ? 3 : 0), val(ptr));
}

// Creates a new wire
__host__ __device__ inline Wire mkwire(Ptr p1, Ptr p2) {
  return (((u64)p1) << 32) | ((u64)p2);
}

// Gets the left element of a wire
__host__ __device__ inline Ptr wire_lft(Wire wire) {
  return wire >> 32;
}

// Gets the right element of a wire
__host__ __device__ inline Ptr wire_rgt(Wire wire) {
  return wire & 0xFFFFFFFF;
}

// Creates a new node
__host__ __device__ inline Node mknode(Ptr p1, Ptr p2) {
  Node node;
  node.ports[P1] = p1;
  node.ports[P2] = p2;
  return node;
}

// Creates a nil node
__host__ __device__ inline Node Node_nil() {
  return mknode(mkptr(NIL, 0), mkptr(NIL, 0));
}

// Gets a reference to the index/port Ptr on the net
__device__ inline Ptr* at(Net* net, Val idx, Port port) {
  return &net->node[idx].ports[port];
}

// Allocates one node in memory
__device__ inline u32 alloc(Worker *worker, Net *net) {
  while (true) {
    u64* ref = (u64*)&net->node[worker->aloc];
    u64  got = atomicCAS((u64*)ref, 0, ((u64)NEO << 32) | (u64)NEO);
    worker->aloc = (worker->aloc + 1) % NODE_SIZE;
    if (got == 0) {
      return (worker->aloc - 1) % NODE_SIZE;
    }
  }
}

// Allocates many nodes in memory
// TODO: use the entire squad to perform this
__device__ inline u32 alloc_many(Worker *worker, Net *net, u32 size) {
  u64 MKNEO = ((u64)NEO << 32) | (u64)NEO;
  u32 space = 0;
  while (true) {
    if (worker->aloc + size - space > NODE_SIZE) {
      worker->aloc = 0;
    }
    u64* ref = (u64*)&net->node[worker->aloc];
    u64  got = atomicCAS(ref, 0, MKNEO);
    if (got != 0) {
      for (u32 i = 0; i < space; ++i) {
        u32  index = (worker->aloc - space + i) % NODE_SIZE;
        Node clear = mknode(mkptr(NIL,0), mkptr(NIL,0));
        u64* ref = (u64*)&net->node[index];
        u64  got = atomicCAS(ref, MKNEO, 0);
      }
      space = 0;
    } else {
      space += 1;
    }
    worker->aloc = (worker->aloc + 1) % NODE_SIZE;
    if (space == size) {
      return (worker->aloc - space) % NODE_SIZE;
    }
  }
}

// Gets the value of a ref; waits if taken.
__device__ Ptr take(Ptr* ref) {
  Ptr got = atomicExch((u32*)ref, TKN);
  u32 K = 0;
  while (got == TKN) {
    //dbug(&K, "take");
    got = atomicExch((u32*)ref, TKN);
  }
  return got;
}

// Attempts to replace 'exp' by 'neo', until it succeeds
__device__ bool replace(Ptr* ref, Ptr exp, Ptr neo) {
  Ptr got = atomicCAS((u32*)ref, exp, neo);
  u32 K = 0;
  while (got != exp) {
    //dbug(&K, "replace");
    got = atomicCAS((u32*)ref, exp, neo);
  }
  return true;
}

// Splits elements of two arrays evenly between each-other
__device__ inline void split(u32 tid, u64* a_len, u64* a_arr, u64* b_len, u64* b_arr, u64 max_len) {
  __syncthreads();
  u64* A_len = *a_len < *b_len ? a_len : b_len;
  u64* B_len = *a_len < *b_len ? b_len : a_len;
  u64* A_arr = *a_len < *b_len ? a_arr : b_arr;
  u64* B_arr = *a_len < *b_len ? b_arr : a_arr;
  bool move  = *A_len + 1 < *B_len;
  u64  min   = *A_len;
  u64  max   = *B_len;
  __syncthreads();
  for (u64 t = 0; t < max_len / (UNIT_SIZE * 2); ++t) {
    u64 i = min + t * (UNIT_SIZE * 2) + tid;
    u64 value;
    if (move && i < max) {
      value = B_arr[i];
      B_arr[i] = 0;
    }
    __syncthreads();
    if (move && i < max) {
      if ((i - min) % 2 == 0) {
        A_arr[min + (t * (UNIT_SIZE * 2) + tid) / 2] = value;
      } else {
        B_arr[min + (t * (UNIT_SIZE * 2) + tid) / 2] = value;
      }
    }
  }
  __syncthreads();
  u64 old_A_len = *A_len;
  u64 old_B_len = *B_len;
  if (move && tid == 0) {
    u64 new_A_len = (*A_len + *B_len) / 2 + (*A_len + *B_len) % 2;
    u64 new_B_len = (*A_len + *B_len) / 2;
    *A_len = new_A_len;
    *B_len = new_B_len;
  }
  __syncthreads();
}

// Pops a redex
__device__ inline Wire pop_redex(Worker* worker) {
  if (worker->quad == A1) {
    Wire redex = mkwire(0,0);
    if (*worker->rlen > 0) {
      u64 index = *worker->rlen - 1;
      *worker->rlen -= 1;
      redex = worker->rbag[index];
      worker->rbag[index] = mkwire(0,0);
    }
    *worker->sm64 = redex;
  }
  __syncwarp();
  Wire got = *worker->sm64;
  __syncwarp();
  *worker->sm64 = 0;
  if (worker->quad <= A2) {
    return mkwire(wire_lft(got), wire_rgt(got));
  } else {
    return mkwire(wire_rgt(got), wire_lft(got));
  }
}

// Puts a redex
__device__ inline void put_redex(Worker* worker, Ptr a_ptr, Ptr b_ptr) {
  //printf("[%04x:%x] put_redex %08x %08x\n", worker->uid, worker->quad, a_ptr, b_ptr);

  // optimization: avoids pushing non-reactive redexes
  bool exclude
    =  is_era(a_ptr) && is_era(b_ptr)
    || is_ref(a_ptr) && is_era(b_ptr)
    || is_era(a_ptr) && is_ref(b_ptr)
    || is_ref(a_ptr) && is_ref(b_ptr);
  if (exclude) {
    worker->rwts += 1;
    return;
  }

  // pushes redex to end of bag
  u32 index = atomicAdd(worker->rlen, 1);
  if (index < RBAG_SIZE - 1) {
    worker->rbag[index] = mkwire(a_ptr, b_ptr);
  }
}

// Adjusts a dereferenced pointer
__device__ Ptr adjust(Worker* worker, Ptr ptr, u32 delta) {
  return mkptr(tag(ptr), has_loc(ptr) ? val(ptr) + delta : val(ptr));
}

// Dereferences a global definition
__device__ void deref(Worker* worker, Net* net, Book* book, Ptr* deref_ptr, Ptr parent) {
  // Loads definition
  Term* term = NULL;
  if (deref_ptr != NULL) {
    term = book->defs[val(*deref_ptr)];
  }

  // Allocates needed space
  if (term != NULL && worker->quad == A1) {
    worker->sm32[0] = alloc_many(worker, net, term->nlen);
  }
  __syncwarp();
  u32 delta = 0;
  if (term != NULL) {
    delta = worker->sm32[0];
  }

  // Loads dereferenced nodes, adjusted
  if (term != NULL) {
    for (u32 i = 0; i < div(term->nlen, UNIT_SIZE); ++i) {
      u32 loc = i * UNIT_SIZE + worker->quad;
      if (loc < term->nlen) {
        //printf("... node %u\n", loc);
        Node got = term->node[loc];
        Ptr  p1  = adjust(worker, got.ports[P1], delta);
        Ptr  p2  = adjust(worker, got.ports[P2], delta);
        replace(at(net, delta + loc, P1), NEO, p1);
        replace(at(net, delta + loc, P2), NEO, p2);
      }
    }
  }

  // Loads dereferenced redexes, adjusted
  if (term != NULL) {
    for (u32 i = 0; i < div(term->alen, UNIT_SIZE); ++i) {
      u32 loc = i * UNIT_SIZE + worker->quad;
      if (loc < term->alen) {
        Wire got = term->acts[loc];
        Ptr  p1  = adjust(worker, wire_lft(got), delta);
        Ptr  p2  = adjust(worker, wire_rgt(got), delta);
        put_redex(worker, p1, p2);
      }
    }
  }

  // Loads dereferenced root, adjusted
  if (term != NULL) {
    *deref_ptr = adjust(worker, term->root, delta);
  }
  __syncwarp();

  // Links root
  if (term != NULL && worker->quad == A1) {
    Ptr* trg = target(net, *deref_ptr);
    if (trg != NULL) {
      *trg = parent;
    }
  }
}

// Atomically links the node in 'src_ref' towards 'trg_ptr'.
__device__ void link(Worker* worker, Net* net, Book* book, Ptr* src_ref, Ptr src_ptr, Ptr dir_ptr) {

  // Create a new redex
  if (is_pri(src_ptr) && is_pri(dir_ptr)) {
    atomicCAS(src_ref, TKN, 0);
    put_redex(worker, *src_ref, dir_ptr);
    return;
  }

  // Move src towards either a VAR or another PRI
  if (is_pri(src_ptr) && is_red(dir_ptr)) {
    while (true) {
      //dbug(&K, "link");

      // Peek the target, which may not be owned by us.
      Ptr* trg_ref = target(net, dir_ptr);
      Ptr  trg_ptr = atomicAdd(trg_ref, 0);

      // If target is a redirection, clear and move forward.
      if (is_red(trg_ptr)) {
        // We own the redirection, so we can mutate it.
        *trg_ref = 0;
        dir_ptr = trg_ptr;
        continue;
      }

      // If target is a variable, try replacing it by the node.
      else if (is_var(trg_ptr)) {
        // Peeks the source node.
        Ptr src_ptr = *src_ref;

        // We don't own the var, so we must try replacing with a CAS.
        if (atomicCAS((u32*)trg_ref, trg_ptr, src_ptr) == trg_ptr) {
          // Collect the orphaned backward path.
          trg_ref = target(net, trg_ptr);
          trg_ptr = *trg_ref;
          while (is_red(trg_ptr)) {
            *trg_ref = 0;
            trg_ref = target(net, trg_ptr);
            trg_ptr = *trg_ref;
          }
          // Clear source location.
          *src_ref = 0;
          return;
        }

        // If the CAS failed, the var changed, so we try again.
        continue;
      }

      // If it is a node, two threads will reach this branch.
      else if (is_pri(trg_ptr) || is_ref(trg_ptr) || trg_ptr == TMP) {

        // Sort references, to avoid deadlocks.
        Ptr *fst_ref = src_ref < trg_ref ? src_ref : trg_ref;
        Ptr *snd_ref = src_ref < trg_ref ? trg_ref : src_ref;

        // Swap first reference by TMP placeholder.
        Ptr fst_ptr = atomicExch((u32*)fst_ref, TMP);

        // First to arrive creates a redex.
        if (fst_ptr != TMP) {
          Ptr snd_ptr = atomicExch((u32*)snd_ref, TMP);
          put_redex(worker, fst_ptr, snd_ptr);
          return;

        // Second to arrive clears up the memory.
        } else {
          *fst_ref = 0;
          replace((u32*)snd_ref, TMP, 0);
          return;
        }
      }

      // If it is taken, we wait.
      else if (trg_ptr == TKN) {
        continue;
      }

      // Shouldn't be reached.
      else {
        return;
      }
    }
  }

  // Optimization: safely shorten redirections
  if (is_red(src_ptr) && is_red(dir_ptr)) {
    while (true) {
      Ptr* ste_ref = target(net, src_ptr);
      Ptr  ste_ptr = *ste_ref;
      if (is_var(ste_ptr)) {
        Ptr* trg_ref = target(net, ste_ptr);
        Ptr  trg_ptr = atomicAdd(trg_ref, 0);
        if (is_red(trg_ptr)) {
          Ptr neo_ptr = mkptr(tag(trg_ptr) - 3, val(trg_ptr));
          Ptr updated = atomicCAS(ste_ref, ste_ptr, neo_ptr);
          if (updated == ste_ptr) {
            *trg_ref = 0;
            continue;
          }
        }
      }
      break;
    }
    return;
  }

}

// Rewrite
// -------

__device__ Worker init_worker(Net* net, bool flip) {
  __shared__ u32 SMEM[GROUP_SIZE * SMEM_SIZE];

  for (u32 i = 0; i < GROUP_SIZE * SMEM_SIZE / BLOCK_SIZE; ++i) {
    SMEM[i * BLOCK_SIZE + threadIdx.x] = 0;
  }
  __syncthreads();

  u32 tid = threadIdx.x;
  u32 gid = blockIdx.x * blockDim.x + tid;
  u32 uid = gid / UNIT_SIZE;
  u32 row = uid / GROUP_SIZE;
  u32 col = uid % GROUP_SIZE;

  Worker worker;
  worker.uid  = flip ? col * GROUP_SIZE + row : row * GROUP_SIZE + col;
  worker.tid  = threadIdx.x;
  worker.aloc = rng(clock() * (gid + 1)) % NODE_SIZE;
  worker.rwts = 0;
  worker.quad = worker.tid % 4;
  worker.port = worker.tid % 2;
  worker.sm32 = (u32*)(SMEM + worker.tid / UNIT_SIZE * SMEM_SIZE);
  worker.sm64 = (u64*)(SMEM + worker.tid / UNIT_SIZE * SMEM_SIZE);
  worker.rlen = net->bags + worker.uid * RBAG_SIZE;
  worker.rbag = worker.rlen + 1;

  return worker;
}

// An active wire is reduced by 4 parallel threads, each one performing "1/4" of
// the work. Each thread will be pointing to a node of the active pair, and an
// aux port of that node. So, when nodes A-B interact, we have 4 thread quads:
// - Thread A1: points to node A and its aux1
// - Thread A2: points to node A and its aux2
// - Thread B1: points to node B and its aux1
// - Thread B2: points to node B and its aux2
// This is organized so that local threads can perform the same instructions
// whenever possible. So, for example, in a commutation rule, all the 4 clones
// would be allocated at the same time.
__global__ void global_rewrite(Net* net, Book* book, u32 repeat, u32 tick, bool flip) {

  // Initializes local vars
  Worker worker = init_worker(net, flip);

  for (u32 turn = 0; turn < repeat; ++turn) {
    // Checks if we're full
    bool is_full = *worker.rlen > RBAG_SIZE - MAX_NEW_REDEX;

    // Pops a redex from local bag
    Wire redex;
    Ptr a_ptr, b_ptr;
    if (!is_full) {
      redex = pop_redex(&worker);
      a_ptr = wire_lft(redex);
      b_ptr = wire_rgt(redex);
    } else {
      //printf("[%04x:%x] full\n", worker.uid, worker.quad);
    }
    __syncwarp();

    // Dereferences
    Ptr* deref_ptr = NULL;
    if (is_ref(a_ptr) && is_ctr(b_ptr)) {
      deref_ptr = &a_ptr;
    }
    if (is_ref(b_ptr) && is_ctr(a_ptr)) {
      deref_ptr = &b_ptr;
    }
    deref(&worker, net, book, deref_ptr, mkptr(NIL,0));

    // Defines type of interaction
    bool rewrite = !is_full && a_ptr != 0 && b_ptr != 0;
    bool var_pri = rewrite && is_var(a_ptr) && is_pri(b_ptr) && worker.port == P1;
    bool era_ctr = rewrite && is_era(a_ptr) && is_ctr(b_ptr);
    bool ctr_era = rewrite && is_ctr(a_ptr) && is_era(b_ptr);
    bool con_con = rewrite && is_ctr(a_ptr) && is_ctr(b_ptr) && tag(a_ptr) == tag(b_ptr);
    bool con_dup = rewrite && is_ctr(a_ptr) && is_ctr(b_ptr) && tag(a_ptr) != tag(b_ptr);

    // Local rewrite variables
    Ptr  ak_dir; // dir to our aux port
    Ptr  bk_dir; // dir to other aux port
    Ptr *ak_ref; // ref to our aux port
    Ptr *bk_ref; // ref to other aux port
    Ptr  ak_ptr; // val of our aux port
    Ptr  mv_ptr; // val of ptr to send to other side
    u32  y0_idx; // idx of other clone idx

    // Inc rewrite count
    if (rewrite && worker.quad == A1) {
      worker.rwts += 1;
    }

    // Gets port here
    if (rewrite && (ctr_era || con_con || con_dup)) {
      ak_dir = mkptr(RD1 + worker.port, val(a_ptr));
      ak_ref = target(net, ak_dir);
      ak_ptr = take(ak_ref);
    }

    // Gets port there
    if (rewrite && (era_ctr || con_con || con_dup)) {
      bk_dir = mkptr(RD1 + worker.port, val(b_ptr));
      bk_ref = target(net, bk_dir);
    }

    // If era_ctr, send an erasure
    if (rewrite && era_ctr) {
      mv_ptr = mkptr(ERA, 0);
    }

    // If con_con, send a redirection
    if (rewrite && con_con) {
      mv_ptr = redir(ak_ptr);
    }

    // If con_dup, alloc clones base index
    if (rewrite && con_dup && worker.quad == A1) {
      worker.sm32[0] = alloc_many(&worker, net, 4);
    }
    __syncwarp();

    // If con_dup, create inner wires between clones
    if (rewrite && con_dup) {
      u32 al_loc = worker.sm32[0];
      u32 cx_loc = al_loc + worker.quad;
      u32 c1_loc = al_loc + (worker.quad <= A2 ? 2 : 0);
      u32 c2_loc = al_loc + (worker.quad <= A2 ? 3 : 1);
      replace(at(net, cx_loc, P1), NEO, mkptr(worker.port == P1 ? VR1 : VR2, c1_loc));
      replace(at(net, cx_loc, P2), NEO, mkptr(worker.port == P1 ? VR1 : VR2, c2_loc));
      mv_ptr = mkptr(tag(a_ptr), cx_loc);
    }
    __syncwarp();

    // Send ptr to other side
    if (rewrite && (era_ctr || con_con || con_dup)) {
      worker.sm32[worker.quad + (worker.quad <= A2 ? 2 : -2)] = mv_ptr;
    }
    __syncwarp();

    // Receive ptr from other side
    if (rewrite && (con_con || ctr_era || con_dup)) {
      *ak_ref = worker.sm32[worker.quad];
    }
    __syncwarp();

    // If var_pri, the var must be a deref root, so we just subst
    if (rewrite && var_pri && worker.port == P1) {
      atomicExch((u32*)target(net, a_ptr), b_ptr);
    }

    // Links the rewritten port
    Ptr* src_ref, src_ptr, dir_ptr;
    if (rewrite && (con_con || ctr_era || con_dup)) {
      src_ref = con_con ? ak_ref : ak_ref;
      src_ptr = *src_ref;
      dir_ptr = con_con ? redir(bk_dir) : redir(ak_ptr);
    }
    __syncwarp();
    if (rewrite && (con_con || ctr_era || con_dup)) {
      link(&worker, net, book, src_ref, src_ptr, dir_ptr);
    }
    __syncwarp();
  }

  // Splits redexes with neighbor
  u32  side  = ((worker.tid / UNIT_SIZE) >> (GROUP_LOG2 - 1 - (tick % GROUP_LOG2))) & 1;
  u32  lpad  = (1 << (GROUP_LOG2 - 1)) >> (tick % GROUP_LOG2);
  u32  gpad  = flip ? lpad * GROUP_SIZE : lpad;
  u32  a_uid = worker.uid;
  u32  b_uid = side ? worker.uid - gpad : worker.uid + gpad;
  u64* a_len = net->bags + a_uid * RBAG_SIZE;
  u64* b_len = net->bags + b_uid * RBAG_SIZE;
  //printf("[%04x:%x] tid=%x split! %04x ~ %04x | flip=%u tick=%u side=%u lpad=%u gpad=%u\n", worker.uid, worker.quad, worker.tid, a_uid, b_uid, flip, tick, side, lpad, gpad);
  split(worker.quad + side * UNIT_SIZE, a_len, a_len+1, b_len, b_len+1, RBAG_SIZE);
  __syncthreads();

  // When the work ends, sum stats
  if (worker.rwts > 0) {
    atomicAdd(&net->rwts, worker.rwts);
  }
}

void do_global_rewrite(Net* net, Book* book, u32 repeat, u32 tick, bool flip) {
  global_rewrite<<<GROUP_SIZE, BLOCK_SIZE>>>(net, book, repeat, tick, flip);
  // print any error launching this kernel:
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(err));
  }
}

// Expand
// ------

// Collects local expansion heads recursively
__device__ void expand(Worker* worker, Net* net, Book* book, Ptr dir, u32* len, u32* lhds) {
  Ptr ptr = *target(net, dir);
  if (is_ctr(ptr)) {
    expand(worker, net, book, mkptr(VR1, val(ptr)), len, lhds);
    expand(worker, net, book, mkptr(VR2, val(ptr)), len, lhds);
  } else if (is_red(ptr)) {
    expand(worker, net, book, ptr, len, lhds);
  } else if (is_ref(ptr) && *len < LHDS_SIZE) {
    lhds[(*len)++] = dir;
  }
}

// Takes an initial head location for each unit
__global__ void global_expand_prepare(Net* net) {
  u32 uid = blockIdx.x * blockDim.x + threadIdx.x;

  // Traverses down
  u32 key = uid;
  Ptr dir = mkptr(VRR, 0);
  Ptr ptr, *ref;
  for (u32 depth = 0; depth < HEAD_LOG2; ++depth) {
    dir = enter(net, dir);
    ref = target(net, dir);
    if (ref != NULL) {
      ptr = *ref;
      if (is_ctr(ptr)) {
        dir = mkptr(key & 1 ? VR1 : VR2, val(ptr));
        key = key >> 1;
      }
    }
  }

  // Takes ptr
  dir = enter(net, dir);
  ref = target(net, dir);
  if (ref != NULL) {
    ptr = atomicExch(ref, TKN);
  }

  // Stores ptr
  if (ptr != TKN) {
    net->head[uid] = mkwire(dir, ptr);
  } else {
    net->head[uid] = mkwire(mkptr(NIL,0), mkptr(NIL,0));
  }

}

// Performs global expansion of heads
__global__ void global_expand(Net* net, Book* book) {
  __shared__ u32 HEAD[GROUP_SIZE * LHDS_SIZE];

  for (u32 i = 0; i < GROUP_SIZE * LHDS_SIZE / BLOCK_SIZE; ++i) {
    HEAD[i * BLOCK_SIZE + threadIdx.x] = 0;
  }
  __syncthreads();

  Worker worker = init_worker(net, 0);

  u32* head = HEAD + worker.tid / UNIT_SIZE * LHDS_SIZE;

  Wire got = net->head[worker.uid];
  Ptr  dir = wire_lft(got);
  Ptr* ref = target(net, dir);
  Ptr  ptr = wire_rgt(got);

  if (worker.quad == A1 && ptr != mkptr(NIL,0)) {
    *ref = ptr;
  }
  __syncthreads();

  u32 len = 0;
  if (worker.quad == A1 && ptr != mkptr(NIL,0)) {
    expand(&worker, net, book, dir, &len, head);
  }
  __syncthreads();

  for (u32 i = 0; i < LHDS_SIZE; ++i) {
    Ptr  dir = head[i];
    Ptr* ref = target(net, dir);
    if (ref != NULL && !is_ref(*ref)) {
      ref = NULL;
    }
    deref(&worker, net, book, ref, dir);
  }
  __syncthreads();
}

// Performs a global head expansion
void do_global_expand(Net* net, Book* book) {
  global_expand_prepare<<<GROUP_SIZE, GROUP_SIZE>>>(net);
  global_expand<<<GROUP_SIZE, BLOCK_SIZE>>>(net, book);
}

// Host<->Device
// -------------

__host__ Net* mknet() {
  Net* net  = (Net*)malloc(sizeof(Net));
  net->root = mkptr(NIL, 0);
  net->rwts = 0;
  net->done = 0;
  net->bags = (Wire*)malloc(BAGS_SIZE * sizeof(Wire));
  net->node = (Node*)malloc(NODE_SIZE * sizeof(Node));
  net->head = (Wire*)malloc(HEAD_SIZE * sizeof(Wire));
  memset(net->bags, 0, BAGS_SIZE * sizeof(Wire));
  memset(net->node, 0, NODE_SIZE * sizeof(Node));
  return net;
}

__host__ Net* net_to_gpu(Net* host_net) {
  // Allocate memory on the device for the Net object, and its data
  Net*  device_net;
  Wire* device_bags;
  Node* device_node;
  Wire* device_head;

  hipMalloc((void**)&device_net, sizeof(Net));
  hipMalloc((void**)&device_bags, BAGS_SIZE * sizeof(Wire));
  hipMalloc((void**)&device_node, NODE_SIZE * sizeof(Node));
  hipMalloc((void**)&device_head, HEAD_SIZE * sizeof(Wire));

  // Copy the host data to the device memory
  hipMemcpy(device_bags, host_net->bags, BAGS_SIZE * sizeof(Wire), hipMemcpyHostToDevice);
  hipMemcpy(device_node, host_net->node, NODE_SIZE * sizeof(Node), hipMemcpyHostToDevice);
  hipMemcpy(device_head, host_net->head, HEAD_SIZE * sizeof(Wire), hipMemcpyHostToDevice);

  // Create a temporary host Net object with device pointers
  Net temp_net  = *host_net;
  temp_net.bags = device_bags;
  temp_net.node = device_node;
  temp_net.head = device_head;

  // Copy the temporary host Net object to the device memory
  hipMemcpy(device_net, &temp_net, sizeof(Net), hipMemcpyHostToDevice);

  // Return the device pointer to the created Net object
  return device_net;
}

__host__ Net* net_to_cpu(Net* device_net) {
  // Create a new host Net object
  Net* host_net = (Net*)malloc(sizeof(Net));

  // Copy the device Net object to the host memory
  hipMemcpy(host_net, device_net, sizeof(Net), hipMemcpyDeviceToHost);

  // Allocate host memory for data
  host_net->bags = (Wire*)malloc(BAGS_SIZE * sizeof(Wire));
  host_net->node = (Node*)malloc(NODE_SIZE * sizeof(Node));
  host_net->head = (Wire*)malloc(HEAD_SIZE * sizeof(Wire));

  // Retrieve the device pointers for data
  Wire* device_bags;
  Node* device_node;
  Ptr*  device_head;
  hipMemcpy(&device_bags, &(device_net->bags), sizeof(Wire*), hipMemcpyDeviceToHost);
  hipMemcpy(&device_node, &(device_net->node), sizeof(Node*), hipMemcpyDeviceToHost);
  hipMemcpy(&device_head, &(device_net->head), sizeof(Wire*), hipMemcpyDeviceToHost);

  // Copy the device data to the host memory
  hipMemcpy(host_net->bags, device_bags, BAGS_SIZE * sizeof(Wire), hipMemcpyDeviceToHost);
  hipMemcpy(host_net->node, device_node, NODE_SIZE * sizeof(Node), hipMemcpyDeviceToHost);
  hipMemcpy(host_net->head, device_head, HEAD_SIZE * sizeof(Wire), hipMemcpyDeviceToHost);

  return host_net;
}

__host__ void net_free_on_gpu(Net* device_net) {
  // Retrieve the device pointers for data
  Wire* device_bags;
  Node* device_node;
  Wire* device_head;
  hipMemcpy(&device_bags, &(device_net->bags), sizeof(Wire*), hipMemcpyDeviceToHost);
  hipMemcpy(&device_node, &(device_net->node), sizeof(Node*), hipMemcpyDeviceToHost);
  hipMemcpy(&device_head, &(device_net->head), sizeof(Wire*), hipMemcpyDeviceToHost);

  // Free the device memory
  hipFree(device_bags);
  hipFree(device_node);
  hipFree(device_head);
  hipFree(device_net);
}

__host__ void net_free_on_cpu(Net* host_net) {
  free(host_net->bags);
  free(host_net->node);
  free(host_net->head);
  free(host_net);
}

// Creates a new book
__host__ __device__ inline Book* mkbook() {
  Book* book = (Book*)malloc(sizeof(Book));
  book->defs = (Term**)malloc(MAX_DEFS * sizeof(Term*));
  memset(book->defs, 0, sizeof(book->defs));
  return book;
}

__host__ Term* term_to_gpu(Term* host_term) {
  // Allocate memory on the device for the Term object, and its data
  Term* device_term;
  Wire* device_acts;
  Node* device_node;

  hipMalloc((void**)&device_term, sizeof(Term));
  hipMalloc((void**)&device_acts, host_term->alen * sizeof(Wire));
  hipMalloc((void**)&device_node, host_term->nlen * sizeof(Node));

  // Copy the host data to the device memory
  hipMemcpy(device_acts, host_term->acts, host_term->alen * sizeof(Wire), hipMemcpyHostToDevice);
  hipMemcpy(device_node, host_term->node, host_term->nlen * sizeof(Node), hipMemcpyHostToDevice);

  // Create a temporary host Term object with device pointers
  Term temp_term = *host_term;
  temp_term.acts = device_acts;
  temp_term.node = device_node;

  // Copy the temporary host Term object to the device memory
  hipMemcpy(device_term, &temp_term, sizeof(Term), hipMemcpyHostToDevice);

  // Return the device pointer to the created Term object
  return device_term;
}

__host__ Term* term_to_cpu(Term* device_term) {
  // Create a new host Term object
  Term* host_term = (Term*)malloc(sizeof(Term));

  // Copy the device Term object to the host memory
  hipMemcpy(host_term, device_term, sizeof(Term), hipMemcpyDeviceToHost);

  // Allocate host memory for data
  host_term->acts = (Wire*)malloc(host_term->alen * sizeof(Wire));
  host_term->node = (Node*)malloc(host_term->nlen * sizeof(Node));

  // Retrieve the device pointers for data
  Wire* device_acts;
  Node* device_node;
  hipMemcpy(&device_acts, &(device_term->acts), sizeof(Wire*), hipMemcpyDeviceToHost);
  hipMemcpy(&device_node, &(device_term->node), sizeof(Node*), hipMemcpyDeviceToHost);

  // Copy the device data to the host memory
  hipMemcpy(host_term->acts, device_acts, host_term->alen * sizeof(Wire), hipMemcpyDeviceToHost);
  hipMemcpy(host_term->node, device_node, host_term->nlen * sizeof(Node), hipMemcpyDeviceToHost);

  return host_term;
}

__host__ Book* book_to_gpu(Book* host_book) {
  Book* device_book;
  Term** device_defs;

  hipMalloc((void**)&device_book, sizeof(Book));
  hipMalloc((void**)&device_defs, MAX_DEFS * sizeof(Term*));
  hipMemset(device_defs, 0, MAX_DEFS * sizeof(Term*));

  for (u32 i = 0; i < MAX_DEFS; ++i) {
    if (host_book->defs[i] != NULL) {
      Term* device_term = term_to_gpu(host_book->defs[i]);
      hipMemcpy(device_defs + i, &device_term, sizeof(Term*), hipMemcpyHostToDevice);
    }
  }

  hipMemcpy(&(device_book->defs), &device_defs, sizeof(Term*), hipMemcpyHostToDevice);

  return device_book;
}

// opposite of book_to_gpu; same style as net_to_cpu and term_to_cpu
__host__ Book* book_to_cpu(Book* device_book) {
  // Create a new host Book object
  Book* host_book = (Book*)malloc(sizeof(Book));

  // Copy the device Book object to the host memory
  hipMemcpy(host_book, device_book, sizeof(Book), hipMemcpyDeviceToHost);

  // Allocate host memory for data
  host_book->defs = (Term**)malloc(MAX_DEFS * sizeof(Term*));

  // Retrieve the device pointer for data
  Term** device_defs;
  hipMemcpy(&device_defs, &(device_book->defs), sizeof(Term**), hipMemcpyDeviceToHost);

  // Copies device_defs into host_book->defs
  hipMemcpy(host_book->defs, device_defs, MAX_DEFS * sizeof(Term*), hipMemcpyDeviceToHost);

  // Copy the device data to the host memory
  for (u32 i = 0; i < MAX_DEFS; ++i) {
    if (host_book->defs[i] != NULL) {
      host_book->defs[i] = term_to_cpu(host_book->defs[i]);
    }
  }

  return host_book;
}

__host__ void book_free_on_gpu(Book* device_book) {
  // TODO
}

__host__ void book_free_on_cpu(Book* host_book) {
  // TODO
}

// Debugging
// ---------

__host__ const char* show_ptr(Ptr ptr, u32 slot) {
  static char buffer[8][20];
  if (ptr == 0) {
    strcpy(buffer[slot], "           ");
    return buffer[slot];
  } else if (ptr == TKN) {
    strcpy(buffer[slot], "[..........]");
    return buffer[slot];
  } else {
    const char* tag_str = NULL;
    switch (tag(ptr)) {
      case VR1: tag_str = "VR1"; break;
      case VR2: tag_str = "VR2"; break;
      case NIL: tag_str = "NIL"; break;
      case REF: tag_str = "REF"; break;
      case NUM: tag_str = "NUM"; break;
      case ERA: tag_str = "ERA"; break;
      case VRR: tag_str = "VRR"; break;
      case RDR: tag_str = "RDR"; break;
      case RD1: tag_str = "RD1"; break;
      case RD2: tag_str = "RD2"; break;
      case CON: tag_str = "CON"; break;
      case DUP: tag_str = "DUP"; break;
      case TRI: tag_str = "TRI"; break;
      default:tag_str = tag(ptr) >= DUP ? "DUP" : "???"; break;
    }
    snprintf(buffer[slot], sizeof(buffer[slot]), "%s:%07X", tag_str, val(ptr));
    return buffer[slot];
  }
}

// Prints a net in hexadecimal, limited to a given size
void print_net(Net* net) {
  printf("Root:\n");
  printf("- %s\n", show_ptr(net->root,0));
  printf("Bags:\n");
  for (u32 i = 0; i < BAGS_SIZE; ++i) {
    if (i % RBAG_SIZE == 0 && net->bags[i] > 0) {
      printf("- [%07X] LEN=%llu\n", i, net->bags[i]);
    } else {
      //Ptr a = wire_lft(net->bags[i]);
      //Ptr b = wire_rgt(net->bags[i]);
      //if (a != 0 || b != 0) {
        //printf("- [%07X] %s %s\n", i, show_ptr(a,0), show_ptr(b,1));
      //}
    }
  }
  //printf("Node:\n");
  //for (u32 i = 0; i < NODE_SIZE; ++i) {
    //Ptr a = net->node[i].ports[P1];
    //Ptr b = net->node[i].ports[P2];
    //if (a != 0 || b != 0) {
      //printf("- [%07X] %s %s\n", i, show_ptr(a,0), show_ptr(b,1));
    //}
  //}
  printf("Rwts: %llu\n", net->rwts);
}
//void print_net(Net* net) {
  ////printf("Root:\n");
  ////printf("- %s\n", show_ptr(net->root,0));
  //printf("net.root = Ptr { data: 0x%08x };\n", net->root);
  ////printf("Bags:\n");
  //for (u32 i = 0; i < BAGS_SIZE; ++i) {
    //if (i % RBAG_SIZE == 0 && net->bags[i] > 0) {
      ////printf("- [%07X] LEN=%llu\n", i, net->bags[i]);
    //} else {
      //Ptr a = wire_lft(net->bags[i]);
      //Ptr b = wire_rgt(net->bags[i]);
      //if (a != 0 || b != 0) {
        ////printf("- [%07X] %s %s\n", i, show_ptr(a,0), show_ptr(b,1));
        //printf("net.acts.push((Ptr { data: 0x%08x }, Ptr { data: 0x%08x }));\n", a, b);
      //}
    //}
  //}
  ////printf("Node:\n");
  //for (u32 i = 0; i < NODE_SIZE; ++i) {
    //Ptr a = net->node[i].ports[P1];
    //Ptr b = net->node[i].ports[P2];
    //if (a != 0 || b != 0) {
      ////printf("- [%07X] %s %s\n", i, show_ptr(a,0), show_ptr(b,1));
      //printf("net.node[0x%07X] = Node::new(Ptr { data: 0x%08x }, Ptr { data: 0x%08x });\n", i, a, b);
    //}
  //}
  //printf("Rwts: %u\n", net->rwts);
  //printf("\n");
//}

// Struct to represent a Map of entries using a simple array of (key,id) pairs
typedef struct {
  u32 keys[65536];
  u32 vals[65536];
  u32 size;
} Map;

// Function to insert a new entry into the map
__host__ void map_insert(Map* map, u32 key, u32 val) {
  map->keys[map->size] = key;
  map->vals[map->size] = val;
  map->size++;
}

// Function to lookup an id in the map by key
__host__ u32 map_lookup(Map* map, u32 key) {
  for (u32 i = 0; i < map->size; ++i) {
    if (map->keys[i] == key) {
      return map->vals[i];
    }
  }
  return map->size;
}

// Recursive function to print a term as a tree with unique variable IDs
__host__ void print_tree_go(Net* net, Ptr ptr, Map* var_ids) {
  if (is_var(ptr)) {
    u32 got = map_lookup(var_ids, ptr);
    if (got == var_ids->size) {
      u32 name = var_ids->size;
      Ptr targ = *target(net, enter(net, ptr));
      map_insert(var_ids, targ, name);
      printf("x%d", name);
    } else {
      printf("x%d", got);
    }
  } else if (is_ref(ptr)) {
    printf("{%x}", val(ptr));
  } else if (tag(ptr) == ERA) {
    printf("*");
  } else {
    switch (tag(ptr)) {
      case RDR: case RD1: case RD2:
        print_tree_go(net, *target(net, ptr), var_ids);
        break;
      default:
        printf("(%d ", tag(ptr) - CON);
        print_tree_go(net, net->node[val(ptr)].ports[P1], var_ids);
        printf(" ");
        print_tree_go(net, net->node[val(ptr)].ports[P2], var_ids);
        printf(")");
    }
  }
}

__host__ void print_tree(Net* net, Ptr ptr) {
  Map var_ids = { .size = 0 };
  print_tree_go(net, ptr, &var_ids);
  printf("\n");
}

// ~
// ~
// ~

// Tests
// -----

__host__ void populate(Book* book) {
  // E
  book->defs[0x0000000f]           = (Term*) malloc(sizeof(Term));
  book->defs[0x0000000f]->root     = 0xa0000000;
  book->defs[0x0000000f]->alen     = 0;
  book->defs[0x0000000f]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x0000000f]->nlen     = 3;
  book->defs[0x0000000f]->node     = (Node*) malloc(3 * sizeof(Node));
  book->defs[0x0000000f]->node[ 0] = (Node) {0x20000000,0xa0000001};
  book->defs[0x0000000f]->node[ 1] = (Node) {0x20000000,0xa0000002};
  book->defs[0x0000000f]->node[ 2] = (Node) {0x50000002,0x40000002};
  // F
  book->defs[0x00000010]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00000010]->root     = 0xa0000000;
  book->defs[0x00000010]->alen     = 0;
  book->defs[0x00000010]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00000010]->nlen     = 2;
  book->defs[0x00000010]->node     = (Node*) malloc(2 * sizeof(Node));
  book->defs[0x00000010]->node[ 0] = (Node) {0x20000000,0xa0000001};
  book->defs[0x00000010]->node[ 1] = (Node) {0x50000001,0x40000001};
  // I
  book->defs[0x00000013]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00000013]->root     = 0xa0000000;
  book->defs[0x00000013]->alen     = 0;
  book->defs[0x00000013]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00000013]->nlen     = 5;
  book->defs[0x00000013]->node     = (Node*) malloc(5 * sizeof(Node));
  book->defs[0x00000013]->node[ 0] = (Node) {0x40000003,0xa0000001};
  book->defs[0x00000013]->node[ 1] = (Node) {0x20000000,0xa0000002};
  book->defs[0x00000013]->node[ 2] = (Node) {0xa0000003,0xa0000004};
  book->defs[0x00000013]->node[ 3] = (Node) {0x40000000,0x50000004};
  book->defs[0x00000013]->node[ 4] = (Node) {0x20000000,0x50000003};
  // O
  book->defs[0x00000019]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00000019]->root     = 0xa0000000;
  book->defs[0x00000019]->alen     = 0;
  book->defs[0x00000019]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00000019]->nlen     = 5;
  book->defs[0x00000019]->node     = (Node*) malloc(5 * sizeof(Node));
  book->defs[0x00000019]->node[ 0] = (Node) {0x40000002,0xa0000001};
  book->defs[0x00000019]->node[ 1] = (Node) {0xa0000002,0xa0000003};
  book->defs[0x00000019]->node[ 2] = (Node) {0x40000000,0x50000004};
  book->defs[0x00000019]->node[ 3] = (Node) {0x20000000,0xa0000004};
  book->defs[0x00000019]->node[ 4] = (Node) {0x20000000,0x50000002};
  // S
  book->defs[0x0000001d]           = (Term*) malloc(sizeof(Term));
  book->defs[0x0000001d]->root     = 0xa0000000;
  book->defs[0x0000001d]->alen     = 0;
  book->defs[0x0000001d]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x0000001d]->nlen     = 4;
  book->defs[0x0000001d]->node     = (Node*) malloc(4 * sizeof(Node));
  book->defs[0x0000001d]->node[ 0] = (Node) {0x40000002,0xa0000001};
  book->defs[0x0000001d]->node[ 1] = (Node) {0xa0000002,0xa0000003};
  book->defs[0x0000001d]->node[ 2] = (Node) {0x40000000,0x50000003};
  book->defs[0x0000001d]->node[ 3] = (Node) {0x20000000,0x50000002};
  // T
  book->defs[0x0000001e]           = (Term*) malloc(sizeof(Term));
  book->defs[0x0000001e]->root     = 0xa0000000;
  book->defs[0x0000001e]->alen     = 0;
  book->defs[0x0000001e]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x0000001e]->nlen     = 2;
  book->defs[0x0000001e]->node     = (Node*) malloc(2 * sizeof(Node));
  book->defs[0x0000001e]->node[ 0] = (Node) {0x50000001,0xa0000001};
  book->defs[0x0000001e]->node[ 1] = (Node) {0x20000000,0x40000000};
  // Z
  book->defs[0x00000024]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00000024]->root     = 0xa0000000;
  book->defs[0x00000024]->alen     = 0;
  book->defs[0x00000024]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00000024]->nlen     = 2;
  book->defs[0x00000024]->node     = (Node*) malloc(2 * sizeof(Node));
  book->defs[0x00000024]->node[ 0] = (Node) {0x20000000,0xa0000001};
  book->defs[0x00000024]->node[ 1] = (Node) {0x50000001,0x40000001};
  // af
  book->defs[0x0000096a]           = (Term*) malloc(sizeof(Term));
  book->defs[0x0000096a]->root     = 0xa0000000;
  book->defs[0x0000096a]->alen     = 0;
  book->defs[0x0000096a]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x0000096a]->nlen     = 3;
  book->defs[0x0000096a]->node     = (Node*) malloc(3 * sizeof(Node));
  book->defs[0x0000096a]->node[ 0] = (Node) {0xa0000001,0x50000002};
  book->defs[0x0000096a]->node[ 1] = (Node) {0x10025a9d,0xa0000002};
  book->defs[0x0000096a]->node[ 2] = (Node) {0x10025aa4,0x50000000};
  // c0
  book->defs[0x000009c1]           = (Term*) malloc(sizeof(Term));
  book->defs[0x000009c1]->root     = 0xa0000000;
  book->defs[0x000009c1]->alen     = 0;
  book->defs[0x000009c1]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x000009c1]->nlen     = 2;
  book->defs[0x000009c1]->node     = (Node*) malloc(2 * sizeof(Node));
  book->defs[0x000009c1]->node[ 0] = (Node) {0x20000000,0xa0000001};
  book->defs[0x000009c1]->node[ 1] = (Node) {0x50000001,0x40000001};
  // c1
  book->defs[0x000009c2]           = (Term*) malloc(sizeof(Term));
  book->defs[0x000009c2]->root     = 0xa0000000;
  book->defs[0x000009c2]->alen     = 0;
  book->defs[0x000009c2]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x000009c2]->nlen     = 3;
  book->defs[0x000009c2]->node     = (Node*) malloc(3 * sizeof(Node));
  book->defs[0x000009c2]->node[ 0] = (Node) {0xa0000001,0xa0000002};
  book->defs[0x000009c2]->node[ 1] = (Node) {0x40000002,0x50000002};
  book->defs[0x000009c2]->node[ 2] = (Node) {0x40000001,0x50000001};
  // c2
  book->defs[0x000009c3]           = (Term*) malloc(sizeof(Term));
  book->defs[0x000009c3]->root     = 0xa0000000;
  book->defs[0x000009c3]->alen     = 0;
  book->defs[0x000009c3]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x000009c3]->nlen     = 5;
  book->defs[0x000009c3]->node     = (Node*) malloc(5 * sizeof(Node));
  book->defs[0x000009c3]->node[ 0] = (Node) {0xb0000001,0xa0000004};
  book->defs[0x000009c3]->node[ 1] = (Node) {0xa0000002,0xa0000003};
  book->defs[0x000009c3]->node[ 2] = (Node) {0x40000004,0x40000003};
  book->defs[0x000009c3]->node[ 3] = (Node) {0x50000002,0x50000004};
  book->defs[0x000009c3]->node[ 4] = (Node) {0x40000002,0x50000003};
  // c3
  book->defs[0x000009c4]           = (Term*) malloc(sizeof(Term));
  book->defs[0x000009c4]->root     = 0xa0000000;
  book->defs[0x000009c4]->alen     = 0;
  book->defs[0x000009c4]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x000009c4]->nlen     = 7;
  book->defs[0x000009c4]->node     = (Node*) malloc(7 * sizeof(Node));
  book->defs[0x000009c4]->node[ 0] = (Node) {0xb0000001,0xa0000006};
  book->defs[0x000009c4]->node[ 1] = (Node) {0xb0000002,0xa0000005};
  book->defs[0x000009c4]->node[ 2] = (Node) {0xa0000003,0xa0000004};
  book->defs[0x000009c4]->node[ 3] = (Node) {0x40000006,0x40000004};
  book->defs[0x000009c4]->node[ 4] = (Node) {0x50000003,0x40000005};
  book->defs[0x000009c4]->node[ 5] = (Node) {0x50000004,0x50000006};
  book->defs[0x000009c4]->node[ 6] = (Node) {0x40000003,0x50000005};
  // c4
  book->defs[0x000009c5]           = (Term*) malloc(sizeof(Term));
  book->defs[0x000009c5]->root     = 0xa0000000;
  book->defs[0x000009c5]->alen     = 0;
  book->defs[0x000009c5]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x000009c5]->nlen     = 9;
  book->defs[0x000009c5]->node     = (Node*) malloc(9 * sizeof(Node));
  book->defs[0x000009c5]->node[ 0] = (Node) {0xb0000001,0xa0000008};
  book->defs[0x000009c5]->node[ 1] = (Node) {0xb0000002,0xa0000007};
  book->defs[0x000009c5]->node[ 2] = (Node) {0xb0000003,0xa0000006};
  book->defs[0x000009c5]->node[ 3] = (Node) {0xa0000004,0xa0000005};
  book->defs[0x000009c5]->node[ 4] = (Node) {0x40000008,0x40000005};
  book->defs[0x000009c5]->node[ 5] = (Node) {0x50000004,0x40000006};
  book->defs[0x000009c5]->node[ 6] = (Node) {0x50000005,0x40000007};
  book->defs[0x000009c5]->node[ 7] = (Node) {0x50000006,0x50000008};
  book->defs[0x000009c5]->node[ 8] = (Node) {0x40000004,0x50000007};
  // c5
  book->defs[0x000009c6]           = (Term*) malloc(sizeof(Term));
  book->defs[0x000009c6]->root     = 0xa0000000;
  book->defs[0x000009c6]->alen     = 0;
  book->defs[0x000009c6]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x000009c6]->nlen     = 11;
  book->defs[0x000009c6]->node     = (Node*) malloc(11 * sizeof(Node));
  book->defs[0x000009c6]->node[ 0] = (Node) {0xb0000001,0xa000000a};
  book->defs[0x000009c6]->node[ 1] = (Node) {0xb0000002,0xa0000009};
  book->defs[0x000009c6]->node[ 2] = (Node) {0xb0000003,0xa0000008};
  book->defs[0x000009c6]->node[ 3] = (Node) {0xb0000004,0xa0000007};
  book->defs[0x000009c6]->node[ 4] = (Node) {0xa0000005,0xa0000006};
  book->defs[0x000009c6]->node[ 5] = (Node) {0x4000000a,0x40000006};
  book->defs[0x000009c6]->node[ 6] = (Node) {0x50000005,0x40000007};
  book->defs[0x000009c6]->node[ 7] = (Node) {0x50000006,0x40000008};
  book->defs[0x000009c6]->node[ 8] = (Node) {0x50000007,0x40000009};
  book->defs[0x000009c6]->node[ 9] = (Node) {0x50000008,0x5000000a};
  book->defs[0x000009c6]->node[10] = (Node) {0x40000005,0x50000009};
  // c6
  book->defs[0x000009c7]           = (Term*) malloc(sizeof(Term));
  book->defs[0x000009c7]->root     = 0xa0000000;
  book->defs[0x000009c7]->alen     = 0;
  book->defs[0x000009c7]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x000009c7]->nlen     = 13;
  book->defs[0x000009c7]->node     = (Node*) malloc(13 * sizeof(Node));
  book->defs[0x000009c7]->node[ 0] = (Node) {0xb0000001,0xa000000c};
  book->defs[0x000009c7]->node[ 1] = (Node) {0xb0000002,0xa000000b};
  book->defs[0x000009c7]->node[ 2] = (Node) {0xb0000003,0xa000000a};
  book->defs[0x000009c7]->node[ 3] = (Node) {0xb0000004,0xa0000009};
  book->defs[0x000009c7]->node[ 4] = (Node) {0xb0000005,0xa0000008};
  book->defs[0x000009c7]->node[ 5] = (Node) {0xa0000006,0xa0000007};
  book->defs[0x000009c7]->node[ 6] = (Node) {0x4000000c,0x40000007};
  book->defs[0x000009c7]->node[ 7] = (Node) {0x50000006,0x40000008};
  book->defs[0x000009c7]->node[ 8] = (Node) {0x50000007,0x40000009};
  book->defs[0x000009c7]->node[ 9] = (Node) {0x50000008,0x4000000a};
  book->defs[0x000009c7]->node[10] = (Node) {0x50000009,0x4000000b};
  book->defs[0x000009c7]->node[11] = (Node) {0x5000000a,0x5000000c};
  book->defs[0x000009c7]->node[12] = (Node) {0x40000006,0x5000000b};
  // c7
  book->defs[0x000009c8]           = (Term*) malloc(sizeof(Term));
  book->defs[0x000009c8]->root     = 0xa0000000;
  book->defs[0x000009c8]->alen     = 0;
  book->defs[0x000009c8]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x000009c8]->nlen     = 15;
  book->defs[0x000009c8]->node     = (Node*) malloc(15 * sizeof(Node));
  book->defs[0x000009c8]->node[ 0] = (Node) {0xb0000001,0xa000000e};
  book->defs[0x000009c8]->node[ 1] = (Node) {0xb0000002,0xa000000d};
  book->defs[0x000009c8]->node[ 2] = (Node) {0xb0000003,0xa000000c};
  book->defs[0x000009c8]->node[ 3] = (Node) {0xb0000004,0xa000000b};
  book->defs[0x000009c8]->node[ 4] = (Node) {0xb0000005,0xa000000a};
  book->defs[0x000009c8]->node[ 5] = (Node) {0xb0000006,0xa0000009};
  book->defs[0x000009c8]->node[ 6] = (Node) {0xa0000007,0xa0000008};
  book->defs[0x000009c8]->node[ 7] = (Node) {0x4000000e,0x40000008};
  book->defs[0x000009c8]->node[ 8] = (Node) {0x50000007,0x40000009};
  book->defs[0x000009c8]->node[ 9] = (Node) {0x50000008,0x4000000a};
  book->defs[0x000009c8]->node[10] = (Node) {0x50000009,0x4000000b};
  book->defs[0x000009c8]->node[11] = (Node) {0x5000000a,0x4000000c};
  book->defs[0x000009c8]->node[12] = (Node) {0x5000000b,0x4000000d};
  book->defs[0x000009c8]->node[13] = (Node) {0x5000000c,0x5000000e};
  book->defs[0x000009c8]->node[14] = (Node) {0x40000007,0x5000000d};
  // c8
  book->defs[0x000009c9]           = (Term*) malloc(sizeof(Term));
  book->defs[0x000009c9]->root     = 0xa0000000;
  book->defs[0x000009c9]->alen     = 0;
  book->defs[0x000009c9]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x000009c9]->nlen     = 17;
  book->defs[0x000009c9]->node     = (Node*) malloc(17 * sizeof(Node));
  book->defs[0x000009c9]->node[ 0] = (Node) {0xb0000001,0xa0000010};
  book->defs[0x000009c9]->node[ 1] = (Node) {0xb0000002,0xa000000f};
  book->defs[0x000009c9]->node[ 2] = (Node) {0xb0000003,0xa000000e};
  book->defs[0x000009c9]->node[ 3] = (Node) {0xb0000004,0xa000000d};
  book->defs[0x000009c9]->node[ 4] = (Node) {0xb0000005,0xa000000c};
  book->defs[0x000009c9]->node[ 5] = (Node) {0xb0000006,0xa000000b};
  book->defs[0x000009c9]->node[ 6] = (Node) {0xb0000007,0xa000000a};
  book->defs[0x000009c9]->node[ 7] = (Node) {0xa0000008,0xa0000009};
  book->defs[0x000009c9]->node[ 8] = (Node) {0x40000010,0x40000009};
  book->defs[0x000009c9]->node[ 9] = (Node) {0x50000008,0x4000000a};
  book->defs[0x000009c9]->node[10] = (Node) {0x50000009,0x4000000b};
  book->defs[0x000009c9]->node[11] = (Node) {0x5000000a,0x4000000c};
  book->defs[0x000009c9]->node[12] = (Node) {0x5000000b,0x4000000d};
  book->defs[0x000009c9]->node[13] = (Node) {0x5000000c,0x4000000e};
  book->defs[0x000009c9]->node[14] = (Node) {0x5000000d,0x4000000f};
  book->defs[0x000009c9]->node[15] = (Node) {0x5000000e,0x50000010};
  book->defs[0x000009c9]->node[16] = (Node) {0x40000008,0x5000000f};
  // c9
  book->defs[0x000009ca]           = (Term*) malloc(sizeof(Term));
  book->defs[0x000009ca]->root     = 0xa0000000;
  book->defs[0x000009ca]->alen     = 0;
  book->defs[0x000009ca]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x000009ca]->nlen     = 19;
  book->defs[0x000009ca]->node     = (Node*) malloc(19 * sizeof(Node));
  book->defs[0x000009ca]->node[ 0] = (Node) {0xb0000001,0xa0000012};
  book->defs[0x000009ca]->node[ 1] = (Node) {0xb0000002,0xa0000011};
  book->defs[0x000009ca]->node[ 2] = (Node) {0xb0000003,0xa0000010};
  book->defs[0x000009ca]->node[ 3] = (Node) {0xb0000004,0xa000000f};
  book->defs[0x000009ca]->node[ 4] = (Node) {0xb0000005,0xa000000e};
  book->defs[0x000009ca]->node[ 5] = (Node) {0xb0000006,0xa000000d};
  book->defs[0x000009ca]->node[ 6] = (Node) {0xb0000007,0xa000000c};
  book->defs[0x000009ca]->node[ 7] = (Node) {0xb0000008,0xa000000b};
  book->defs[0x000009ca]->node[ 8] = (Node) {0xa0000009,0xa000000a};
  book->defs[0x000009ca]->node[ 9] = (Node) {0x40000012,0x4000000a};
  book->defs[0x000009ca]->node[10] = (Node) {0x50000009,0x4000000b};
  book->defs[0x000009ca]->node[11] = (Node) {0x5000000a,0x4000000c};
  book->defs[0x000009ca]->node[12] = (Node) {0x5000000b,0x4000000d};
  book->defs[0x000009ca]->node[13] = (Node) {0x5000000c,0x4000000e};
  book->defs[0x000009ca]->node[14] = (Node) {0x5000000d,0x4000000f};
  book->defs[0x000009ca]->node[15] = (Node) {0x5000000e,0x40000010};
  book->defs[0x000009ca]->node[16] = (Node) {0x5000000f,0x40000011};
  book->defs[0x000009ca]->node[17] = (Node) {0x50000010,0x50000012};
  book->defs[0x000009ca]->node[18] = (Node) {0x40000009,0x50000011};
  // id
  book->defs[0x00000b68]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00000b68]->root     = 0xa0000000;
  book->defs[0x00000b68]->alen     = 0;
  book->defs[0x00000b68]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00000b68]->nlen     = 1;
  book->defs[0x00000b68]->node     = (Node*) malloc(1 * sizeof(Node));
  book->defs[0x00000b68]->node[ 0] = (Node) {0x50000000,0x40000000};
  // k0
  book->defs[0x00000bc1]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00000bc1]->root     = 0xa0000000;
  book->defs[0x00000bc1]->alen     = 0;
  book->defs[0x00000bc1]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00000bc1]->nlen     = 2;
  book->defs[0x00000bc1]->node     = (Node*) malloc(2 * sizeof(Node));
  book->defs[0x00000bc1]->node[ 0] = (Node) {0x20000000,0xa0000001};
  book->defs[0x00000bc1]->node[ 1] = (Node) {0x50000001,0x40000001};
  // k1
  book->defs[0x00000bc2]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00000bc2]->root     = 0xa0000000;
  book->defs[0x00000bc2]->alen     = 0;
  book->defs[0x00000bc2]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00000bc2]->nlen     = 3;
  book->defs[0x00000bc2]->node     = (Node*) malloc(3 * sizeof(Node));
  book->defs[0x00000bc2]->node[ 0] = (Node) {0xa0000001,0xa0000002};
  book->defs[0x00000bc2]->node[ 1] = (Node) {0x40000002,0x50000002};
  book->defs[0x00000bc2]->node[ 2] = (Node) {0x40000001,0x50000001};
  // k2
  book->defs[0x00000bc3]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00000bc3]->root     = 0xa0000000;
  book->defs[0x00000bc3]->alen     = 0;
  book->defs[0x00000bc3]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00000bc3]->nlen     = 5;
  book->defs[0x00000bc3]->node     = (Node*) malloc(5 * sizeof(Node));
  book->defs[0x00000bc3]->node[ 0] = (Node) {0xc0000001,0xa0000004};
  book->defs[0x00000bc3]->node[ 1] = (Node) {0xa0000002,0xa0000003};
  book->defs[0x00000bc3]->node[ 2] = (Node) {0x40000004,0x40000003};
  book->defs[0x00000bc3]->node[ 3] = (Node) {0x50000002,0x50000004};
  book->defs[0x00000bc3]->node[ 4] = (Node) {0x40000002,0x50000003};
  // k3
  book->defs[0x00000bc4]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00000bc4]->root     = 0xa0000000;
  book->defs[0x00000bc4]->alen     = 0;
  book->defs[0x00000bc4]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00000bc4]->nlen     = 7;
  book->defs[0x00000bc4]->node     = (Node*) malloc(7 * sizeof(Node));
  book->defs[0x00000bc4]->node[ 0] = (Node) {0xc0000001,0xa0000006};
  book->defs[0x00000bc4]->node[ 1] = (Node) {0xc0000002,0xa0000005};
  book->defs[0x00000bc4]->node[ 2] = (Node) {0xa0000003,0xa0000004};
  book->defs[0x00000bc4]->node[ 3] = (Node) {0x40000006,0x40000004};
  book->defs[0x00000bc4]->node[ 4] = (Node) {0x50000003,0x40000005};
  book->defs[0x00000bc4]->node[ 5] = (Node) {0x50000004,0x50000006};
  book->defs[0x00000bc4]->node[ 6] = (Node) {0x40000003,0x50000005};
  // k4
  book->defs[0x00000bc5]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00000bc5]->root     = 0xa0000000;
  book->defs[0x00000bc5]->alen     = 0;
  book->defs[0x00000bc5]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00000bc5]->nlen     = 9;
  book->defs[0x00000bc5]->node     = (Node*) malloc(9 * sizeof(Node));
  book->defs[0x00000bc5]->node[ 0] = (Node) {0xc0000001,0xa0000008};
  book->defs[0x00000bc5]->node[ 1] = (Node) {0xc0000002,0xa0000007};
  book->defs[0x00000bc5]->node[ 2] = (Node) {0xc0000003,0xa0000006};
  book->defs[0x00000bc5]->node[ 3] = (Node) {0xa0000004,0xa0000005};
  book->defs[0x00000bc5]->node[ 4] = (Node) {0x40000008,0x40000005};
  book->defs[0x00000bc5]->node[ 5] = (Node) {0x50000004,0x40000006};
  book->defs[0x00000bc5]->node[ 6] = (Node) {0x50000005,0x40000007};
  book->defs[0x00000bc5]->node[ 7] = (Node) {0x50000006,0x50000008};
  book->defs[0x00000bc5]->node[ 8] = (Node) {0x40000004,0x50000007};
  // k5
  book->defs[0x00000bc6]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00000bc6]->root     = 0xa0000000;
  book->defs[0x00000bc6]->alen     = 0;
  book->defs[0x00000bc6]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00000bc6]->nlen     = 11;
  book->defs[0x00000bc6]->node     = (Node*) malloc(11 * sizeof(Node));
  book->defs[0x00000bc6]->node[ 0] = (Node) {0xc0000001,0xa000000a};
  book->defs[0x00000bc6]->node[ 1] = (Node) {0xc0000002,0xa0000009};
  book->defs[0x00000bc6]->node[ 2] = (Node) {0xc0000003,0xa0000008};
  book->defs[0x00000bc6]->node[ 3] = (Node) {0xc0000004,0xa0000007};
  book->defs[0x00000bc6]->node[ 4] = (Node) {0xa0000005,0xa0000006};
  book->defs[0x00000bc6]->node[ 5] = (Node) {0x4000000a,0x40000006};
  book->defs[0x00000bc6]->node[ 6] = (Node) {0x50000005,0x40000007};
  book->defs[0x00000bc6]->node[ 7] = (Node) {0x50000006,0x40000008};
  book->defs[0x00000bc6]->node[ 8] = (Node) {0x50000007,0x40000009};
  book->defs[0x00000bc6]->node[ 9] = (Node) {0x50000008,0x5000000a};
  book->defs[0x00000bc6]->node[10] = (Node) {0x40000005,0x50000009};
  // k6
  book->defs[0x00000bc7]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00000bc7]->root     = 0xa0000000;
  book->defs[0x00000bc7]->alen     = 0;
  book->defs[0x00000bc7]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00000bc7]->nlen     = 13;
  book->defs[0x00000bc7]->node     = (Node*) malloc(13 * sizeof(Node));
  book->defs[0x00000bc7]->node[ 0] = (Node) {0xc0000001,0xa000000c};
  book->defs[0x00000bc7]->node[ 1] = (Node) {0xc0000002,0xa000000b};
  book->defs[0x00000bc7]->node[ 2] = (Node) {0xc0000003,0xa000000a};
  book->defs[0x00000bc7]->node[ 3] = (Node) {0xc0000004,0xa0000009};
  book->defs[0x00000bc7]->node[ 4] = (Node) {0xc0000005,0xa0000008};
  book->defs[0x00000bc7]->node[ 5] = (Node) {0xa0000006,0xa0000007};
  book->defs[0x00000bc7]->node[ 6] = (Node) {0x4000000c,0x40000007};
  book->defs[0x00000bc7]->node[ 7] = (Node) {0x50000006,0x40000008};
  book->defs[0x00000bc7]->node[ 8] = (Node) {0x50000007,0x40000009};
  book->defs[0x00000bc7]->node[ 9] = (Node) {0x50000008,0x4000000a};
  book->defs[0x00000bc7]->node[10] = (Node) {0x50000009,0x4000000b};
  book->defs[0x00000bc7]->node[11] = (Node) {0x5000000a,0x5000000c};
  book->defs[0x00000bc7]->node[12] = (Node) {0x40000006,0x5000000b};
  // k7
  book->defs[0x00000bc8]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00000bc8]->root     = 0xa0000000;
  book->defs[0x00000bc8]->alen     = 0;
  book->defs[0x00000bc8]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00000bc8]->nlen     = 15;
  book->defs[0x00000bc8]->node     = (Node*) malloc(15 * sizeof(Node));
  book->defs[0x00000bc8]->node[ 0] = (Node) {0xc0000001,0xa000000e};
  book->defs[0x00000bc8]->node[ 1] = (Node) {0xc0000002,0xa000000d};
  book->defs[0x00000bc8]->node[ 2] = (Node) {0xc0000003,0xa000000c};
  book->defs[0x00000bc8]->node[ 3] = (Node) {0xc0000004,0xa000000b};
  book->defs[0x00000bc8]->node[ 4] = (Node) {0xc0000005,0xa000000a};
  book->defs[0x00000bc8]->node[ 5] = (Node) {0xc0000006,0xa0000009};
  book->defs[0x00000bc8]->node[ 6] = (Node) {0xa0000007,0xa0000008};
  book->defs[0x00000bc8]->node[ 7] = (Node) {0x4000000e,0x40000008};
  book->defs[0x00000bc8]->node[ 8] = (Node) {0x50000007,0x40000009};
  book->defs[0x00000bc8]->node[ 9] = (Node) {0x50000008,0x4000000a};
  book->defs[0x00000bc8]->node[10] = (Node) {0x50000009,0x4000000b};
  book->defs[0x00000bc8]->node[11] = (Node) {0x5000000a,0x4000000c};
  book->defs[0x00000bc8]->node[12] = (Node) {0x5000000b,0x4000000d};
  book->defs[0x00000bc8]->node[13] = (Node) {0x5000000c,0x5000000e};
  book->defs[0x00000bc8]->node[14] = (Node) {0x40000007,0x5000000d};
  // k8
  book->defs[0x00000bc9]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00000bc9]->root     = 0xa0000000;
  book->defs[0x00000bc9]->alen     = 0;
  book->defs[0x00000bc9]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00000bc9]->nlen     = 17;
  book->defs[0x00000bc9]->node     = (Node*) malloc(17 * sizeof(Node));
  book->defs[0x00000bc9]->node[ 0] = (Node) {0xc0000001,0xa0000010};
  book->defs[0x00000bc9]->node[ 1] = (Node) {0xc0000002,0xa000000f};
  book->defs[0x00000bc9]->node[ 2] = (Node) {0xc0000003,0xa000000e};
  book->defs[0x00000bc9]->node[ 3] = (Node) {0xc0000004,0xa000000d};
  book->defs[0x00000bc9]->node[ 4] = (Node) {0xc0000005,0xa000000c};
  book->defs[0x00000bc9]->node[ 5] = (Node) {0xc0000006,0xa000000b};
  book->defs[0x00000bc9]->node[ 6] = (Node) {0xc0000007,0xa000000a};
  book->defs[0x00000bc9]->node[ 7] = (Node) {0xa0000008,0xa0000009};
  book->defs[0x00000bc9]->node[ 8] = (Node) {0x40000010,0x40000009};
  book->defs[0x00000bc9]->node[ 9] = (Node) {0x50000008,0x4000000a};
  book->defs[0x00000bc9]->node[10] = (Node) {0x50000009,0x4000000b};
  book->defs[0x00000bc9]->node[11] = (Node) {0x5000000a,0x4000000c};
  book->defs[0x00000bc9]->node[12] = (Node) {0x5000000b,0x4000000d};
  book->defs[0x00000bc9]->node[13] = (Node) {0x5000000c,0x4000000e};
  book->defs[0x00000bc9]->node[14] = (Node) {0x5000000d,0x4000000f};
  book->defs[0x00000bc9]->node[15] = (Node) {0x5000000e,0x50000010};
  book->defs[0x00000bc9]->node[16] = (Node) {0x40000008,0x5000000f};
  // k9
  book->defs[0x00000bca]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00000bca]->root     = 0xa0000000;
  book->defs[0x00000bca]->alen     = 0;
  book->defs[0x00000bca]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00000bca]->nlen     = 19;
  book->defs[0x00000bca]->node     = (Node*) malloc(19 * sizeof(Node));
  book->defs[0x00000bca]->node[ 0] = (Node) {0xc0000001,0xa0000012};
  book->defs[0x00000bca]->node[ 1] = (Node) {0xc0000002,0xa0000011};
  book->defs[0x00000bca]->node[ 2] = (Node) {0xc0000003,0xa0000010};
  book->defs[0x00000bca]->node[ 3] = (Node) {0xc0000004,0xa000000f};
  book->defs[0x00000bca]->node[ 4] = (Node) {0xc0000005,0xa000000e};
  book->defs[0x00000bca]->node[ 5] = (Node) {0xc0000006,0xa000000d};
  book->defs[0x00000bca]->node[ 6] = (Node) {0xc0000007,0xa000000c};
  book->defs[0x00000bca]->node[ 7] = (Node) {0xc0000008,0xa000000b};
  book->defs[0x00000bca]->node[ 8] = (Node) {0xa0000009,0xa000000a};
  book->defs[0x00000bca]->node[ 9] = (Node) {0x40000012,0x4000000a};
  book->defs[0x00000bca]->node[10] = (Node) {0x50000009,0x4000000b};
  book->defs[0x00000bca]->node[11] = (Node) {0x5000000a,0x4000000c};
  book->defs[0x00000bca]->node[12] = (Node) {0x5000000b,0x4000000d};
  book->defs[0x00000bca]->node[13] = (Node) {0x5000000c,0x4000000e};
  book->defs[0x00000bca]->node[14] = (Node) {0x5000000d,0x4000000f};
  book->defs[0x00000bca]->node[15] = (Node) {0x5000000e,0x40000010};
  book->defs[0x00000bca]->node[16] = (Node) {0x5000000f,0x40000011};
  book->defs[0x00000bca]->node[17] = (Node) {0x50000010,0x50000012};
  book->defs[0x00000bca]->node[18] = (Node) {0x40000009,0x50000011};
  // afS
  book->defs[0x00025a9d]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00025a9d]->root     = 0xa0000000;
  book->defs[0x00025a9d]->alen     = 3;
  book->defs[0x00025a9d]->acts     = (Wire*) malloc(3 * sizeof(Wire));
  book->defs[0x00025a9d]->acts[ 0] = mkwire(0xa0000002,0x1000096a);
  book->defs[0x00025a9d]->acts[ 1] = mkwire(0xa0000003,0x10025ca8);
  book->defs[0x00025a9d]->acts[ 2] = mkwire(0xa0000005,0x1000096a);
  book->defs[0x00025a9d]->nlen     = 6;
  book->defs[0x00025a9d]->node     = (Node*) malloc(6 * sizeof(Node));
  book->defs[0x00025a9d]->node[ 0] = (Node) {0xb0000001,0x50000004};
  book->defs[0x00025a9d]->node[ 1] = (Node) {0x40000005,0x40000002};
  book->defs[0x00025a9d]->node[ 2] = (Node) {0x50000001,0x40000004};
  book->defs[0x00025a9d]->node[ 3] = (Node) {0x50000005,0xa0000004};
  book->defs[0x00025a9d]->node[ 4] = (Node) {0x50000002,0x50000000};
  book->defs[0x00025a9d]->node[ 5] = (Node) {0x40000001,0x40000003};
  // afZ
  book->defs[0x00025aa4]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00025aa4]->root     = 0x1000001e;
  book->defs[0x00025aa4]->alen     = 0;
  book->defs[0x00025aa4]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00025aa4]->nlen     = 0;
  book->defs[0x00025aa4]->node     = (Node*) malloc(0 * sizeof(Node));
  // and
  book->defs[0x00025ca8]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00025ca8]->root     = 0xa0000000;
  book->defs[0x00025ca8]->alen     = 0;
  book->defs[0x00025ca8]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00025ca8]->nlen     = 9;
  book->defs[0x00025ca8]->node     = (Node*) malloc(9 * sizeof(Node));
  book->defs[0x00025ca8]->node[ 0] = (Node) {0xa0000001,0x50000005};
  book->defs[0x00025ca8]->node[ 1] = (Node) {0xa0000002,0xa0000005};
  book->defs[0x00025ca8]->node[ 2] = (Node) {0xa0000003,0x50000004};
  book->defs[0x00025ca8]->node[ 3] = (Node) {0x1000001e,0xa0000004};
  book->defs[0x00025ca8]->node[ 4] = (Node) {0x10000010,0x50000002};
  book->defs[0x00025ca8]->node[ 5] = (Node) {0xa0000006,0x50000000};
  book->defs[0x00025ca8]->node[ 6] = (Node) {0xa0000007,0x50000008};
  book->defs[0x00025ca8]->node[ 7] = (Node) {0x10000010,0xa0000008};
  book->defs[0x00025ca8]->node[ 8] = (Node) {0x10000010,0x50000006};
  // brn
  book->defs[0x00026db2]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00026db2]->root     = 0xa0000000;
  book->defs[0x00026db2]->alen     = 0;
  book->defs[0x00026db2]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00026db2]->nlen     = 3;
  book->defs[0x00026db2]->node     = (Node*) malloc(3 * sizeof(Node));
  book->defs[0x00026db2]->node[ 0] = (Node) {0xa0000001,0x50000002};
  book->defs[0x00026db2]->node[ 1] = (Node) {0x109b6c9d,0xa0000002};
  book->defs[0x00026db2]->node[ 2] = (Node) {0x109b6ca4,0x50000000};
  // c10
  book->defs[0x00027081]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00027081]->root     = 0xa0000000;
  book->defs[0x00027081]->alen     = 0;
  book->defs[0x00027081]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00027081]->nlen     = 21;
  book->defs[0x00027081]->node     = (Node*) malloc(21 * sizeof(Node));
  book->defs[0x00027081]->node[ 0] = (Node) {0xb0000001,0xa0000014};
  book->defs[0x00027081]->node[ 1] = (Node) {0xb0000002,0xa0000013};
  book->defs[0x00027081]->node[ 2] = (Node) {0xb0000003,0xa0000012};
  book->defs[0x00027081]->node[ 3] = (Node) {0xb0000004,0xa0000011};
  book->defs[0x00027081]->node[ 4] = (Node) {0xb0000005,0xa0000010};
  book->defs[0x00027081]->node[ 5] = (Node) {0xb0000006,0xa000000f};
  book->defs[0x00027081]->node[ 6] = (Node) {0xb0000007,0xa000000e};
  book->defs[0x00027081]->node[ 7] = (Node) {0xb0000008,0xa000000d};
  book->defs[0x00027081]->node[ 8] = (Node) {0xb0000009,0xa000000c};
  book->defs[0x00027081]->node[ 9] = (Node) {0xa000000a,0xa000000b};
  book->defs[0x00027081]->node[10] = (Node) {0x40000014,0x4000000b};
  book->defs[0x00027081]->node[11] = (Node) {0x5000000a,0x4000000c};
  book->defs[0x00027081]->node[12] = (Node) {0x5000000b,0x4000000d};
  book->defs[0x00027081]->node[13] = (Node) {0x5000000c,0x4000000e};
  book->defs[0x00027081]->node[14] = (Node) {0x5000000d,0x4000000f};
  book->defs[0x00027081]->node[15] = (Node) {0x5000000e,0x40000010};
  book->defs[0x00027081]->node[16] = (Node) {0x5000000f,0x40000011};
  book->defs[0x00027081]->node[17] = (Node) {0x50000010,0x40000012};
  book->defs[0x00027081]->node[18] = (Node) {0x50000011,0x40000013};
  book->defs[0x00027081]->node[19] = (Node) {0x50000012,0x50000014};
  book->defs[0x00027081]->node[20] = (Node) {0x4000000a,0x50000013};
  // c11
  book->defs[0x00027082]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00027082]->root     = 0xa0000000;
  book->defs[0x00027082]->alen     = 0;
  book->defs[0x00027082]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00027082]->nlen     = 23;
  book->defs[0x00027082]->node     = (Node*) malloc(23 * sizeof(Node));
  book->defs[0x00027082]->node[ 0] = (Node) {0xb0000001,0xa0000016};
  book->defs[0x00027082]->node[ 1] = (Node) {0xb0000002,0xa0000015};
  book->defs[0x00027082]->node[ 2] = (Node) {0xb0000003,0xa0000014};
  book->defs[0x00027082]->node[ 3] = (Node) {0xb0000004,0xa0000013};
  book->defs[0x00027082]->node[ 4] = (Node) {0xb0000005,0xa0000012};
  book->defs[0x00027082]->node[ 5] = (Node) {0xb0000006,0xa0000011};
  book->defs[0x00027082]->node[ 6] = (Node) {0xb0000007,0xa0000010};
  book->defs[0x00027082]->node[ 7] = (Node) {0xb0000008,0xa000000f};
  book->defs[0x00027082]->node[ 8] = (Node) {0xb0000009,0xa000000e};
  book->defs[0x00027082]->node[ 9] = (Node) {0xb000000a,0xa000000d};
  book->defs[0x00027082]->node[10] = (Node) {0xa000000b,0xa000000c};
  book->defs[0x00027082]->node[11] = (Node) {0x40000016,0x4000000c};
  book->defs[0x00027082]->node[12] = (Node) {0x5000000b,0x4000000d};
  book->defs[0x00027082]->node[13] = (Node) {0x5000000c,0x4000000e};
  book->defs[0x00027082]->node[14] = (Node) {0x5000000d,0x4000000f};
  book->defs[0x00027082]->node[15] = (Node) {0x5000000e,0x40000010};
  book->defs[0x00027082]->node[16] = (Node) {0x5000000f,0x40000011};
  book->defs[0x00027082]->node[17] = (Node) {0x50000010,0x40000012};
  book->defs[0x00027082]->node[18] = (Node) {0x50000011,0x40000013};
  book->defs[0x00027082]->node[19] = (Node) {0x50000012,0x40000014};
  book->defs[0x00027082]->node[20] = (Node) {0x50000013,0x40000015};
  book->defs[0x00027082]->node[21] = (Node) {0x50000014,0x50000016};
  book->defs[0x00027082]->node[22] = (Node) {0x4000000b,0x50000015};
  // c12
  book->defs[0x00027083]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00027083]->root     = 0xa0000000;
  book->defs[0x00027083]->alen     = 0;
  book->defs[0x00027083]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00027083]->nlen     = 25;
  book->defs[0x00027083]->node     = (Node*) malloc(25 * sizeof(Node));
  book->defs[0x00027083]->node[ 0] = (Node) {0xb0000001,0xa0000018};
  book->defs[0x00027083]->node[ 1] = (Node) {0xb0000002,0xa0000017};
  book->defs[0x00027083]->node[ 2] = (Node) {0xb0000003,0xa0000016};
  book->defs[0x00027083]->node[ 3] = (Node) {0xb0000004,0xa0000015};
  book->defs[0x00027083]->node[ 4] = (Node) {0xb0000005,0xa0000014};
  book->defs[0x00027083]->node[ 5] = (Node) {0xb0000006,0xa0000013};
  book->defs[0x00027083]->node[ 6] = (Node) {0xb0000007,0xa0000012};
  book->defs[0x00027083]->node[ 7] = (Node) {0xb0000008,0xa0000011};
  book->defs[0x00027083]->node[ 8] = (Node) {0xb0000009,0xa0000010};
  book->defs[0x00027083]->node[ 9] = (Node) {0xb000000a,0xa000000f};
  book->defs[0x00027083]->node[10] = (Node) {0xb000000b,0xa000000e};
  book->defs[0x00027083]->node[11] = (Node) {0xa000000c,0xa000000d};
  book->defs[0x00027083]->node[12] = (Node) {0x40000018,0x4000000d};
  book->defs[0x00027083]->node[13] = (Node) {0x5000000c,0x4000000e};
  book->defs[0x00027083]->node[14] = (Node) {0x5000000d,0x4000000f};
  book->defs[0x00027083]->node[15] = (Node) {0x5000000e,0x40000010};
  book->defs[0x00027083]->node[16] = (Node) {0x5000000f,0x40000011};
  book->defs[0x00027083]->node[17] = (Node) {0x50000010,0x40000012};
  book->defs[0x00027083]->node[18] = (Node) {0x50000011,0x40000013};
  book->defs[0x00027083]->node[19] = (Node) {0x50000012,0x40000014};
  book->defs[0x00027083]->node[20] = (Node) {0x50000013,0x40000015};
  book->defs[0x00027083]->node[21] = (Node) {0x50000014,0x40000016};
  book->defs[0x00027083]->node[22] = (Node) {0x50000015,0x40000017};
  book->defs[0x00027083]->node[23] = (Node) {0x50000016,0x50000018};
  book->defs[0x00027083]->node[24] = (Node) {0x4000000c,0x50000017};
  // c13
  book->defs[0x00027084]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00027084]->root     = 0xa0000000;
  book->defs[0x00027084]->alen     = 0;
  book->defs[0x00027084]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00027084]->nlen     = 27;
  book->defs[0x00027084]->node     = (Node*) malloc(27 * sizeof(Node));
  book->defs[0x00027084]->node[ 0] = (Node) {0xb0000001,0xa000001a};
  book->defs[0x00027084]->node[ 1] = (Node) {0xb0000002,0xa0000019};
  book->defs[0x00027084]->node[ 2] = (Node) {0xb0000003,0xa0000018};
  book->defs[0x00027084]->node[ 3] = (Node) {0xb0000004,0xa0000017};
  book->defs[0x00027084]->node[ 4] = (Node) {0xb0000005,0xa0000016};
  book->defs[0x00027084]->node[ 5] = (Node) {0xb0000006,0xa0000015};
  book->defs[0x00027084]->node[ 6] = (Node) {0xb0000007,0xa0000014};
  book->defs[0x00027084]->node[ 7] = (Node) {0xb0000008,0xa0000013};
  book->defs[0x00027084]->node[ 8] = (Node) {0xb0000009,0xa0000012};
  book->defs[0x00027084]->node[ 9] = (Node) {0xb000000a,0xa0000011};
  book->defs[0x00027084]->node[10] = (Node) {0xb000000b,0xa0000010};
  book->defs[0x00027084]->node[11] = (Node) {0xb000000c,0xa000000f};
  book->defs[0x00027084]->node[12] = (Node) {0xa000000d,0xa000000e};
  book->defs[0x00027084]->node[13] = (Node) {0x4000001a,0x4000000e};
  book->defs[0x00027084]->node[14] = (Node) {0x5000000d,0x4000000f};
  book->defs[0x00027084]->node[15] = (Node) {0x5000000e,0x40000010};
  book->defs[0x00027084]->node[16] = (Node) {0x5000000f,0x40000011};
  book->defs[0x00027084]->node[17] = (Node) {0x50000010,0x40000012};
  book->defs[0x00027084]->node[18] = (Node) {0x50000011,0x40000013};
  book->defs[0x00027084]->node[19] = (Node) {0x50000012,0x40000014};
  book->defs[0x00027084]->node[20] = (Node) {0x50000013,0x40000015};
  book->defs[0x00027084]->node[21] = (Node) {0x50000014,0x40000016};
  book->defs[0x00027084]->node[22] = (Node) {0x50000015,0x40000017};
  book->defs[0x00027084]->node[23] = (Node) {0x50000016,0x40000018};
  book->defs[0x00027084]->node[24] = (Node) {0x50000017,0x40000019};
  book->defs[0x00027084]->node[25] = (Node) {0x50000018,0x5000001a};
  book->defs[0x00027084]->node[26] = (Node) {0x4000000d,0x50000019};
  // c14
  book->defs[0x00027085]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00027085]->root     = 0xa0000000;
  book->defs[0x00027085]->alen     = 0;
  book->defs[0x00027085]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00027085]->nlen     = 29;
  book->defs[0x00027085]->node     = (Node*) malloc(29 * sizeof(Node));
  book->defs[0x00027085]->node[ 0] = (Node) {0xb0000001,0xa000001c};
  book->defs[0x00027085]->node[ 1] = (Node) {0xb0000002,0xa000001b};
  book->defs[0x00027085]->node[ 2] = (Node) {0xb0000003,0xa000001a};
  book->defs[0x00027085]->node[ 3] = (Node) {0xb0000004,0xa0000019};
  book->defs[0x00027085]->node[ 4] = (Node) {0xb0000005,0xa0000018};
  book->defs[0x00027085]->node[ 5] = (Node) {0xb0000006,0xa0000017};
  book->defs[0x00027085]->node[ 6] = (Node) {0xb0000007,0xa0000016};
  book->defs[0x00027085]->node[ 7] = (Node) {0xb0000008,0xa0000015};
  book->defs[0x00027085]->node[ 8] = (Node) {0xb0000009,0xa0000014};
  book->defs[0x00027085]->node[ 9] = (Node) {0xb000000a,0xa0000013};
  book->defs[0x00027085]->node[10] = (Node) {0xb000000b,0xa0000012};
  book->defs[0x00027085]->node[11] = (Node) {0xb000000c,0xa0000011};
  book->defs[0x00027085]->node[12] = (Node) {0xb000000d,0xa0000010};
  book->defs[0x00027085]->node[13] = (Node) {0xa000000e,0xa000000f};
  book->defs[0x00027085]->node[14] = (Node) {0x4000001c,0x4000000f};
  book->defs[0x00027085]->node[15] = (Node) {0x5000000e,0x40000010};
  book->defs[0x00027085]->node[16] = (Node) {0x5000000f,0x40000011};
  book->defs[0x00027085]->node[17] = (Node) {0x50000010,0x40000012};
  book->defs[0x00027085]->node[18] = (Node) {0x50000011,0x40000013};
  book->defs[0x00027085]->node[19] = (Node) {0x50000012,0x40000014};
  book->defs[0x00027085]->node[20] = (Node) {0x50000013,0x40000015};
  book->defs[0x00027085]->node[21] = (Node) {0x50000014,0x40000016};
  book->defs[0x00027085]->node[22] = (Node) {0x50000015,0x40000017};
  book->defs[0x00027085]->node[23] = (Node) {0x50000016,0x40000018};
  book->defs[0x00027085]->node[24] = (Node) {0x50000017,0x40000019};
  book->defs[0x00027085]->node[25] = (Node) {0x50000018,0x4000001a};
  book->defs[0x00027085]->node[26] = (Node) {0x50000019,0x4000001b};
  book->defs[0x00027085]->node[27] = (Node) {0x5000001a,0x5000001c};
  book->defs[0x00027085]->node[28] = (Node) {0x4000000e,0x5000001b};
  // c15
  book->defs[0x00027086]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00027086]->root     = 0xa0000000;
  book->defs[0x00027086]->alen     = 0;
  book->defs[0x00027086]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00027086]->nlen     = 31;
  book->defs[0x00027086]->node     = (Node*) malloc(31 * sizeof(Node));
  book->defs[0x00027086]->node[ 0] = (Node) {0xb0000001,0xa000001e};
  book->defs[0x00027086]->node[ 1] = (Node) {0xb0000002,0xa000001d};
  book->defs[0x00027086]->node[ 2] = (Node) {0xb0000003,0xa000001c};
  book->defs[0x00027086]->node[ 3] = (Node) {0xb0000004,0xa000001b};
  book->defs[0x00027086]->node[ 4] = (Node) {0xb0000005,0xa000001a};
  book->defs[0x00027086]->node[ 5] = (Node) {0xb0000006,0xa0000019};
  book->defs[0x00027086]->node[ 6] = (Node) {0xb0000007,0xa0000018};
  book->defs[0x00027086]->node[ 7] = (Node) {0xb0000008,0xa0000017};
  book->defs[0x00027086]->node[ 8] = (Node) {0xb0000009,0xa0000016};
  book->defs[0x00027086]->node[ 9] = (Node) {0xb000000a,0xa0000015};
  book->defs[0x00027086]->node[10] = (Node) {0xb000000b,0xa0000014};
  book->defs[0x00027086]->node[11] = (Node) {0xb000000c,0xa0000013};
  book->defs[0x00027086]->node[12] = (Node) {0xb000000d,0xa0000012};
  book->defs[0x00027086]->node[13] = (Node) {0xb000000e,0xa0000011};
  book->defs[0x00027086]->node[14] = (Node) {0xa000000f,0xa0000010};
  book->defs[0x00027086]->node[15] = (Node) {0x4000001e,0x40000010};
  book->defs[0x00027086]->node[16] = (Node) {0x5000000f,0x40000011};
  book->defs[0x00027086]->node[17] = (Node) {0x50000010,0x40000012};
  book->defs[0x00027086]->node[18] = (Node) {0x50000011,0x40000013};
  book->defs[0x00027086]->node[19] = (Node) {0x50000012,0x40000014};
  book->defs[0x00027086]->node[20] = (Node) {0x50000013,0x40000015};
  book->defs[0x00027086]->node[21] = (Node) {0x50000014,0x40000016};
  book->defs[0x00027086]->node[22] = (Node) {0x50000015,0x40000017};
  book->defs[0x00027086]->node[23] = (Node) {0x50000016,0x40000018};
  book->defs[0x00027086]->node[24] = (Node) {0x50000017,0x40000019};
  book->defs[0x00027086]->node[25] = (Node) {0x50000018,0x4000001a};
  book->defs[0x00027086]->node[26] = (Node) {0x50000019,0x4000001b};
  book->defs[0x00027086]->node[27] = (Node) {0x5000001a,0x4000001c};
  book->defs[0x00027086]->node[28] = (Node) {0x5000001b,0x4000001d};
  book->defs[0x00027086]->node[29] = (Node) {0x5000001c,0x5000001e};
  book->defs[0x00027086]->node[30] = (Node) {0x4000000f,0x5000001d};
  // c16
  book->defs[0x00027087]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00027087]->root     = 0xa0000000;
  book->defs[0x00027087]->alen     = 0;
  book->defs[0x00027087]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00027087]->nlen     = 33;
  book->defs[0x00027087]->node     = (Node*) malloc(33 * sizeof(Node));
  book->defs[0x00027087]->node[ 0] = (Node) {0xb0000001,0xa0000020};
  book->defs[0x00027087]->node[ 1] = (Node) {0xb0000002,0xa000001f};
  book->defs[0x00027087]->node[ 2] = (Node) {0xb0000003,0xa000001e};
  book->defs[0x00027087]->node[ 3] = (Node) {0xb0000004,0xa000001d};
  book->defs[0x00027087]->node[ 4] = (Node) {0xb0000005,0xa000001c};
  book->defs[0x00027087]->node[ 5] = (Node) {0xb0000006,0xa000001b};
  book->defs[0x00027087]->node[ 6] = (Node) {0xb0000007,0xa000001a};
  book->defs[0x00027087]->node[ 7] = (Node) {0xb0000008,0xa0000019};
  book->defs[0x00027087]->node[ 8] = (Node) {0xb0000009,0xa0000018};
  book->defs[0x00027087]->node[ 9] = (Node) {0xb000000a,0xa0000017};
  book->defs[0x00027087]->node[10] = (Node) {0xb000000b,0xa0000016};
  book->defs[0x00027087]->node[11] = (Node) {0xb000000c,0xa0000015};
  book->defs[0x00027087]->node[12] = (Node) {0xb000000d,0xa0000014};
  book->defs[0x00027087]->node[13] = (Node) {0xb000000e,0xa0000013};
  book->defs[0x00027087]->node[14] = (Node) {0xb000000f,0xa0000012};
  book->defs[0x00027087]->node[15] = (Node) {0xa0000010,0xa0000011};
  book->defs[0x00027087]->node[16] = (Node) {0x00000000,0x40000011};
  book->defs[0x00027087]->node[17] = (Node) {0x50000010,0x40000012};
  book->defs[0x00027087]->node[18] = (Node) {0x50000011,0x40000013};
  book->defs[0x00027087]->node[19] = (Node) {0x50000012,0x40000014};
  book->defs[0x00027087]->node[20] = (Node) {0x50000013,0x40000015};
  book->defs[0x00027087]->node[21] = (Node) {0x50000014,0x40000016};
  book->defs[0x00027087]->node[22] = (Node) {0x50000015,0x40000017};
  book->defs[0x00027087]->node[23] = (Node) {0x50000016,0x40000018};
  book->defs[0x00027087]->node[24] = (Node) {0x50000017,0x40000019};
  book->defs[0x00027087]->node[25] = (Node) {0x50000018,0x4000001a};
  book->defs[0x00027087]->node[26] = (Node) {0x50000019,0x4000001b};
  book->defs[0x00027087]->node[27] = (Node) {0x5000001a,0x4000001c};
  book->defs[0x00027087]->node[28] = (Node) {0x5000001b,0x4000001d};
  book->defs[0x00027087]->node[29] = (Node) {0x5000001c,0x4000001e};
  book->defs[0x00027087]->node[30] = (Node) {0x5000001d,0x4000001f};
  book->defs[0x00027087]->node[31] = (Node) {0x5000001e,0x50000020};
  book->defs[0x00027087]->node[32] = (Node) {0x00000000,0x5000001f};
  // c17
  book->defs[0x00027088]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00027088]->root     = 0xa0000000;
  book->defs[0x00027088]->alen     = 0;
  book->defs[0x00027088]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00027088]->nlen     = 35;
  book->defs[0x00027088]->node     = (Node*) malloc(35 * sizeof(Node));
  book->defs[0x00027088]->node[ 0] = (Node) {0xb0000001,0xa0000022};
  book->defs[0x00027088]->node[ 1] = (Node) {0xb0000002,0xa0000021};
  book->defs[0x00027088]->node[ 2] = (Node) {0xb0000003,0xa0000020};
  book->defs[0x00027088]->node[ 3] = (Node) {0xb0000004,0xa000001f};
  book->defs[0x00027088]->node[ 4] = (Node) {0xb0000005,0xa000001e};
  book->defs[0x00027088]->node[ 5] = (Node) {0xb0000006,0xa000001d};
  book->defs[0x00027088]->node[ 6] = (Node) {0xb0000007,0xa000001c};
  book->defs[0x00027088]->node[ 7] = (Node) {0xb0000008,0xa000001b};
  book->defs[0x00027088]->node[ 8] = (Node) {0xb0000009,0xa000001a};
  book->defs[0x00027088]->node[ 9] = (Node) {0xb000000a,0xa0000019};
  book->defs[0x00027088]->node[10] = (Node) {0xb000000b,0xa0000018};
  book->defs[0x00027088]->node[11] = (Node) {0xb000000c,0xa0000017};
  book->defs[0x00027088]->node[12] = (Node) {0xb000000d,0xa0000016};
  book->defs[0x00027088]->node[13] = (Node) {0xb000000e,0xa0000015};
  book->defs[0x00027088]->node[14] = (Node) {0xb000000f,0xa0000014};
  book->defs[0x00027088]->node[15] = (Node) {0xb0000010,0xa0000013};
  book->defs[0x00027088]->node[16] = (Node) {0xa0000011,0xa0000012};
  book->defs[0x00027088]->node[17] = (Node) {0x40000022,0x40000012};
  book->defs[0x00027088]->node[18] = (Node) {0x50000011,0x40000013};
  book->defs[0x00027088]->node[19] = (Node) {0x50000012,0x40000014};
  book->defs[0x00027088]->node[20] = (Node) {0x50000013,0x40000015};
  book->defs[0x00027088]->node[21] = (Node) {0x50000014,0x40000016};
  book->defs[0x00027088]->node[22] = (Node) {0x50000015,0x40000017};
  book->defs[0x00027088]->node[23] = (Node) {0x50000016,0x40000018};
  book->defs[0x00027088]->node[24] = (Node) {0x50000017,0x40000019};
  book->defs[0x00027088]->node[25] = (Node) {0x50000018,0x4000001a};
  book->defs[0x00027088]->node[26] = (Node) {0x50000019,0x4000001b};
  book->defs[0x00027088]->node[27] = (Node) {0x5000001a,0x4000001c};
  book->defs[0x00027088]->node[28] = (Node) {0x5000001b,0x4000001d};
  book->defs[0x00027088]->node[29] = (Node) {0x5000001c,0x4000001e};
  book->defs[0x00027088]->node[30] = (Node) {0x5000001d,0x4000001f};
  book->defs[0x00027088]->node[31] = (Node) {0x5000001e,0x40000020};
  book->defs[0x00027088]->node[32] = (Node) {0x5000001f,0x40000021};
  book->defs[0x00027088]->node[33] = (Node) {0x50000020,0x50000022};
  book->defs[0x00027088]->node[34] = (Node) {0x40000011,0x50000021};
  // c18
  book->defs[0x00027089]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00027089]->root     = 0xa0000000;
  book->defs[0x00027089]->alen     = 0;
  book->defs[0x00027089]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00027089]->nlen     = 37;
  book->defs[0x00027089]->node     = (Node*) malloc(37 * sizeof(Node));
  book->defs[0x00027089]->node[ 0] = (Node) {0xb0000001,0xa0000024};
  book->defs[0x00027089]->node[ 1] = (Node) {0xb0000002,0xa0000023};
  book->defs[0x00027089]->node[ 2] = (Node) {0xb0000003,0xa0000022};
  book->defs[0x00027089]->node[ 3] = (Node) {0xb0000004,0xa0000021};
  book->defs[0x00027089]->node[ 4] = (Node) {0xb0000005,0xa0000020};
  book->defs[0x00027089]->node[ 5] = (Node) {0xb0000006,0xa000001f};
  book->defs[0x00027089]->node[ 6] = (Node) {0xb0000007,0xa000001e};
  book->defs[0x00027089]->node[ 7] = (Node) {0xb0000008,0xa000001d};
  book->defs[0x00027089]->node[ 8] = (Node) {0xb0000009,0xa000001c};
  book->defs[0x00027089]->node[ 9] = (Node) {0xb000000a,0xa000001b};
  book->defs[0x00027089]->node[10] = (Node) {0xb000000b,0xa000001a};
  book->defs[0x00027089]->node[11] = (Node) {0xb000000c,0xa0000019};
  book->defs[0x00027089]->node[12] = (Node) {0xb000000d,0xa0000018};
  book->defs[0x00027089]->node[13] = (Node) {0xb000000e,0xa0000017};
  book->defs[0x00027089]->node[14] = (Node) {0xb000000f,0xa0000016};
  book->defs[0x00027089]->node[15] = (Node) {0xb0000010,0xa0000015};
  book->defs[0x00027089]->node[16] = (Node) {0xb0000011,0xa0000014};
  book->defs[0x00027089]->node[17] = (Node) {0xa0000012,0xa0000013};
  book->defs[0x00027089]->node[18] = (Node) {0x40000024,0x40000013};
  book->defs[0x00027089]->node[19] = (Node) {0x50000012,0x40000014};
  book->defs[0x00027089]->node[20] = (Node) {0x50000013,0x40000015};
  book->defs[0x00027089]->node[21] = (Node) {0x50000014,0x40000016};
  book->defs[0x00027089]->node[22] = (Node) {0x50000015,0x40000017};
  book->defs[0x00027089]->node[23] = (Node) {0x50000016,0x40000018};
  book->defs[0x00027089]->node[24] = (Node) {0x50000017,0x40000019};
  book->defs[0x00027089]->node[25] = (Node) {0x50000018,0x4000001a};
  book->defs[0x00027089]->node[26] = (Node) {0x50000019,0x4000001b};
  book->defs[0x00027089]->node[27] = (Node) {0x5000001a,0x4000001c};
  book->defs[0x00027089]->node[28] = (Node) {0x5000001b,0x4000001d};
  book->defs[0x00027089]->node[29] = (Node) {0x5000001c,0x4000001e};
  book->defs[0x00027089]->node[30] = (Node) {0x5000001d,0x4000001f};
  book->defs[0x00027089]->node[31] = (Node) {0x5000001e,0x40000020};
  book->defs[0x00027089]->node[32] = (Node) {0x5000001f,0x40000021};
  book->defs[0x00027089]->node[33] = (Node) {0x50000020,0x40000022};
  book->defs[0x00027089]->node[34] = (Node) {0x50000021,0x40000023};
  book->defs[0x00027089]->node[35] = (Node) {0x50000022,0x50000024};
  book->defs[0x00027089]->node[36] = (Node) {0x40000012,0x50000023};
  // c19
  book->defs[0x0002708a]           = (Term*) malloc(sizeof(Term));
  book->defs[0x0002708a]->root     = 0xa0000000;
  book->defs[0x0002708a]->alen     = 0;
  book->defs[0x0002708a]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x0002708a]->nlen     = 39;
  book->defs[0x0002708a]->node     = (Node*) malloc(39 * sizeof(Node));
  book->defs[0x0002708a]->node[ 0] = (Node) {0xb0000001,0xa0000026};
  book->defs[0x0002708a]->node[ 1] = (Node) {0xb0000002,0xa0000025};
  book->defs[0x0002708a]->node[ 2] = (Node) {0xb0000003,0xa0000024};
  book->defs[0x0002708a]->node[ 3] = (Node) {0xb0000004,0xa0000023};
  book->defs[0x0002708a]->node[ 4] = (Node) {0xb0000005,0xa0000022};
  book->defs[0x0002708a]->node[ 5] = (Node) {0xb0000006,0xa0000021};
  book->defs[0x0002708a]->node[ 6] = (Node) {0xb0000007,0xa0000020};
  book->defs[0x0002708a]->node[ 7] = (Node) {0xb0000008,0xa000001f};
  book->defs[0x0002708a]->node[ 8] = (Node) {0xb0000009,0xa000001e};
  book->defs[0x0002708a]->node[ 9] = (Node) {0xb000000a,0xa000001d};
  book->defs[0x0002708a]->node[10] = (Node) {0xb000000b,0xa000001c};
  book->defs[0x0002708a]->node[11] = (Node) {0xb000000c,0xa000001b};
  book->defs[0x0002708a]->node[12] = (Node) {0xb000000d,0xa000001a};
  book->defs[0x0002708a]->node[13] = (Node) {0xb000000e,0xa0000019};
  book->defs[0x0002708a]->node[14] = (Node) {0xb000000f,0xa0000018};
  book->defs[0x0002708a]->node[15] = (Node) {0xb0000010,0xa0000017};
  book->defs[0x0002708a]->node[16] = (Node) {0xb0000011,0xa0000016};
  book->defs[0x0002708a]->node[17] = (Node) {0xb0000012,0xa0000015};
  book->defs[0x0002708a]->node[18] = (Node) {0xa0000013,0xa0000014};
  book->defs[0x0002708a]->node[19] = (Node) {0x40000026,0x40000014};
  book->defs[0x0002708a]->node[20] = (Node) {0x50000013,0x40000015};
  book->defs[0x0002708a]->node[21] = (Node) {0x50000014,0x40000016};
  book->defs[0x0002708a]->node[22] = (Node) {0x50000015,0x40000017};
  book->defs[0x0002708a]->node[23] = (Node) {0x50000016,0x40000018};
  book->defs[0x0002708a]->node[24] = (Node) {0x50000017,0x40000019};
  book->defs[0x0002708a]->node[25] = (Node) {0x50000018,0x4000001a};
  book->defs[0x0002708a]->node[26] = (Node) {0x50000019,0x4000001b};
  book->defs[0x0002708a]->node[27] = (Node) {0x5000001a,0x4000001c};
  book->defs[0x0002708a]->node[28] = (Node) {0x5000001b,0x4000001d};
  book->defs[0x0002708a]->node[29] = (Node) {0x5000001c,0x4000001e};
  book->defs[0x0002708a]->node[30] = (Node) {0x5000001d,0x4000001f};
  book->defs[0x0002708a]->node[31] = (Node) {0x5000001e,0x40000020};
  book->defs[0x0002708a]->node[32] = (Node) {0x5000001f,0x40000021};
  book->defs[0x0002708a]->node[33] = (Node) {0x50000020,0x40000022};
  book->defs[0x0002708a]->node[34] = (Node) {0x50000021,0x40000023};
  book->defs[0x0002708a]->node[35] = (Node) {0x50000022,0x40000024};
  book->defs[0x0002708a]->node[36] = (Node) {0x50000023,0x40000025};
  book->defs[0x0002708a]->node[37] = (Node) {0x50000024,0x50000026};
  book->defs[0x0002708a]->node[38] = (Node) {0x40000013,0x50000025};
  // c20
  book->defs[0x000270c1]           = (Term*) malloc(sizeof(Term));
  book->defs[0x000270c1]->root     = 0xa0000000;
  book->defs[0x000270c1]->alen     = 0;
  book->defs[0x000270c1]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x000270c1]->nlen     = 41;
  book->defs[0x000270c1]->node     = (Node*) malloc(41 * sizeof(Node));
  book->defs[0x000270c1]->node[ 0] = (Node) {0xb0000001,0xa0000028};
  book->defs[0x000270c1]->node[ 1] = (Node) {0xb0000002,0xa0000027};
  book->defs[0x000270c1]->node[ 2] = (Node) {0xb0000003,0xa0000026};
  book->defs[0x000270c1]->node[ 3] = (Node) {0xb0000004,0xa0000025};
  book->defs[0x000270c1]->node[ 4] = (Node) {0xb0000005,0xa0000024};
  book->defs[0x000270c1]->node[ 5] = (Node) {0xb0000006,0xa0000023};
  book->defs[0x000270c1]->node[ 6] = (Node) {0xb0000007,0xa0000022};
  book->defs[0x000270c1]->node[ 7] = (Node) {0xb0000008,0xa0000021};
  book->defs[0x000270c1]->node[ 8] = (Node) {0xb0000009,0xa0000020};
  book->defs[0x000270c1]->node[ 9] = (Node) {0xb000000a,0xa000001f};
  book->defs[0x000270c1]->node[10] = (Node) {0xb000000b,0xa000001e};
  book->defs[0x000270c1]->node[11] = (Node) {0xb000000c,0xa000001d};
  book->defs[0x000270c1]->node[12] = (Node) {0xb000000d,0xa000001c};
  book->defs[0x000270c1]->node[13] = (Node) {0xb000000e,0xa000001b};
  book->defs[0x000270c1]->node[14] = (Node) {0xb000000f,0xa000001a};
  book->defs[0x000270c1]->node[15] = (Node) {0xb0000010,0xa0000019};
  book->defs[0x000270c1]->node[16] = (Node) {0xb0000011,0xa0000018};
  book->defs[0x000270c1]->node[17] = (Node) {0xb0000012,0xa0000017};
  book->defs[0x000270c1]->node[18] = (Node) {0xb0000013,0xa0000016};
  book->defs[0x000270c1]->node[19] = (Node) {0xa0000014,0xa0000015};
  book->defs[0x000270c1]->node[20] = (Node) {0x40000028,0x40000015};
  book->defs[0x000270c1]->node[21] = (Node) {0x50000014,0x40000016};
  book->defs[0x000270c1]->node[22] = (Node) {0x50000015,0x40000017};
  book->defs[0x000270c1]->node[23] = (Node) {0x50000016,0x40000018};
  book->defs[0x000270c1]->node[24] = (Node) {0x50000017,0x40000019};
  book->defs[0x000270c1]->node[25] = (Node) {0x50000018,0x4000001a};
  book->defs[0x000270c1]->node[26] = (Node) {0x50000019,0x4000001b};
  book->defs[0x000270c1]->node[27] = (Node) {0x5000001a,0x4000001c};
  book->defs[0x000270c1]->node[28] = (Node) {0x5000001b,0x4000001d};
  book->defs[0x000270c1]->node[29] = (Node) {0x5000001c,0x4000001e};
  book->defs[0x000270c1]->node[30] = (Node) {0x5000001d,0x4000001f};
  book->defs[0x000270c1]->node[31] = (Node) {0x5000001e,0x40000020};
  book->defs[0x000270c1]->node[32] = (Node) {0x5000001f,0x40000021};
  book->defs[0x000270c1]->node[33] = (Node) {0x50000020,0x40000022};
  book->defs[0x000270c1]->node[34] = (Node) {0x50000021,0x40000023};
  book->defs[0x000270c1]->node[35] = (Node) {0x50000022,0x40000024};
  book->defs[0x000270c1]->node[36] = (Node) {0x50000023,0x40000025};
  book->defs[0x000270c1]->node[37] = (Node) {0x50000024,0x40000026};
  book->defs[0x000270c1]->node[38] = (Node) {0x50000025,0x40000027};
  book->defs[0x000270c1]->node[39] = (Node) {0x50000026,0x50000028};
  book->defs[0x000270c1]->node[40] = (Node) {0x40000014,0x50000027};
  // c21
  book->defs[0x000270c2]           = (Term*) malloc(sizeof(Term));
  book->defs[0x000270c2]->root     = 0xa0000000;
  book->defs[0x000270c2]->alen     = 0;
  book->defs[0x000270c2]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x000270c2]->nlen     = 43;
  book->defs[0x000270c2]->node     = (Node*) malloc(43 * sizeof(Node));
  book->defs[0x000270c2]->node[ 0] = (Node) {0xb0000001,0xa000002a};
  book->defs[0x000270c2]->node[ 1] = (Node) {0xb0000002,0xa0000029};
  book->defs[0x000270c2]->node[ 2] = (Node) {0xb0000003,0xa0000028};
  book->defs[0x000270c2]->node[ 3] = (Node) {0xb0000004,0xa0000027};
  book->defs[0x000270c2]->node[ 4] = (Node) {0xb0000005,0xa0000026};
  book->defs[0x000270c2]->node[ 5] = (Node) {0xb0000006,0xa0000025};
  book->defs[0x000270c2]->node[ 6] = (Node) {0xb0000007,0xa0000024};
  book->defs[0x000270c2]->node[ 7] = (Node) {0xb0000008,0xa0000023};
  book->defs[0x000270c2]->node[ 8] = (Node) {0xb0000009,0xa0000022};
  book->defs[0x000270c2]->node[ 9] = (Node) {0xb000000a,0xa0000021};
  book->defs[0x000270c2]->node[10] = (Node) {0xb000000b,0xa0000020};
  book->defs[0x000270c2]->node[11] = (Node) {0xb000000c,0xa000001f};
  book->defs[0x000270c2]->node[12] = (Node) {0xb000000d,0xa000001e};
  book->defs[0x000270c2]->node[13] = (Node) {0xb000000e,0xa000001d};
  book->defs[0x000270c2]->node[14] = (Node) {0xb000000f,0xa000001c};
  book->defs[0x000270c2]->node[15] = (Node) {0xb0000010,0xa000001b};
  book->defs[0x000270c2]->node[16] = (Node) {0xb0000011,0xa000001a};
  book->defs[0x000270c2]->node[17] = (Node) {0xb0000012,0xa0000019};
  book->defs[0x000270c2]->node[18] = (Node) {0xb0000013,0xa0000018};
  book->defs[0x000270c2]->node[19] = (Node) {0xb0000014,0xa0000017};
  book->defs[0x000270c2]->node[20] = (Node) {0xa0000015,0xa0000016};
  book->defs[0x000270c2]->node[21] = (Node) {0x4000002a,0x40000016};
  book->defs[0x000270c2]->node[22] = (Node) {0x50000015,0x40000017};
  book->defs[0x000270c2]->node[23] = (Node) {0x50000016,0x40000018};
  book->defs[0x000270c2]->node[24] = (Node) {0x50000017,0x40000019};
  book->defs[0x000270c2]->node[25] = (Node) {0x50000018,0x4000001a};
  book->defs[0x000270c2]->node[26] = (Node) {0x50000019,0x4000001b};
  book->defs[0x000270c2]->node[27] = (Node) {0x5000001a,0x4000001c};
  book->defs[0x000270c2]->node[28] = (Node) {0x5000001b,0x4000001d};
  book->defs[0x000270c2]->node[29] = (Node) {0x5000001c,0x4000001e};
  book->defs[0x000270c2]->node[30] = (Node) {0x5000001d,0x4000001f};
  book->defs[0x000270c2]->node[31] = (Node) {0x5000001e,0x40000020};
  book->defs[0x000270c2]->node[32] = (Node) {0x5000001f,0x40000021};
  book->defs[0x000270c2]->node[33] = (Node) {0x50000020,0x40000022};
  book->defs[0x000270c2]->node[34] = (Node) {0x50000021,0x40000023};
  book->defs[0x000270c2]->node[35] = (Node) {0x50000022,0x40000024};
  book->defs[0x000270c2]->node[36] = (Node) {0x50000023,0x40000025};
  book->defs[0x000270c2]->node[37] = (Node) {0x50000024,0x40000026};
  book->defs[0x000270c2]->node[38] = (Node) {0x50000025,0x40000027};
  book->defs[0x000270c2]->node[39] = (Node) {0x50000026,0x40000028};
  book->defs[0x000270c2]->node[40] = (Node) {0x50000027,0x40000029};
  book->defs[0x000270c2]->node[41] = (Node) {0x50000028,0x5000002a};
  book->defs[0x000270c2]->node[42] = (Node) {0x40000015,0x50000029};
  // c22
  book->defs[0x000270c3]           = (Term*) malloc(sizeof(Term));
  book->defs[0x000270c3]->root     = 0xa0000000;
  book->defs[0x000270c3]->alen     = 0;
  book->defs[0x000270c3]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x000270c3]->nlen     = 45;
  book->defs[0x000270c3]->node     = (Node*) malloc(45 * sizeof(Node));
  book->defs[0x000270c3]->node[ 0] = (Node) {0xb0000001,0xa000002c};
  book->defs[0x000270c3]->node[ 1] = (Node) {0xb0000002,0xa000002b};
  book->defs[0x000270c3]->node[ 2] = (Node) {0xb0000003,0xa000002a};
  book->defs[0x000270c3]->node[ 3] = (Node) {0xb0000004,0xa0000029};
  book->defs[0x000270c3]->node[ 4] = (Node) {0xb0000005,0xa0000028};
  book->defs[0x000270c3]->node[ 5] = (Node) {0xb0000006,0xa0000027};
  book->defs[0x000270c3]->node[ 6] = (Node) {0xb0000007,0xa0000026};
  book->defs[0x000270c3]->node[ 7] = (Node) {0xb0000008,0xa0000025};
  book->defs[0x000270c3]->node[ 8] = (Node) {0xb0000009,0xa0000024};
  book->defs[0x000270c3]->node[ 9] = (Node) {0xb000000a,0xa0000023};
  book->defs[0x000270c3]->node[10] = (Node) {0xb000000b,0xa0000022};
  book->defs[0x000270c3]->node[11] = (Node) {0xb000000c,0xa0000021};
  book->defs[0x000270c3]->node[12] = (Node) {0xb000000d,0xa0000020};
  book->defs[0x000270c3]->node[13] = (Node) {0xb000000e,0xa000001f};
  book->defs[0x000270c3]->node[14] = (Node) {0xb000000f,0xa000001e};
  book->defs[0x000270c3]->node[15] = (Node) {0xb0000010,0xa000001d};
  book->defs[0x000270c3]->node[16] = (Node) {0xb0000011,0xa000001c};
  book->defs[0x000270c3]->node[17] = (Node) {0xb0000012,0xa000001b};
  book->defs[0x000270c3]->node[18] = (Node) {0xb0000013,0xa000001a};
  book->defs[0x000270c3]->node[19] = (Node) {0xb0000014,0xa0000019};
  book->defs[0x000270c3]->node[20] = (Node) {0xb0000015,0xa0000018};
  book->defs[0x000270c3]->node[21] = (Node) {0xa0000016,0xa0000017};
  book->defs[0x000270c3]->node[22] = (Node) {0x4000002c,0x40000017};
  book->defs[0x000270c3]->node[23] = (Node) {0x50000016,0x40000018};
  book->defs[0x000270c3]->node[24] = (Node) {0x50000017,0x40000019};
  book->defs[0x000270c3]->node[25] = (Node) {0x50000018,0x4000001a};
  book->defs[0x000270c3]->node[26] = (Node) {0x50000019,0x4000001b};
  book->defs[0x000270c3]->node[27] = (Node) {0x5000001a,0x4000001c};
  book->defs[0x000270c3]->node[28] = (Node) {0x5000001b,0x4000001d};
  book->defs[0x000270c3]->node[29] = (Node) {0x5000001c,0x4000001e};
  book->defs[0x000270c3]->node[30] = (Node) {0x5000001d,0x4000001f};
  book->defs[0x000270c3]->node[31] = (Node) {0x5000001e,0x40000020};
  book->defs[0x000270c3]->node[32] = (Node) {0x5000001f,0x40000021};
  book->defs[0x000270c3]->node[33] = (Node) {0x50000020,0x40000022};
  book->defs[0x000270c3]->node[34] = (Node) {0x50000021,0x40000023};
  book->defs[0x000270c3]->node[35] = (Node) {0x50000022,0x40000024};
  book->defs[0x000270c3]->node[36] = (Node) {0x50000023,0x40000025};
  book->defs[0x000270c3]->node[37] = (Node) {0x50000024,0x40000026};
  book->defs[0x000270c3]->node[38] = (Node) {0x50000025,0x40000027};
  book->defs[0x000270c3]->node[39] = (Node) {0x50000026,0x40000028};
  book->defs[0x000270c3]->node[40] = (Node) {0x50000027,0x40000029};
  book->defs[0x000270c3]->node[41] = (Node) {0x50000028,0x4000002a};
  book->defs[0x000270c3]->node[42] = (Node) {0x50000029,0x4000002b};
  book->defs[0x000270c3]->node[43] = (Node) {0x5000002a,0x5000002c};
  book->defs[0x000270c3]->node[44] = (Node) {0x40000016,0x5000002b};
  // c23
  book->defs[0x000270c4]           = (Term*) malloc(sizeof(Term));
  book->defs[0x000270c4]->root     = 0xa0000000;
  book->defs[0x000270c4]->alen     = 0;
  book->defs[0x000270c4]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x000270c4]->nlen     = 47;
  book->defs[0x000270c4]->node     = (Node*) malloc(47 * sizeof(Node));
  book->defs[0x000270c4]->node[ 0] = (Node) {0xb0000001,0xa000002e};
  book->defs[0x000270c4]->node[ 1] = (Node) {0xb0000002,0xa000002d};
  book->defs[0x000270c4]->node[ 2] = (Node) {0xb0000003,0xa000002c};
  book->defs[0x000270c4]->node[ 3] = (Node) {0xb0000004,0xa000002b};
  book->defs[0x000270c4]->node[ 4] = (Node) {0xb0000005,0xa000002a};
  book->defs[0x000270c4]->node[ 5] = (Node) {0xb0000006,0xa0000029};
  book->defs[0x000270c4]->node[ 6] = (Node) {0xb0000007,0xa0000028};
  book->defs[0x000270c4]->node[ 7] = (Node) {0xb0000008,0xa0000027};
  book->defs[0x000270c4]->node[ 8] = (Node) {0xb0000009,0xa0000026};
  book->defs[0x000270c4]->node[ 9] = (Node) {0xb000000a,0xa0000025};
  book->defs[0x000270c4]->node[10] = (Node) {0xb000000b,0xa0000024};
  book->defs[0x000270c4]->node[11] = (Node) {0xb000000c,0xa0000023};
  book->defs[0x000270c4]->node[12] = (Node) {0xb000000d,0xa0000022};
  book->defs[0x000270c4]->node[13] = (Node) {0xb000000e,0xa0000021};
  book->defs[0x000270c4]->node[14] = (Node) {0xb000000f,0xa0000020};
  book->defs[0x000270c4]->node[15] = (Node) {0xb0000010,0xa000001f};
  book->defs[0x000270c4]->node[16] = (Node) {0xb0000011,0xa000001e};
  book->defs[0x000270c4]->node[17] = (Node) {0xb0000012,0xa000001d};
  book->defs[0x000270c4]->node[18] = (Node) {0xb0000013,0xa000001c};
  book->defs[0x000270c4]->node[19] = (Node) {0xb0000014,0xa000001b};
  book->defs[0x000270c4]->node[20] = (Node) {0xb0000015,0xa000001a};
  book->defs[0x000270c4]->node[21] = (Node) {0xb0000016,0xa0000019};
  book->defs[0x000270c4]->node[22] = (Node) {0xa0000017,0xa0000018};
  book->defs[0x000270c4]->node[23] = (Node) {0x4000002e,0x40000018};
  book->defs[0x000270c4]->node[24] = (Node) {0x50000017,0x40000019};
  book->defs[0x000270c4]->node[25] = (Node) {0x50000018,0x4000001a};
  book->defs[0x000270c4]->node[26] = (Node) {0x50000019,0x4000001b};
  book->defs[0x000270c4]->node[27] = (Node) {0x5000001a,0x4000001c};
  book->defs[0x000270c4]->node[28] = (Node) {0x5000001b,0x4000001d};
  book->defs[0x000270c4]->node[29] = (Node) {0x5000001c,0x4000001e};
  book->defs[0x000270c4]->node[30] = (Node) {0x5000001d,0x4000001f};
  book->defs[0x000270c4]->node[31] = (Node) {0x5000001e,0x40000020};
  book->defs[0x000270c4]->node[32] = (Node) {0x5000001f,0x40000021};
  book->defs[0x000270c4]->node[33] = (Node) {0x50000020,0x40000022};
  book->defs[0x000270c4]->node[34] = (Node) {0x50000021,0x40000023};
  book->defs[0x000270c4]->node[35] = (Node) {0x50000022,0x40000024};
  book->defs[0x000270c4]->node[36] = (Node) {0x50000023,0x40000025};
  book->defs[0x000270c4]->node[37] = (Node) {0x50000024,0x40000026};
  book->defs[0x000270c4]->node[38] = (Node) {0x50000025,0x40000027};
  book->defs[0x000270c4]->node[39] = (Node) {0x50000026,0x40000028};
  book->defs[0x000270c4]->node[40] = (Node) {0x50000027,0x40000029};
  book->defs[0x000270c4]->node[41] = (Node) {0x50000028,0x4000002a};
  book->defs[0x000270c4]->node[42] = (Node) {0x50000029,0x4000002b};
  book->defs[0x000270c4]->node[43] = (Node) {0x5000002a,0x4000002c};
  book->defs[0x000270c4]->node[44] = (Node) {0x5000002b,0x4000002d};
  book->defs[0x000270c4]->node[45] = (Node) {0x5000002c,0x5000002e};
  book->defs[0x000270c4]->node[46] = (Node) {0x40000017,0x5000002d};
  // c24
  book->defs[0x000270c5]           = (Term*) malloc(sizeof(Term));
  book->defs[0x000270c5]->root     = 0xa0000000;
  book->defs[0x000270c5]->alen     = 0;
  book->defs[0x000270c5]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x000270c5]->nlen     = 49;
  book->defs[0x000270c5]->node     = (Node*) malloc(49 * sizeof(Node));
  book->defs[0x000270c5]->node[ 0] = (Node) {0xb0000001,0xa0000030};
  book->defs[0x000270c5]->node[ 1] = (Node) {0xb0000002,0xa000002f};
  book->defs[0x000270c5]->node[ 2] = (Node) {0xb0000003,0xa000002e};
  book->defs[0x000270c5]->node[ 3] = (Node) {0xb0000004,0xa000002d};
  book->defs[0x000270c5]->node[ 4] = (Node) {0xb0000005,0xa000002c};
  book->defs[0x000270c5]->node[ 5] = (Node) {0xb0000006,0xa000002b};
  book->defs[0x000270c5]->node[ 6] = (Node) {0xb0000007,0xa000002a};
  book->defs[0x000270c5]->node[ 7] = (Node) {0xb0000008,0xa0000029};
  book->defs[0x000270c5]->node[ 8] = (Node) {0xb0000009,0xa0000028};
  book->defs[0x000270c5]->node[ 9] = (Node) {0xb000000a,0xa0000027};
  book->defs[0x000270c5]->node[10] = (Node) {0xb000000b,0xa0000026};
  book->defs[0x000270c5]->node[11] = (Node) {0xb000000c,0xa0000025};
  book->defs[0x000270c5]->node[12] = (Node) {0xb000000d,0xa0000024};
  book->defs[0x000270c5]->node[13] = (Node) {0xb000000e,0xa0000023};
  book->defs[0x000270c5]->node[14] = (Node) {0xb000000f,0xa0000022};
  book->defs[0x000270c5]->node[15] = (Node) {0xb0000010,0xa0000021};
  book->defs[0x000270c5]->node[16] = (Node) {0xb0000011,0xa0000020};
  book->defs[0x000270c5]->node[17] = (Node) {0xb0000012,0xa000001f};
  book->defs[0x000270c5]->node[18] = (Node) {0xb0000013,0xa000001e};
  book->defs[0x000270c5]->node[19] = (Node) {0xb0000014,0xa000001d};
  book->defs[0x000270c5]->node[20] = (Node) {0xb0000015,0xa000001c};
  book->defs[0x000270c5]->node[21] = (Node) {0xb0000016,0xa000001b};
  book->defs[0x000270c5]->node[22] = (Node) {0xb0000017,0xa000001a};
  book->defs[0x000270c5]->node[23] = (Node) {0xa0000018,0xa0000019};
  book->defs[0x000270c5]->node[24] = (Node) {0x40000030,0x40000019};
  book->defs[0x000270c5]->node[25] = (Node) {0x50000018,0x4000001a};
  book->defs[0x000270c5]->node[26] = (Node) {0x50000019,0x4000001b};
  book->defs[0x000270c5]->node[27] = (Node) {0x5000001a,0x4000001c};
  book->defs[0x000270c5]->node[28] = (Node) {0x5000001b,0x4000001d};
  book->defs[0x000270c5]->node[29] = (Node) {0x5000001c,0x4000001e};
  book->defs[0x000270c5]->node[30] = (Node) {0x5000001d,0x4000001f};
  book->defs[0x000270c5]->node[31] = (Node) {0x5000001e,0x40000020};
  book->defs[0x000270c5]->node[32] = (Node) {0x5000001f,0x40000021};
  book->defs[0x000270c5]->node[33] = (Node) {0x50000020,0x40000022};
  book->defs[0x000270c5]->node[34] = (Node) {0x50000021,0x40000023};
  book->defs[0x000270c5]->node[35] = (Node) {0x50000022,0x40000024};
  book->defs[0x000270c5]->node[36] = (Node) {0x50000023,0x40000025};
  book->defs[0x000270c5]->node[37] = (Node) {0x50000024,0x40000026};
  book->defs[0x000270c5]->node[38] = (Node) {0x50000025,0x40000027};
  book->defs[0x000270c5]->node[39] = (Node) {0x50000026,0x40000028};
  book->defs[0x000270c5]->node[40] = (Node) {0x50000027,0x40000029};
  book->defs[0x000270c5]->node[41] = (Node) {0x50000028,0x4000002a};
  book->defs[0x000270c5]->node[42] = (Node) {0x50000029,0x4000002b};
  book->defs[0x000270c5]->node[43] = (Node) {0x5000002a,0x4000002c};
  book->defs[0x000270c5]->node[44] = (Node) {0x5000002b,0x4000002d};
  book->defs[0x000270c5]->node[45] = (Node) {0x5000002c,0x4000002e};
  book->defs[0x000270c5]->node[46] = (Node) {0x5000002d,0x4000002f};
  book->defs[0x000270c5]->node[47] = (Node) {0x5000002e,0x50000030};
  book->defs[0x000270c5]->node[48] = (Node) {0x40000018,0x5000002f};
  // c25
  book->defs[0x000270c6]           = (Term*) malloc(sizeof(Term));
  book->defs[0x000270c6]->root     = 0xa0000000;
  book->defs[0x000270c6]->alen     = 0;
  book->defs[0x000270c6]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x000270c6]->nlen     = 51;
  book->defs[0x000270c6]->node     = (Node*) malloc(51 * sizeof(Node));
  book->defs[0x000270c6]->node[ 0] = (Node) {0xb0000001,0xa0000032};
  book->defs[0x000270c6]->node[ 1] = (Node) {0xb0000002,0xa0000031};
  book->defs[0x000270c6]->node[ 2] = (Node) {0xb0000003,0xa0000030};
  book->defs[0x000270c6]->node[ 3] = (Node) {0xb0000004,0xa000002f};
  book->defs[0x000270c6]->node[ 4] = (Node) {0xb0000005,0xa000002e};
  book->defs[0x000270c6]->node[ 5] = (Node) {0xb0000006,0xa000002d};
  book->defs[0x000270c6]->node[ 6] = (Node) {0xb0000007,0xa000002c};
  book->defs[0x000270c6]->node[ 7] = (Node) {0xb0000008,0xa000002b};
  book->defs[0x000270c6]->node[ 8] = (Node) {0xb0000009,0xa000002a};
  book->defs[0x000270c6]->node[ 9] = (Node) {0xb000000a,0xa0000029};
  book->defs[0x000270c6]->node[10] = (Node) {0xb000000b,0xa0000028};
  book->defs[0x000270c6]->node[11] = (Node) {0xb000000c,0xa0000027};
  book->defs[0x000270c6]->node[12] = (Node) {0xb000000d,0xa0000026};
  book->defs[0x000270c6]->node[13] = (Node) {0xb000000e,0xa0000025};
  book->defs[0x000270c6]->node[14] = (Node) {0xb000000f,0xa0000024};
  book->defs[0x000270c6]->node[15] = (Node) {0xb0000010,0xa0000023};
  book->defs[0x000270c6]->node[16] = (Node) {0xb0000011,0xa0000022};
  book->defs[0x000270c6]->node[17] = (Node) {0xb0000012,0xa0000021};
  book->defs[0x000270c6]->node[18] = (Node) {0xb0000013,0xa0000020};
  book->defs[0x000270c6]->node[19] = (Node) {0xb0000014,0xa000001f};
  book->defs[0x000270c6]->node[20] = (Node) {0xb0000015,0xa000001e};
  book->defs[0x000270c6]->node[21] = (Node) {0xb0000016,0xa000001d};
  book->defs[0x000270c6]->node[22] = (Node) {0xb0000017,0xa000001c};
  book->defs[0x000270c6]->node[23] = (Node) {0xb0000018,0xa000001b};
  book->defs[0x000270c6]->node[24] = (Node) {0xa0000019,0xa000001a};
  book->defs[0x000270c6]->node[25] = (Node) {0x00000000,0x40000032};
  book->defs[0x000270c6]->node[26] = (Node) {0x50000019,0x4000001b};
  book->defs[0x000270c6]->node[27] = (Node) {0x5000001a,0x4000001c};
  book->defs[0x000270c6]->node[28] = (Node) {0x5000001b,0x4000001d};
  book->defs[0x000270c6]->node[29] = (Node) {0x5000001c,0x4000001e};
  book->defs[0x000270c6]->node[30] = (Node) {0x5000001d,0x4000001f};
  book->defs[0x000270c6]->node[31] = (Node) {0x5000001e,0x40000020};
  book->defs[0x000270c6]->node[32] = (Node) {0x5000001f,0x40000021};
  book->defs[0x000270c6]->node[33] = (Node) {0x50000020,0x40000022};
  book->defs[0x000270c6]->node[34] = (Node) {0x50000021,0x40000023};
  book->defs[0x000270c6]->node[35] = (Node) {0x50000022,0x40000024};
  book->defs[0x000270c6]->node[36] = (Node) {0x50000023,0x40000025};
  book->defs[0x000270c6]->node[37] = (Node) {0x50000024,0x40000026};
  book->defs[0x000270c6]->node[38] = (Node) {0x50000025,0x40000027};
  book->defs[0x000270c6]->node[39] = (Node) {0x50000026,0x40000028};
  book->defs[0x000270c6]->node[40] = (Node) {0x50000027,0x40000029};
  book->defs[0x000270c6]->node[41] = (Node) {0x50000028,0x4000002a};
  book->defs[0x000270c6]->node[42] = (Node) {0x50000029,0x4000002b};
  book->defs[0x000270c6]->node[43] = (Node) {0x5000002a,0x4000002c};
  book->defs[0x000270c6]->node[44] = (Node) {0x5000002b,0x4000002d};
  book->defs[0x000270c6]->node[45] = (Node) {0x5000002c,0x4000002e};
  book->defs[0x000270c6]->node[46] = (Node) {0x5000002d,0x4000002f};
  book->defs[0x000270c6]->node[47] = (Node) {0x5000002e,0x40000030};
  book->defs[0x000270c6]->node[48] = (Node) {0x5000002f,0x40000031};
  book->defs[0x000270c6]->node[49] = (Node) {0x50000030,0x50000032};
  book->defs[0x000270c6]->node[50] = (Node) {0x50000019,0x50000031};
  // c26
  book->defs[0x000270c7]           = (Term*) malloc(sizeof(Term));
  book->defs[0x000270c7]->root     = 0xa0000000;
  book->defs[0x000270c7]->alen     = 0;
  book->defs[0x000270c7]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x000270c7]->nlen     = 53;
  book->defs[0x000270c7]->node     = (Node*) malloc(53 * sizeof(Node));
  book->defs[0x000270c7]->node[ 0] = (Node) {0xb0000001,0xa0000034};
  book->defs[0x000270c7]->node[ 1] = (Node) {0xb0000002,0xa0000033};
  book->defs[0x000270c7]->node[ 2] = (Node) {0xb0000003,0xa0000032};
  book->defs[0x000270c7]->node[ 3] = (Node) {0xb0000004,0xa0000031};
  book->defs[0x000270c7]->node[ 4] = (Node) {0xb0000005,0xa0000030};
  book->defs[0x000270c7]->node[ 5] = (Node) {0xb0000006,0xa000002f};
  book->defs[0x000270c7]->node[ 6] = (Node) {0xb0000007,0xa000002e};
  book->defs[0x000270c7]->node[ 7] = (Node) {0xb0000008,0xa000002d};
  book->defs[0x000270c7]->node[ 8] = (Node) {0xb0000009,0xa000002c};
  book->defs[0x000270c7]->node[ 9] = (Node) {0xb000000a,0xa000002b};
  book->defs[0x000270c7]->node[10] = (Node) {0xb000000b,0xa000002a};
  book->defs[0x000270c7]->node[11] = (Node) {0xb000000c,0xa0000029};
  book->defs[0x000270c7]->node[12] = (Node) {0xb000000d,0xa0000028};
  book->defs[0x000270c7]->node[13] = (Node) {0xb000000e,0xa0000027};
  book->defs[0x000270c7]->node[14] = (Node) {0xb000000f,0xa0000026};
  book->defs[0x000270c7]->node[15] = (Node) {0xb0000010,0xa0000025};
  book->defs[0x000270c7]->node[16] = (Node) {0xb0000011,0xa0000024};
  book->defs[0x000270c7]->node[17] = (Node) {0xb0000012,0xa0000023};
  book->defs[0x000270c7]->node[18] = (Node) {0xb0000013,0xa0000022};
  book->defs[0x000270c7]->node[19] = (Node) {0xb0000014,0xa0000021};
  book->defs[0x000270c7]->node[20] = (Node) {0xb0000015,0xa0000020};
  book->defs[0x000270c7]->node[21] = (Node) {0xb0000016,0xa000001f};
  book->defs[0x000270c7]->node[22] = (Node) {0xb0000017,0xa000001e};
  book->defs[0x000270c7]->node[23] = (Node) {0xb0000018,0xa000001d};
  book->defs[0x000270c7]->node[24] = (Node) {0xb0000019,0xa000001c};
  book->defs[0x000270c7]->node[25] = (Node) {0xa000001a,0xa000001b};
  book->defs[0x000270c7]->node[26] = (Node) {0x00000000,0x4000001b};
  book->defs[0x000270c7]->node[27] = (Node) {0x5000001a,0x40000034};
  book->defs[0x000270c7]->node[28] = (Node) {0x5000001b,0x4000001d};
  book->defs[0x000270c7]->node[29] = (Node) {0x5000001c,0x4000001e};
  book->defs[0x000270c7]->node[30] = (Node) {0x5000001d,0x4000001f};
  book->defs[0x000270c7]->node[31] = (Node) {0x5000001e,0x40000020};
  book->defs[0x000270c7]->node[32] = (Node) {0x5000001f,0x40000021};
  book->defs[0x000270c7]->node[33] = (Node) {0x50000020,0x40000022};
  book->defs[0x000270c7]->node[34] = (Node) {0x50000021,0x40000023};
  book->defs[0x000270c7]->node[35] = (Node) {0x50000022,0x40000024};
  book->defs[0x000270c7]->node[36] = (Node) {0x50000023,0x40000025};
  book->defs[0x000270c7]->node[37] = (Node) {0x50000024,0x40000026};
  book->defs[0x000270c7]->node[38] = (Node) {0x50000025,0x40000027};
  book->defs[0x000270c7]->node[39] = (Node) {0x50000026,0x40000028};
  book->defs[0x000270c7]->node[40] = (Node) {0x50000027,0x40000029};
  book->defs[0x000270c7]->node[41] = (Node) {0x50000028,0x4000002a};
  book->defs[0x000270c7]->node[42] = (Node) {0x50000029,0x4000002b};
  book->defs[0x000270c7]->node[43] = (Node) {0x5000002a,0x4000002c};
  book->defs[0x000270c7]->node[44] = (Node) {0x5000002b,0x4000002d};
  book->defs[0x000270c7]->node[45] = (Node) {0x5000002c,0x4000002e};
  book->defs[0x000270c7]->node[46] = (Node) {0x5000002d,0x4000002f};
  book->defs[0x000270c7]->node[47] = (Node) {0x5000002e,0x40000030};
  book->defs[0x000270c7]->node[48] = (Node) {0x5000002f,0x40000031};
  book->defs[0x000270c7]->node[49] = (Node) {0x50000030,0x40000032};
  book->defs[0x000270c7]->node[50] = (Node) {0x50000031,0x40000033};
  book->defs[0x000270c7]->node[51] = (Node) {0x50000032,0x50000034};
  book->defs[0x000270c7]->node[52] = (Node) {0x5000001b,0x50000033};
  // c_s
  book->defs[0x00027ff7]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00027ff7]->root     = 0xa0000000;
  book->defs[0x00027ff7]->alen     = 0;
  book->defs[0x00027ff7]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00027ff7]->nlen     = 7;
  book->defs[0x00027ff7]->node     = (Node*) malloc(7 * sizeof(Node));
  book->defs[0x00027ff7]->node[ 0] = (Node) {0xa0000001,0xa0000003};
  book->defs[0x00027ff7]->node[ 1] = (Node) {0x50000004,0xa0000002};
  book->defs[0x00027ff7]->node[ 2] = (Node) {0x40000006,0x40000005};
  book->defs[0x00027ff7]->node[ 3] = (Node) {0xb0000004,0xa0000006};
  book->defs[0x00027ff7]->node[ 4] = (Node) {0xa0000005,0x40000001};
  book->defs[0x00027ff7]->node[ 5] = (Node) {0x50000002,0x50000006};
  book->defs[0x00027ff7]->node[ 6] = (Node) {0x40000002,0x50000005};
  // c_z
  book->defs[0x00027ffe]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00027ffe]->root     = 0xa0000000;
  book->defs[0x00027ffe]->alen     = 0;
  book->defs[0x00027ffe]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00027ffe]->nlen     = 2;
  book->defs[0x00027ffe]->node     = (Node*) malloc(2 * sizeof(Node));
  book->defs[0x00027ffe]->node[ 0] = (Node) {0x20000000,0xa0000001};
  book->defs[0x00027ffe]->node[ 1] = (Node) {0x50000001,0x40000001};
  // dec
  book->defs[0x00028a67]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00028a67]->root     = 0xa0000000;
  book->defs[0x00028a67]->alen     = 0;
  book->defs[0x00028a67]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00028a67]->nlen     = 4;
  book->defs[0x00028a67]->node     = (Node*) malloc(4 * sizeof(Node));
  book->defs[0x00028a67]->node[ 0] = (Node) {0xa0000001,0x50000003};
  book->defs[0x00028a67]->node[ 1] = (Node) {0x10a299d9,0xa0000002};
  book->defs[0x00028a67]->node[ 2] = (Node) {0x10a299d3,0xa0000003};
  book->defs[0x00028a67]->node[ 3] = (Node) {0x1000000f,0x50000000};
  // ex0
  book->defs[0x00029f01]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00029f01]->root     = 0x50000000;
  book->defs[0x00029f01]->alen     = 1;
  book->defs[0x00029f01]->acts     = (Wire*) malloc(1 * sizeof(Wire));
  book->defs[0x00029f01]->acts[ 0] = mkwire(0x100009c3,0xa0000000);
  book->defs[0x00029f01]->nlen     = 1;
  book->defs[0x00029f01]->node     = (Node*) malloc(1 * sizeof(Node));
  book->defs[0x00029f01]->node[ 0] = (Node) {0x10000bc3,0x30000000};
  // ex1
  book->defs[0x00029f02]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00029f02]->root     = 0x50000001;
  book->defs[0x00029f02]->alen     = 1;
  book->defs[0x00029f02]->acts     = (Wire*) malloc(1 * sizeof(Wire));
  book->defs[0x00029f02]->acts[ 0] = mkwire(0x100270c5,0xa0000000);
  book->defs[0x00029f02]->nlen     = 2;
  book->defs[0x00029f02]->node     = (Node*) malloc(2 * sizeof(Node));
  book->defs[0x00029f02]->node[ 0] = (Node) {0x1002bff7,0xa0000001};
  book->defs[0x00029f02]->node[ 1] = (Node) {0x1002bffe,0x30000000};
  // ex2
  book->defs[0x00029f03]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00029f03]->root     = 0x50000002;
  book->defs[0x00029f03]->alen     = 2;
  book->defs[0x00029f03]->acts     = (Wire*) malloc(2 * sizeof(Wire));
  book->defs[0x00029f03]->acts[ 0] = mkwire(0x100270c1,0xa0000000);
  book->defs[0x00029f03]->acts[ 1] = mkwire(0x10036e72,0xa0000002);
  book->defs[0x00029f03]->nlen     = 3;
  book->defs[0x00029f03]->node     = (Node*) malloc(3 * sizeof(Node));
  book->defs[0x00029f03]->node[ 0] = (Node) {0x10000013,0xa0000001};
  book->defs[0x00029f03]->node[ 1] = (Node) {0x1000000f,0x40000002};
  book->defs[0x00029f03]->node[ 2] = (Node) {0x50000001,0x30000000};
  // ex3
  book->defs[0x00029f04]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00029f04]->root     = 0x50000002;
  book->defs[0x00029f04]->alen     = 2;
  book->defs[0x00029f04]->acts     = (Wire*) malloc(2 * sizeof(Wire));
  book->defs[0x00029f04]->acts[ 0] = mkwire(0x10027088,0xa0000000);
  book->defs[0x00029f04]->acts[ 1] = mkwire(0x10026db2,0xa0000002);
  book->defs[0x00029f04]->nlen     = 3;
  book->defs[0x00029f04]->node     = (Node*) malloc(3 * sizeof(Node));
  book->defs[0x00029f04]->node[ 0] = (Node) {0x1000001d,0xa0000001};
  book->defs[0x00029f04]->node[ 1] = (Node) {0x10000024,0x40000002};
  book->defs[0x00029f04]->node[ 2] = (Node) {0x50000001,0x30000000};
  // ex4
  book->defs[0x00029f05]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00029f05]->root     = 0x50000000;
  book->defs[0x00029f05]->alen     = 2;
  book->defs[0x00029f05]->acts     = (Wire*) malloc(2 * sizeof(Wire));
  book->defs[0x00029f05]->acts[ 0] = mkwire(0xa0000000,0x1000096a);
  book->defs[0x00029f05]->acts[ 1] = mkwire(0xa0000001,0x100009c2);
  book->defs[0x00029f05]->nlen     = 3;
  book->defs[0x00029f05]->node     = (Node*) malloc(3 * sizeof(Node));
  book->defs[0x00029f05]->node[ 0] = (Node) {0x50000002,0x30000000};
  book->defs[0x00029f05]->node[ 1] = (Node) {0x1000001d,0xa0000002};
  book->defs[0x00029f05]->node[ 2] = (Node) {0x10000024,0x40000000};
  // g_s
  book->defs[0x0002bff7]           = (Term*) malloc(sizeof(Term));
  book->defs[0x0002bff7]->root     = 0xa0000000;
  book->defs[0x0002bff7]->alen     = 0;
  book->defs[0x0002bff7]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x0002bff7]->nlen     = 5;
  book->defs[0x0002bff7]->node     = (Node*) malloc(5 * sizeof(Node));
  book->defs[0x0002bff7]->node[ 0] = (Node) {0xc0000001,0xa0000002};
  book->defs[0x0002bff7]->node[ 1] = (Node) {0x40000003,0x40000004};
  book->defs[0x0002bff7]->node[ 2] = (Node) {0xa0000003,0x50000004};
  book->defs[0x0002bff7]->node[ 3] = (Node) {0x40000001,0xa0000004};
  book->defs[0x0002bff7]->node[ 4] = (Node) {0x50000001,0x50000002};
  // g_z
  book->defs[0x0002bffe]           = (Term*) malloc(sizeof(Term));
  book->defs[0x0002bffe]->root     = 0xa0000000;
  book->defs[0x0002bffe]->alen     = 0;
  book->defs[0x0002bffe]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x0002bffe]->nlen     = 1;
  book->defs[0x0002bffe]->node     = (Node*) malloc(1 * sizeof(Node));
  book->defs[0x0002bffe]->node[ 0] = (Node) {0x50000000,0x40000000};
  // k10
  book->defs[0x0002f081]           = (Term*) malloc(sizeof(Term));
  book->defs[0x0002f081]->root     = 0xa0000000;
  book->defs[0x0002f081]->alen     = 0;
  book->defs[0x0002f081]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x0002f081]->nlen     = 21;
  book->defs[0x0002f081]->node     = (Node*) malloc(21 * sizeof(Node));
  book->defs[0x0002f081]->node[ 0] = (Node) {0xc0000001,0xa0000014};
  book->defs[0x0002f081]->node[ 1] = (Node) {0xc0000002,0xa0000013};
  book->defs[0x0002f081]->node[ 2] = (Node) {0xc0000003,0xa0000012};
  book->defs[0x0002f081]->node[ 3] = (Node) {0xc0000004,0xa0000011};
  book->defs[0x0002f081]->node[ 4] = (Node) {0xc0000005,0xa0000010};
  book->defs[0x0002f081]->node[ 5] = (Node) {0xc0000006,0xa000000f};
  book->defs[0x0002f081]->node[ 6] = (Node) {0xc0000007,0xa000000e};
  book->defs[0x0002f081]->node[ 7] = (Node) {0xc0000008,0xa000000d};
  book->defs[0x0002f081]->node[ 8] = (Node) {0xc0000009,0xa000000c};
  book->defs[0x0002f081]->node[ 9] = (Node) {0xa000000a,0xa000000b};
  book->defs[0x0002f081]->node[10] = (Node) {0x40000014,0x4000000b};
  book->defs[0x0002f081]->node[11] = (Node) {0x5000000a,0x4000000c};
  book->defs[0x0002f081]->node[12] = (Node) {0x5000000b,0x4000000d};
  book->defs[0x0002f081]->node[13] = (Node) {0x5000000c,0x4000000e};
  book->defs[0x0002f081]->node[14] = (Node) {0x5000000d,0x4000000f};
  book->defs[0x0002f081]->node[15] = (Node) {0x5000000e,0x40000010};
  book->defs[0x0002f081]->node[16] = (Node) {0x5000000f,0x40000011};
  book->defs[0x0002f081]->node[17] = (Node) {0x50000010,0x40000012};
  book->defs[0x0002f081]->node[18] = (Node) {0x50000011,0x40000013};
  book->defs[0x0002f081]->node[19] = (Node) {0x50000012,0x50000014};
  book->defs[0x0002f081]->node[20] = (Node) {0x4000000a,0x50000013};
  // k11
  book->defs[0x0002f082]           = (Term*) malloc(sizeof(Term));
  book->defs[0x0002f082]->root     = 0xa0000000;
  book->defs[0x0002f082]->alen     = 0;
  book->defs[0x0002f082]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x0002f082]->nlen     = 23;
  book->defs[0x0002f082]->node     = (Node*) malloc(23 * sizeof(Node));
  book->defs[0x0002f082]->node[ 0] = (Node) {0xc0000001,0xa0000016};
  book->defs[0x0002f082]->node[ 1] = (Node) {0xc0000002,0xa0000015};
  book->defs[0x0002f082]->node[ 2] = (Node) {0xc0000003,0xa0000014};
  book->defs[0x0002f082]->node[ 3] = (Node) {0xc0000004,0xa0000013};
  book->defs[0x0002f082]->node[ 4] = (Node) {0xc0000005,0xa0000012};
  book->defs[0x0002f082]->node[ 5] = (Node) {0xc0000006,0xa0000011};
  book->defs[0x0002f082]->node[ 6] = (Node) {0xc0000007,0xa0000010};
  book->defs[0x0002f082]->node[ 7] = (Node) {0xc0000008,0xa000000f};
  book->defs[0x0002f082]->node[ 8] = (Node) {0xc0000009,0xa000000e};
  book->defs[0x0002f082]->node[ 9] = (Node) {0xc000000a,0xa000000d};
  book->defs[0x0002f082]->node[10] = (Node) {0xa000000b,0xa000000c};
  book->defs[0x0002f082]->node[11] = (Node) {0x40000016,0x4000000c};
  book->defs[0x0002f082]->node[12] = (Node) {0x5000000b,0x4000000d};
  book->defs[0x0002f082]->node[13] = (Node) {0x5000000c,0x4000000e};
  book->defs[0x0002f082]->node[14] = (Node) {0x5000000d,0x4000000f};
  book->defs[0x0002f082]->node[15] = (Node) {0x5000000e,0x40000010};
  book->defs[0x0002f082]->node[16] = (Node) {0x5000000f,0x40000011};
  book->defs[0x0002f082]->node[17] = (Node) {0x50000010,0x40000012};
  book->defs[0x0002f082]->node[18] = (Node) {0x50000011,0x40000013};
  book->defs[0x0002f082]->node[19] = (Node) {0x50000012,0x40000014};
  book->defs[0x0002f082]->node[20] = (Node) {0x50000013,0x40000015};
  book->defs[0x0002f082]->node[21] = (Node) {0x50000014,0x50000016};
  book->defs[0x0002f082]->node[22] = (Node) {0x4000000b,0x50000015};
  // k12
  book->defs[0x0002f083]           = (Term*) malloc(sizeof(Term));
  book->defs[0x0002f083]->root     = 0xa0000000;
  book->defs[0x0002f083]->alen     = 0;
  book->defs[0x0002f083]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x0002f083]->nlen     = 25;
  book->defs[0x0002f083]->node     = (Node*) malloc(25 * sizeof(Node));
  book->defs[0x0002f083]->node[ 0] = (Node) {0xc0000001,0xa0000018};
  book->defs[0x0002f083]->node[ 1] = (Node) {0xc0000002,0xa0000017};
  book->defs[0x0002f083]->node[ 2] = (Node) {0xc0000003,0xa0000016};
  book->defs[0x0002f083]->node[ 3] = (Node) {0xc0000004,0xa0000015};
  book->defs[0x0002f083]->node[ 4] = (Node) {0xc0000005,0xa0000014};
  book->defs[0x0002f083]->node[ 5] = (Node) {0xc0000006,0xa0000013};
  book->defs[0x0002f083]->node[ 6] = (Node) {0xc0000007,0xa0000012};
  book->defs[0x0002f083]->node[ 7] = (Node) {0xc0000008,0xa0000011};
  book->defs[0x0002f083]->node[ 8] = (Node) {0xc0000009,0xa0000010};
  book->defs[0x0002f083]->node[ 9] = (Node) {0xc000000a,0xa000000f};
  book->defs[0x0002f083]->node[10] = (Node) {0xc000000b,0xa000000e};
  book->defs[0x0002f083]->node[11] = (Node) {0xa000000c,0xa000000d};
  book->defs[0x0002f083]->node[12] = (Node) {0x40000018,0x4000000d};
  book->defs[0x0002f083]->node[13] = (Node) {0x5000000c,0x4000000e};
  book->defs[0x0002f083]->node[14] = (Node) {0x5000000d,0x4000000f};
  book->defs[0x0002f083]->node[15] = (Node) {0x5000000e,0x40000010};
  book->defs[0x0002f083]->node[16] = (Node) {0x5000000f,0x40000011};
  book->defs[0x0002f083]->node[17] = (Node) {0x50000010,0x40000012};
  book->defs[0x0002f083]->node[18] = (Node) {0x50000011,0x40000013};
  book->defs[0x0002f083]->node[19] = (Node) {0x50000012,0x40000014};
  book->defs[0x0002f083]->node[20] = (Node) {0x50000013,0x40000015};
  book->defs[0x0002f083]->node[21] = (Node) {0x50000014,0x40000016};
  book->defs[0x0002f083]->node[22] = (Node) {0x50000015,0x40000017};
  book->defs[0x0002f083]->node[23] = (Node) {0x50000016,0x50000018};
  book->defs[0x0002f083]->node[24] = (Node) {0x4000000c,0x50000017};
  // k13
  book->defs[0x0002f084]           = (Term*) malloc(sizeof(Term));
  book->defs[0x0002f084]->root     = 0xa0000000;
  book->defs[0x0002f084]->alen     = 0;
  book->defs[0x0002f084]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x0002f084]->nlen     = 27;
  book->defs[0x0002f084]->node     = (Node*) malloc(27 * sizeof(Node));
  book->defs[0x0002f084]->node[ 0] = (Node) {0xc0000001,0xa000001a};
  book->defs[0x0002f084]->node[ 1] = (Node) {0xc0000002,0xa0000019};
  book->defs[0x0002f084]->node[ 2] = (Node) {0xc0000003,0xa0000018};
  book->defs[0x0002f084]->node[ 3] = (Node) {0xc0000004,0xa0000017};
  book->defs[0x0002f084]->node[ 4] = (Node) {0xc0000005,0xa0000016};
  book->defs[0x0002f084]->node[ 5] = (Node) {0xc0000006,0xa0000015};
  book->defs[0x0002f084]->node[ 6] = (Node) {0xc0000007,0xa0000014};
  book->defs[0x0002f084]->node[ 7] = (Node) {0xc0000008,0xa0000013};
  book->defs[0x0002f084]->node[ 8] = (Node) {0xc0000009,0xa0000012};
  book->defs[0x0002f084]->node[ 9] = (Node) {0xc000000a,0xa0000011};
  book->defs[0x0002f084]->node[10] = (Node) {0xc000000b,0xa0000010};
  book->defs[0x0002f084]->node[11] = (Node) {0xc000000c,0xa000000f};
  book->defs[0x0002f084]->node[12] = (Node) {0xa000000d,0xa000000e};
  book->defs[0x0002f084]->node[13] = (Node) {0x4000001a,0x4000000e};
  book->defs[0x0002f084]->node[14] = (Node) {0x5000000d,0x4000000f};
  book->defs[0x0002f084]->node[15] = (Node) {0x5000000e,0x40000010};
  book->defs[0x0002f084]->node[16] = (Node) {0x5000000f,0x40000011};
  book->defs[0x0002f084]->node[17] = (Node) {0x50000010,0x40000012};
  book->defs[0x0002f084]->node[18] = (Node) {0x50000011,0x40000013};
  book->defs[0x0002f084]->node[19] = (Node) {0x50000012,0x40000014};
  book->defs[0x0002f084]->node[20] = (Node) {0x50000013,0x40000015};
  book->defs[0x0002f084]->node[21] = (Node) {0x50000014,0x40000016};
  book->defs[0x0002f084]->node[22] = (Node) {0x50000015,0x40000017};
  book->defs[0x0002f084]->node[23] = (Node) {0x50000016,0x40000018};
  book->defs[0x0002f084]->node[24] = (Node) {0x50000017,0x40000019};
  book->defs[0x0002f084]->node[25] = (Node) {0x50000018,0x5000001a};
  book->defs[0x0002f084]->node[26] = (Node) {0x4000000d,0x50000019};
  // k14
  book->defs[0x0002f085]           = (Term*) malloc(sizeof(Term));
  book->defs[0x0002f085]->root     = 0xa0000000;
  book->defs[0x0002f085]->alen     = 0;
  book->defs[0x0002f085]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x0002f085]->nlen     = 29;
  book->defs[0x0002f085]->node     = (Node*) malloc(29 * sizeof(Node));
  book->defs[0x0002f085]->node[ 0] = (Node) {0xc0000001,0xa000001c};
  book->defs[0x0002f085]->node[ 1] = (Node) {0xc0000002,0xa000001b};
  book->defs[0x0002f085]->node[ 2] = (Node) {0xc0000003,0xa000001a};
  book->defs[0x0002f085]->node[ 3] = (Node) {0xc0000004,0xa0000019};
  book->defs[0x0002f085]->node[ 4] = (Node) {0xc0000005,0xa0000018};
  book->defs[0x0002f085]->node[ 5] = (Node) {0xc0000006,0xa0000017};
  book->defs[0x0002f085]->node[ 6] = (Node) {0xc0000007,0xa0000016};
  book->defs[0x0002f085]->node[ 7] = (Node) {0xc0000008,0xa0000015};
  book->defs[0x0002f085]->node[ 8] = (Node) {0xc0000009,0xa0000014};
  book->defs[0x0002f085]->node[ 9] = (Node) {0xc000000a,0xa0000013};
  book->defs[0x0002f085]->node[10] = (Node) {0xc000000b,0xa0000012};
  book->defs[0x0002f085]->node[11] = (Node) {0xc000000c,0xa0000011};
  book->defs[0x0002f085]->node[12] = (Node) {0xc000000d,0xa0000010};
  book->defs[0x0002f085]->node[13] = (Node) {0xa000000e,0xa000000f};
  book->defs[0x0002f085]->node[14] = (Node) {0x4000001c,0x4000000f};
  book->defs[0x0002f085]->node[15] = (Node) {0x5000000e,0x40000010};
  book->defs[0x0002f085]->node[16] = (Node) {0x5000000f,0x40000011};
  book->defs[0x0002f085]->node[17] = (Node) {0x50000010,0x40000012};
  book->defs[0x0002f085]->node[18] = (Node) {0x50000011,0x40000013};
  book->defs[0x0002f085]->node[19] = (Node) {0x50000012,0x40000014};
  book->defs[0x0002f085]->node[20] = (Node) {0x50000013,0x40000015};
  book->defs[0x0002f085]->node[21] = (Node) {0x50000014,0x40000016};
  book->defs[0x0002f085]->node[22] = (Node) {0x50000015,0x40000017};
  book->defs[0x0002f085]->node[23] = (Node) {0x50000016,0x40000018};
  book->defs[0x0002f085]->node[24] = (Node) {0x50000017,0x40000019};
  book->defs[0x0002f085]->node[25] = (Node) {0x50000018,0x4000001a};
  book->defs[0x0002f085]->node[26] = (Node) {0x50000019,0x4000001b};
  book->defs[0x0002f085]->node[27] = (Node) {0x5000001a,0x5000001c};
  book->defs[0x0002f085]->node[28] = (Node) {0x4000000e,0x5000001b};
  // k15
  book->defs[0x0002f086]           = (Term*) malloc(sizeof(Term));
  book->defs[0x0002f086]->root     = 0xa0000000;
  book->defs[0x0002f086]->alen     = 0;
  book->defs[0x0002f086]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x0002f086]->nlen     = 31;
  book->defs[0x0002f086]->node     = (Node*) malloc(31 * sizeof(Node));
  book->defs[0x0002f086]->node[ 0] = (Node) {0xc0000001,0xa000001e};
  book->defs[0x0002f086]->node[ 1] = (Node) {0xc0000002,0xa000001d};
  book->defs[0x0002f086]->node[ 2] = (Node) {0xc0000003,0xa000001c};
  book->defs[0x0002f086]->node[ 3] = (Node) {0xc0000004,0xa000001b};
  book->defs[0x0002f086]->node[ 4] = (Node) {0xc0000005,0xa000001a};
  book->defs[0x0002f086]->node[ 5] = (Node) {0xc0000006,0xa0000019};
  book->defs[0x0002f086]->node[ 6] = (Node) {0xc0000007,0xa0000018};
  book->defs[0x0002f086]->node[ 7] = (Node) {0xc0000008,0xa0000017};
  book->defs[0x0002f086]->node[ 8] = (Node) {0xc0000009,0xa0000016};
  book->defs[0x0002f086]->node[ 9] = (Node) {0xc000000a,0xa0000015};
  book->defs[0x0002f086]->node[10] = (Node) {0xc000000b,0xa0000014};
  book->defs[0x0002f086]->node[11] = (Node) {0xc000000c,0xa0000013};
  book->defs[0x0002f086]->node[12] = (Node) {0xc000000d,0xa0000012};
  book->defs[0x0002f086]->node[13] = (Node) {0xc000000e,0xa0000011};
  book->defs[0x0002f086]->node[14] = (Node) {0xa000000f,0xa0000010};
  book->defs[0x0002f086]->node[15] = (Node) {0x4000001e,0x40000010};
  book->defs[0x0002f086]->node[16] = (Node) {0x5000000f,0x40000011};
  book->defs[0x0002f086]->node[17] = (Node) {0x50000010,0x40000012};
  book->defs[0x0002f086]->node[18] = (Node) {0x50000011,0x40000013};
  book->defs[0x0002f086]->node[19] = (Node) {0x50000012,0x40000014};
  book->defs[0x0002f086]->node[20] = (Node) {0x50000013,0x40000015};
  book->defs[0x0002f086]->node[21] = (Node) {0x50000014,0x40000016};
  book->defs[0x0002f086]->node[22] = (Node) {0x50000015,0x40000017};
  book->defs[0x0002f086]->node[23] = (Node) {0x50000016,0x40000018};
  book->defs[0x0002f086]->node[24] = (Node) {0x50000017,0x40000019};
  book->defs[0x0002f086]->node[25] = (Node) {0x50000018,0x4000001a};
  book->defs[0x0002f086]->node[26] = (Node) {0x50000019,0x4000001b};
  book->defs[0x0002f086]->node[27] = (Node) {0x5000001a,0x4000001c};
  book->defs[0x0002f086]->node[28] = (Node) {0x5000001b,0x4000001d};
  book->defs[0x0002f086]->node[29] = (Node) {0x5000001c,0x5000001e};
  book->defs[0x0002f086]->node[30] = (Node) {0x4000000f,0x5000001d};
  // k16
  book->defs[0x0002f087]           = (Term*) malloc(sizeof(Term));
  book->defs[0x0002f087]->root     = 0xa0000000;
  book->defs[0x0002f087]->alen     = 0;
  book->defs[0x0002f087]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x0002f087]->nlen     = 33;
  book->defs[0x0002f087]->node     = (Node*) malloc(33 * sizeof(Node));
  book->defs[0x0002f087]->node[ 0] = (Node) {0xc0000001,0xa0000020};
  book->defs[0x0002f087]->node[ 1] = (Node) {0xc0000002,0xa000001f};
  book->defs[0x0002f087]->node[ 2] = (Node) {0xc0000003,0xa000001e};
  book->defs[0x0002f087]->node[ 3] = (Node) {0xc0000004,0xa000001d};
  book->defs[0x0002f087]->node[ 4] = (Node) {0xc0000005,0xa000001c};
  book->defs[0x0002f087]->node[ 5] = (Node) {0xc0000006,0xa000001b};
  book->defs[0x0002f087]->node[ 6] = (Node) {0xc0000007,0xa000001a};
  book->defs[0x0002f087]->node[ 7] = (Node) {0xc0000008,0xa0000019};
  book->defs[0x0002f087]->node[ 8] = (Node) {0xc0000009,0xa0000018};
  book->defs[0x0002f087]->node[ 9] = (Node) {0xc000000a,0xa0000017};
  book->defs[0x0002f087]->node[10] = (Node) {0xc000000b,0xa0000016};
  book->defs[0x0002f087]->node[11] = (Node) {0xc000000c,0xa0000015};
  book->defs[0x0002f087]->node[12] = (Node) {0xc000000d,0xa0000014};
  book->defs[0x0002f087]->node[13] = (Node) {0xc000000e,0xa0000013};
  book->defs[0x0002f087]->node[14] = (Node) {0xc000000f,0xa0000012};
  book->defs[0x0002f087]->node[15] = (Node) {0xa0000010,0xa0000011};
  book->defs[0x0002f087]->node[16] = (Node) {0x40000020,0x40000011};
  book->defs[0x0002f087]->node[17] = (Node) {0x50000010,0x40000012};
  book->defs[0x0002f087]->node[18] = (Node) {0x50000011,0x40000013};
  book->defs[0x0002f087]->node[19] = (Node) {0x50000012,0x40000014};
  book->defs[0x0002f087]->node[20] = (Node) {0x50000013,0x40000015};
  book->defs[0x0002f087]->node[21] = (Node) {0x50000014,0x40000016};
  book->defs[0x0002f087]->node[22] = (Node) {0x50000015,0x40000017};
  book->defs[0x0002f087]->node[23] = (Node) {0x50000016,0x40000018};
  book->defs[0x0002f087]->node[24] = (Node) {0x50000017,0x40000019};
  book->defs[0x0002f087]->node[25] = (Node) {0x50000018,0x4000001a};
  book->defs[0x0002f087]->node[26] = (Node) {0x50000019,0x4000001b};
  book->defs[0x0002f087]->node[27] = (Node) {0x5000001a,0x4000001c};
  book->defs[0x0002f087]->node[28] = (Node) {0x5000001b,0x4000001d};
  book->defs[0x0002f087]->node[29] = (Node) {0x5000001c,0x4000001e};
  book->defs[0x0002f087]->node[30] = (Node) {0x5000001d,0x4000001f};
  book->defs[0x0002f087]->node[31] = (Node) {0x5000001e,0x50000020};
  book->defs[0x0002f087]->node[32] = (Node) {0x40000010,0x5000001f};
  // k17
  book->defs[0x0002f088]           = (Term*) malloc(sizeof(Term));
  book->defs[0x0002f088]->root     = 0xa0000000;
  book->defs[0x0002f088]->alen     = 0;
  book->defs[0x0002f088]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x0002f088]->nlen     = 35;
  book->defs[0x0002f088]->node     = (Node*) malloc(35 * sizeof(Node));
  book->defs[0x0002f088]->node[ 0] = (Node) {0xc0000001,0xa0000022};
  book->defs[0x0002f088]->node[ 1] = (Node) {0xc0000002,0xa0000021};
  book->defs[0x0002f088]->node[ 2] = (Node) {0xc0000003,0xa0000020};
  book->defs[0x0002f088]->node[ 3] = (Node) {0xc0000004,0xa000001f};
  book->defs[0x0002f088]->node[ 4] = (Node) {0xc0000005,0xa000001e};
  book->defs[0x0002f088]->node[ 5] = (Node) {0xc0000006,0xa000001d};
  book->defs[0x0002f088]->node[ 6] = (Node) {0xc0000007,0xa000001c};
  book->defs[0x0002f088]->node[ 7] = (Node) {0xc0000008,0xa000001b};
  book->defs[0x0002f088]->node[ 8] = (Node) {0xc0000009,0xa000001a};
  book->defs[0x0002f088]->node[ 9] = (Node) {0xc000000a,0xa0000019};
  book->defs[0x0002f088]->node[10] = (Node) {0xc000000b,0xa0000018};
  book->defs[0x0002f088]->node[11] = (Node) {0xc000000c,0xa0000017};
  book->defs[0x0002f088]->node[12] = (Node) {0xc000000d,0xa0000016};
  book->defs[0x0002f088]->node[13] = (Node) {0xc000000e,0xa0000015};
  book->defs[0x0002f088]->node[14] = (Node) {0xc000000f,0xa0000014};
  book->defs[0x0002f088]->node[15] = (Node) {0xc0000010,0xa0000013};
  book->defs[0x0002f088]->node[16] = (Node) {0xa0000011,0xa0000012};
  book->defs[0x0002f088]->node[17] = (Node) {0x40000022,0x40000012};
  book->defs[0x0002f088]->node[18] = (Node) {0x50000011,0x40000013};
  book->defs[0x0002f088]->node[19] = (Node) {0x50000012,0x40000014};
  book->defs[0x0002f088]->node[20] = (Node) {0x50000013,0x40000015};
  book->defs[0x0002f088]->node[21] = (Node) {0x50000014,0x40000016};
  book->defs[0x0002f088]->node[22] = (Node) {0x50000015,0x40000017};
  book->defs[0x0002f088]->node[23] = (Node) {0x50000016,0x40000018};
  book->defs[0x0002f088]->node[24] = (Node) {0x50000017,0x40000019};
  book->defs[0x0002f088]->node[25] = (Node) {0x50000018,0x4000001a};
  book->defs[0x0002f088]->node[26] = (Node) {0x50000019,0x4000001b};
  book->defs[0x0002f088]->node[27] = (Node) {0x5000001a,0x4000001c};
  book->defs[0x0002f088]->node[28] = (Node) {0x5000001b,0x4000001d};
  book->defs[0x0002f088]->node[29] = (Node) {0x5000001c,0x4000001e};
  book->defs[0x0002f088]->node[30] = (Node) {0x5000001d,0x4000001f};
  book->defs[0x0002f088]->node[31] = (Node) {0x5000001e,0x40000020};
  book->defs[0x0002f088]->node[32] = (Node) {0x5000001f,0x40000021};
  book->defs[0x0002f088]->node[33] = (Node) {0x50000020,0x50000022};
  book->defs[0x0002f088]->node[34] = (Node) {0x40000011,0x50000021};
  // k18
  book->defs[0x0002f089]           = (Term*) malloc(sizeof(Term));
  book->defs[0x0002f089]->root     = 0xa0000000;
  book->defs[0x0002f089]->alen     = 0;
  book->defs[0x0002f089]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x0002f089]->nlen     = 37;
  book->defs[0x0002f089]->node     = (Node*) malloc(37 * sizeof(Node));
  book->defs[0x0002f089]->node[ 0] = (Node) {0xc0000001,0xa0000024};
  book->defs[0x0002f089]->node[ 1] = (Node) {0xc0000002,0xa0000023};
  book->defs[0x0002f089]->node[ 2] = (Node) {0xc0000003,0xa0000022};
  book->defs[0x0002f089]->node[ 3] = (Node) {0xc0000004,0xa0000021};
  book->defs[0x0002f089]->node[ 4] = (Node) {0xc0000005,0xa0000020};
  book->defs[0x0002f089]->node[ 5] = (Node) {0xc0000006,0xa000001f};
  book->defs[0x0002f089]->node[ 6] = (Node) {0xc0000007,0xa000001e};
  book->defs[0x0002f089]->node[ 7] = (Node) {0xc0000008,0xa000001d};
  book->defs[0x0002f089]->node[ 8] = (Node) {0xc0000009,0xa000001c};
  book->defs[0x0002f089]->node[ 9] = (Node) {0xc000000a,0xa000001b};
  book->defs[0x0002f089]->node[10] = (Node) {0xc000000b,0xa000001a};
  book->defs[0x0002f089]->node[11] = (Node) {0xc000000c,0xa0000019};
  book->defs[0x0002f089]->node[12] = (Node) {0xc000000d,0xa0000018};
  book->defs[0x0002f089]->node[13] = (Node) {0xc000000e,0xa0000017};
  book->defs[0x0002f089]->node[14] = (Node) {0xc000000f,0xa0000016};
  book->defs[0x0002f089]->node[15] = (Node) {0xc0000010,0xa0000015};
  book->defs[0x0002f089]->node[16] = (Node) {0xc0000011,0xa0000014};
  book->defs[0x0002f089]->node[17] = (Node) {0xa0000012,0xa0000013};
  book->defs[0x0002f089]->node[18] = (Node) {0x40000024,0x40000013};
  book->defs[0x0002f089]->node[19] = (Node) {0x50000012,0x40000014};
  book->defs[0x0002f089]->node[20] = (Node) {0x50000013,0x40000015};
  book->defs[0x0002f089]->node[21] = (Node) {0x50000014,0x40000016};
  book->defs[0x0002f089]->node[22] = (Node) {0x50000015,0x40000017};
  book->defs[0x0002f089]->node[23] = (Node) {0x50000016,0x40000018};
  book->defs[0x0002f089]->node[24] = (Node) {0x50000017,0x40000019};
  book->defs[0x0002f089]->node[25] = (Node) {0x50000018,0x4000001a};
  book->defs[0x0002f089]->node[26] = (Node) {0x50000019,0x4000001b};
  book->defs[0x0002f089]->node[27] = (Node) {0x5000001a,0x4000001c};
  book->defs[0x0002f089]->node[28] = (Node) {0x5000001b,0x4000001d};
  book->defs[0x0002f089]->node[29] = (Node) {0x5000001c,0x4000001e};
  book->defs[0x0002f089]->node[30] = (Node) {0x5000001d,0x4000001f};
  book->defs[0x0002f089]->node[31] = (Node) {0x5000001e,0x40000020};
  book->defs[0x0002f089]->node[32] = (Node) {0x5000001f,0x40000021};
  book->defs[0x0002f089]->node[33] = (Node) {0x50000020,0x40000022};
  book->defs[0x0002f089]->node[34] = (Node) {0x50000021,0x40000023};
  book->defs[0x0002f089]->node[35] = (Node) {0x50000022,0x50000024};
  book->defs[0x0002f089]->node[36] = (Node) {0x40000012,0x50000023};
  // k19
  book->defs[0x0002f08a]           = (Term*) malloc(sizeof(Term));
  book->defs[0x0002f08a]->root     = 0xa0000000;
  book->defs[0x0002f08a]->alen     = 0;
  book->defs[0x0002f08a]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x0002f08a]->nlen     = 39;
  book->defs[0x0002f08a]->node     = (Node*) malloc(39 * sizeof(Node));
  book->defs[0x0002f08a]->node[ 0] = (Node) {0xc0000001,0xa0000026};
  book->defs[0x0002f08a]->node[ 1] = (Node) {0xc0000002,0xa0000025};
  book->defs[0x0002f08a]->node[ 2] = (Node) {0xc0000003,0xa0000024};
  book->defs[0x0002f08a]->node[ 3] = (Node) {0xc0000004,0xa0000023};
  book->defs[0x0002f08a]->node[ 4] = (Node) {0xc0000005,0xa0000022};
  book->defs[0x0002f08a]->node[ 5] = (Node) {0xc0000006,0xa0000021};
  book->defs[0x0002f08a]->node[ 6] = (Node) {0xc0000007,0xa0000020};
  book->defs[0x0002f08a]->node[ 7] = (Node) {0xc0000008,0xa000001f};
  book->defs[0x0002f08a]->node[ 8] = (Node) {0xc0000009,0xa000001e};
  book->defs[0x0002f08a]->node[ 9] = (Node) {0xc000000a,0xa000001d};
  book->defs[0x0002f08a]->node[10] = (Node) {0xc000000b,0xa000001c};
  book->defs[0x0002f08a]->node[11] = (Node) {0xc000000c,0xa000001b};
  book->defs[0x0002f08a]->node[12] = (Node) {0xc000000d,0xa000001a};
  book->defs[0x0002f08a]->node[13] = (Node) {0xc000000e,0xa0000019};
  book->defs[0x0002f08a]->node[14] = (Node) {0xc000000f,0xa0000018};
  book->defs[0x0002f08a]->node[15] = (Node) {0xc0000010,0xa0000017};
  book->defs[0x0002f08a]->node[16] = (Node) {0xc0000011,0xa0000016};
  book->defs[0x0002f08a]->node[17] = (Node) {0xc0000012,0xa0000015};
  book->defs[0x0002f08a]->node[18] = (Node) {0xa0000013,0xa0000014};
  book->defs[0x0002f08a]->node[19] = (Node) {0x40000026,0x40000014};
  book->defs[0x0002f08a]->node[20] = (Node) {0x50000013,0x40000015};
  book->defs[0x0002f08a]->node[21] = (Node) {0x50000014,0x40000016};
  book->defs[0x0002f08a]->node[22] = (Node) {0x50000015,0x40000017};
  book->defs[0x0002f08a]->node[23] = (Node) {0x50000016,0x40000018};
  book->defs[0x0002f08a]->node[24] = (Node) {0x50000017,0x40000019};
  book->defs[0x0002f08a]->node[25] = (Node) {0x50000018,0x4000001a};
  book->defs[0x0002f08a]->node[26] = (Node) {0x50000019,0x4000001b};
  book->defs[0x0002f08a]->node[27] = (Node) {0x5000001a,0x4000001c};
  book->defs[0x0002f08a]->node[28] = (Node) {0x5000001b,0x4000001d};
  book->defs[0x0002f08a]->node[29] = (Node) {0x5000001c,0x4000001e};
  book->defs[0x0002f08a]->node[30] = (Node) {0x5000001d,0x4000001f};
  book->defs[0x0002f08a]->node[31] = (Node) {0x5000001e,0x40000020};
  book->defs[0x0002f08a]->node[32] = (Node) {0x5000001f,0x40000021};
  book->defs[0x0002f08a]->node[33] = (Node) {0x50000020,0x40000022};
  book->defs[0x0002f08a]->node[34] = (Node) {0x50000021,0x40000023};
  book->defs[0x0002f08a]->node[35] = (Node) {0x50000022,0x40000024};
  book->defs[0x0002f08a]->node[36] = (Node) {0x50000023,0x40000025};
  book->defs[0x0002f08a]->node[37] = (Node) {0x50000024,0x50000026};
  book->defs[0x0002f08a]->node[38] = (Node) {0x40000013,0x50000025};
  // k20
  book->defs[0x0002f0c1]           = (Term*) malloc(sizeof(Term));
  book->defs[0x0002f0c1]->root     = 0xa0000000;
  book->defs[0x0002f0c1]->alen     = 0;
  book->defs[0x0002f0c1]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x0002f0c1]->nlen     = 41;
  book->defs[0x0002f0c1]->node     = (Node*) malloc(41 * sizeof(Node));
  book->defs[0x0002f0c1]->node[ 0] = (Node) {0xc0000001,0xa0000028};
  book->defs[0x0002f0c1]->node[ 1] = (Node) {0xc0000002,0xa0000027};
  book->defs[0x0002f0c1]->node[ 2] = (Node) {0xc0000003,0xa0000026};
  book->defs[0x0002f0c1]->node[ 3] = (Node) {0xc0000004,0xa0000025};
  book->defs[0x0002f0c1]->node[ 4] = (Node) {0xc0000005,0xa0000024};
  book->defs[0x0002f0c1]->node[ 5] = (Node) {0xc0000006,0xa0000023};
  book->defs[0x0002f0c1]->node[ 6] = (Node) {0xc0000007,0xa0000022};
  book->defs[0x0002f0c1]->node[ 7] = (Node) {0xc0000008,0xa0000021};
  book->defs[0x0002f0c1]->node[ 8] = (Node) {0xc0000009,0xa0000020};
  book->defs[0x0002f0c1]->node[ 9] = (Node) {0xc000000a,0xa000001f};
  book->defs[0x0002f0c1]->node[10] = (Node) {0xc000000b,0xa000001e};
  book->defs[0x0002f0c1]->node[11] = (Node) {0xc000000c,0xa000001d};
  book->defs[0x0002f0c1]->node[12] = (Node) {0xc000000d,0xa000001c};
  book->defs[0x0002f0c1]->node[13] = (Node) {0xc000000e,0xa000001b};
  book->defs[0x0002f0c1]->node[14] = (Node) {0xc000000f,0xa000001a};
  book->defs[0x0002f0c1]->node[15] = (Node) {0xc0000010,0xa0000019};
  book->defs[0x0002f0c1]->node[16] = (Node) {0xc0000011,0xa0000018};
  book->defs[0x0002f0c1]->node[17] = (Node) {0xc0000012,0xa0000017};
  book->defs[0x0002f0c1]->node[18] = (Node) {0xc0000013,0xa0000016};
  book->defs[0x0002f0c1]->node[19] = (Node) {0xa0000014,0xa0000015};
  book->defs[0x0002f0c1]->node[20] = (Node) {0x40000028,0x40000015};
  book->defs[0x0002f0c1]->node[21] = (Node) {0x50000014,0x40000016};
  book->defs[0x0002f0c1]->node[22] = (Node) {0x50000015,0x40000017};
  book->defs[0x0002f0c1]->node[23] = (Node) {0x50000016,0x40000018};
  book->defs[0x0002f0c1]->node[24] = (Node) {0x50000017,0x40000019};
  book->defs[0x0002f0c1]->node[25] = (Node) {0x50000018,0x4000001a};
  book->defs[0x0002f0c1]->node[26] = (Node) {0x50000019,0x4000001b};
  book->defs[0x0002f0c1]->node[27] = (Node) {0x5000001a,0x4000001c};
  book->defs[0x0002f0c1]->node[28] = (Node) {0x5000001b,0x4000001d};
  book->defs[0x0002f0c1]->node[29] = (Node) {0x5000001c,0x4000001e};
  book->defs[0x0002f0c1]->node[30] = (Node) {0x5000001d,0x4000001f};
  book->defs[0x0002f0c1]->node[31] = (Node) {0x5000001e,0x40000020};
  book->defs[0x0002f0c1]->node[32] = (Node) {0x5000001f,0x40000021};
  book->defs[0x0002f0c1]->node[33] = (Node) {0x50000020,0x40000022};
  book->defs[0x0002f0c1]->node[34] = (Node) {0x50000021,0x40000023};
  book->defs[0x0002f0c1]->node[35] = (Node) {0x50000022,0x40000024};
  book->defs[0x0002f0c1]->node[36] = (Node) {0x50000023,0x40000025};
  book->defs[0x0002f0c1]->node[37] = (Node) {0x50000024,0x40000026};
  book->defs[0x0002f0c1]->node[38] = (Node) {0x50000025,0x40000027};
  book->defs[0x0002f0c1]->node[39] = (Node) {0x50000026,0x50000028};
  book->defs[0x0002f0c1]->node[40] = (Node) {0x40000014,0x50000027};
  // k21
  book->defs[0x0002f0c2]           = (Term*) malloc(sizeof(Term));
  book->defs[0x0002f0c2]->root     = 0xa0000000;
  book->defs[0x0002f0c2]->alen     = 0;
  book->defs[0x0002f0c2]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x0002f0c2]->nlen     = 43;
  book->defs[0x0002f0c2]->node     = (Node*) malloc(43 * sizeof(Node));
  book->defs[0x0002f0c2]->node[ 0] = (Node) {0xc0000001,0xa000002a};
  book->defs[0x0002f0c2]->node[ 1] = (Node) {0xc0000002,0xa0000029};
  book->defs[0x0002f0c2]->node[ 2] = (Node) {0xc0000003,0xa0000028};
  book->defs[0x0002f0c2]->node[ 3] = (Node) {0xc0000004,0xa0000027};
  book->defs[0x0002f0c2]->node[ 4] = (Node) {0xc0000005,0xa0000026};
  book->defs[0x0002f0c2]->node[ 5] = (Node) {0xc0000006,0xa0000025};
  book->defs[0x0002f0c2]->node[ 6] = (Node) {0xc0000007,0xa0000024};
  book->defs[0x0002f0c2]->node[ 7] = (Node) {0xc0000008,0xa0000023};
  book->defs[0x0002f0c2]->node[ 8] = (Node) {0xc0000009,0xa0000022};
  book->defs[0x0002f0c2]->node[ 9] = (Node) {0xc000000a,0xa0000021};
  book->defs[0x0002f0c2]->node[10] = (Node) {0xc000000b,0xa0000020};
  book->defs[0x0002f0c2]->node[11] = (Node) {0xc000000c,0xa000001f};
  book->defs[0x0002f0c2]->node[12] = (Node) {0xc000000d,0xa000001e};
  book->defs[0x0002f0c2]->node[13] = (Node) {0xc000000e,0xa000001d};
  book->defs[0x0002f0c2]->node[14] = (Node) {0xc000000f,0xa000001c};
  book->defs[0x0002f0c2]->node[15] = (Node) {0xc0000010,0xa000001b};
  book->defs[0x0002f0c2]->node[16] = (Node) {0xc0000011,0xa000001a};
  book->defs[0x0002f0c2]->node[17] = (Node) {0xc0000012,0xa0000019};
  book->defs[0x0002f0c2]->node[18] = (Node) {0xc0000013,0xa0000018};
  book->defs[0x0002f0c2]->node[19] = (Node) {0xc0000014,0xa0000017};
  book->defs[0x0002f0c2]->node[20] = (Node) {0xa0000015,0xa0000016};
  book->defs[0x0002f0c2]->node[21] = (Node) {0x4000002a,0x40000016};
  book->defs[0x0002f0c2]->node[22] = (Node) {0x50000015,0x40000017};
  book->defs[0x0002f0c2]->node[23] = (Node) {0x50000016,0x40000018};
  book->defs[0x0002f0c2]->node[24] = (Node) {0x50000017,0x40000019};
  book->defs[0x0002f0c2]->node[25] = (Node) {0x50000018,0x4000001a};
  book->defs[0x0002f0c2]->node[26] = (Node) {0x50000019,0x4000001b};
  book->defs[0x0002f0c2]->node[27] = (Node) {0x5000001a,0x4000001c};
  book->defs[0x0002f0c2]->node[28] = (Node) {0x5000001b,0x4000001d};
  book->defs[0x0002f0c2]->node[29] = (Node) {0x5000001c,0x4000001e};
  book->defs[0x0002f0c2]->node[30] = (Node) {0x5000001d,0x4000001f};
  book->defs[0x0002f0c2]->node[31] = (Node) {0x5000001e,0x40000020};
  book->defs[0x0002f0c2]->node[32] = (Node) {0x5000001f,0x40000021};
  book->defs[0x0002f0c2]->node[33] = (Node) {0x50000020,0x40000022};
  book->defs[0x0002f0c2]->node[34] = (Node) {0x50000021,0x40000023};
  book->defs[0x0002f0c2]->node[35] = (Node) {0x50000022,0x40000024};
  book->defs[0x0002f0c2]->node[36] = (Node) {0x50000023,0x40000025};
  book->defs[0x0002f0c2]->node[37] = (Node) {0x50000024,0x40000026};
  book->defs[0x0002f0c2]->node[38] = (Node) {0x50000025,0x40000027};
  book->defs[0x0002f0c2]->node[39] = (Node) {0x50000026,0x40000028};
  book->defs[0x0002f0c2]->node[40] = (Node) {0x50000027,0x40000029};
  book->defs[0x0002f0c2]->node[41] = (Node) {0x50000028,0x5000002a};
  book->defs[0x0002f0c2]->node[42] = (Node) {0x40000015,0x50000029};
  // k22
  book->defs[0x0002f0c3]           = (Term*) malloc(sizeof(Term));
  book->defs[0x0002f0c3]->root     = 0xa0000000;
  book->defs[0x0002f0c3]->alen     = 0;
  book->defs[0x0002f0c3]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x0002f0c3]->nlen     = 45;
  book->defs[0x0002f0c3]->node     = (Node*) malloc(45 * sizeof(Node));
  book->defs[0x0002f0c3]->node[ 0] = (Node) {0xc0000001,0xa000002c};
  book->defs[0x0002f0c3]->node[ 1] = (Node) {0xc0000002,0xa000002b};
  book->defs[0x0002f0c3]->node[ 2] = (Node) {0xc0000003,0xa000002a};
  book->defs[0x0002f0c3]->node[ 3] = (Node) {0xc0000004,0xa0000029};
  book->defs[0x0002f0c3]->node[ 4] = (Node) {0xc0000005,0xa0000028};
  book->defs[0x0002f0c3]->node[ 5] = (Node) {0xc0000006,0xa0000027};
  book->defs[0x0002f0c3]->node[ 6] = (Node) {0xc0000007,0xa0000026};
  book->defs[0x0002f0c3]->node[ 7] = (Node) {0xc0000008,0xa0000025};
  book->defs[0x0002f0c3]->node[ 8] = (Node) {0xc0000009,0xa0000024};
  book->defs[0x0002f0c3]->node[ 9] = (Node) {0xc000000a,0xa0000023};
  book->defs[0x0002f0c3]->node[10] = (Node) {0xc000000b,0xa0000022};
  book->defs[0x0002f0c3]->node[11] = (Node) {0xc000000c,0xa0000021};
  book->defs[0x0002f0c3]->node[12] = (Node) {0xc000000d,0xa0000020};
  book->defs[0x0002f0c3]->node[13] = (Node) {0xc000000e,0xa000001f};
  book->defs[0x0002f0c3]->node[14] = (Node) {0xc000000f,0xa000001e};
  book->defs[0x0002f0c3]->node[15] = (Node) {0xc0000010,0xa000001d};
  book->defs[0x0002f0c3]->node[16] = (Node) {0xc0000011,0xa000001c};
  book->defs[0x0002f0c3]->node[17] = (Node) {0xc0000012,0xa000001b};
  book->defs[0x0002f0c3]->node[18] = (Node) {0xc0000013,0xa000001a};
  book->defs[0x0002f0c3]->node[19] = (Node) {0xc0000014,0xa0000019};
  book->defs[0x0002f0c3]->node[20] = (Node) {0xc0000015,0xa0000018};
  book->defs[0x0002f0c3]->node[21] = (Node) {0xa0000016,0xa0000017};
  book->defs[0x0002f0c3]->node[22] = (Node) {0x4000002c,0x40000017};
  book->defs[0x0002f0c3]->node[23] = (Node) {0x50000016,0x40000018};
  book->defs[0x0002f0c3]->node[24] = (Node) {0x50000017,0x40000019};
  book->defs[0x0002f0c3]->node[25] = (Node) {0x50000018,0x4000001a};
  book->defs[0x0002f0c3]->node[26] = (Node) {0x50000019,0x4000001b};
  book->defs[0x0002f0c3]->node[27] = (Node) {0x5000001a,0x4000001c};
  book->defs[0x0002f0c3]->node[28] = (Node) {0x5000001b,0x4000001d};
  book->defs[0x0002f0c3]->node[29] = (Node) {0x5000001c,0x4000001e};
  book->defs[0x0002f0c3]->node[30] = (Node) {0x5000001d,0x4000001f};
  book->defs[0x0002f0c3]->node[31] = (Node) {0x5000001e,0x40000020};
  book->defs[0x0002f0c3]->node[32] = (Node) {0x5000001f,0x40000021};
  book->defs[0x0002f0c3]->node[33] = (Node) {0x50000020,0x40000022};
  book->defs[0x0002f0c3]->node[34] = (Node) {0x50000021,0x40000023};
  book->defs[0x0002f0c3]->node[35] = (Node) {0x50000022,0x40000024};
  book->defs[0x0002f0c3]->node[36] = (Node) {0x50000023,0x40000025};
  book->defs[0x0002f0c3]->node[37] = (Node) {0x50000024,0x40000026};
  book->defs[0x0002f0c3]->node[38] = (Node) {0x50000025,0x40000027};
  book->defs[0x0002f0c3]->node[39] = (Node) {0x50000026,0x40000028};
  book->defs[0x0002f0c3]->node[40] = (Node) {0x50000027,0x40000029};
  book->defs[0x0002f0c3]->node[41] = (Node) {0x50000028,0x4000002a};
  book->defs[0x0002f0c3]->node[42] = (Node) {0x50000029,0x4000002b};
  book->defs[0x0002f0c3]->node[43] = (Node) {0x5000002a,0x5000002c};
  book->defs[0x0002f0c3]->node[44] = (Node) {0x40000016,0x5000002b};
  // k23
  book->defs[0x0002f0c4]           = (Term*) malloc(sizeof(Term));
  book->defs[0x0002f0c4]->root     = 0xa0000000;
  book->defs[0x0002f0c4]->alen     = 0;
  book->defs[0x0002f0c4]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x0002f0c4]->nlen     = 47;
  book->defs[0x0002f0c4]->node     = (Node*) malloc(47 * sizeof(Node));
  book->defs[0x0002f0c4]->node[ 0] = (Node) {0xc0000001,0xa000002e};
  book->defs[0x0002f0c4]->node[ 1] = (Node) {0xc0000002,0xa000002d};
  book->defs[0x0002f0c4]->node[ 2] = (Node) {0xc0000003,0xa000002c};
  book->defs[0x0002f0c4]->node[ 3] = (Node) {0xc0000004,0xa000002b};
  book->defs[0x0002f0c4]->node[ 4] = (Node) {0xc0000005,0xa000002a};
  book->defs[0x0002f0c4]->node[ 5] = (Node) {0xc0000006,0xa0000029};
  book->defs[0x0002f0c4]->node[ 6] = (Node) {0xc0000007,0xa0000028};
  book->defs[0x0002f0c4]->node[ 7] = (Node) {0xc0000008,0xa0000027};
  book->defs[0x0002f0c4]->node[ 8] = (Node) {0xc0000009,0xa0000026};
  book->defs[0x0002f0c4]->node[ 9] = (Node) {0xc000000a,0xa0000025};
  book->defs[0x0002f0c4]->node[10] = (Node) {0xc000000b,0xa0000024};
  book->defs[0x0002f0c4]->node[11] = (Node) {0xc000000c,0xa0000023};
  book->defs[0x0002f0c4]->node[12] = (Node) {0xc000000d,0xa0000022};
  book->defs[0x0002f0c4]->node[13] = (Node) {0xc000000e,0xa0000021};
  book->defs[0x0002f0c4]->node[14] = (Node) {0xc000000f,0xa0000020};
  book->defs[0x0002f0c4]->node[15] = (Node) {0xc0000010,0xa000001f};
  book->defs[0x0002f0c4]->node[16] = (Node) {0xc0000011,0xa000001e};
  book->defs[0x0002f0c4]->node[17] = (Node) {0xc0000012,0xa000001d};
  book->defs[0x0002f0c4]->node[18] = (Node) {0xc0000013,0xa000001c};
  book->defs[0x0002f0c4]->node[19] = (Node) {0xc0000014,0xa000001b};
  book->defs[0x0002f0c4]->node[20] = (Node) {0xc0000015,0xa000001a};
  book->defs[0x0002f0c4]->node[21] = (Node) {0xc0000016,0xa0000019};
  book->defs[0x0002f0c4]->node[22] = (Node) {0xa0000017,0xa0000018};
  book->defs[0x0002f0c4]->node[23] = (Node) {0x4000002e,0x40000018};
  book->defs[0x0002f0c4]->node[24] = (Node) {0x50000017,0x40000019};
  book->defs[0x0002f0c4]->node[25] = (Node) {0x50000018,0x4000001a};
  book->defs[0x0002f0c4]->node[26] = (Node) {0x50000019,0x4000001b};
  book->defs[0x0002f0c4]->node[27] = (Node) {0x5000001a,0x4000001c};
  book->defs[0x0002f0c4]->node[28] = (Node) {0x5000001b,0x4000001d};
  book->defs[0x0002f0c4]->node[29] = (Node) {0x5000001c,0x4000001e};
  book->defs[0x0002f0c4]->node[30] = (Node) {0x5000001d,0x4000001f};
  book->defs[0x0002f0c4]->node[31] = (Node) {0x5000001e,0x40000020};
  book->defs[0x0002f0c4]->node[32] = (Node) {0x5000001f,0x40000021};
  book->defs[0x0002f0c4]->node[33] = (Node) {0x50000020,0x40000022};
  book->defs[0x0002f0c4]->node[34] = (Node) {0x50000021,0x40000023};
  book->defs[0x0002f0c4]->node[35] = (Node) {0x50000022,0x40000024};
  book->defs[0x0002f0c4]->node[36] = (Node) {0x50000023,0x40000025};
  book->defs[0x0002f0c4]->node[37] = (Node) {0x50000024,0x40000026};
  book->defs[0x0002f0c4]->node[38] = (Node) {0x50000025,0x40000027};
  book->defs[0x0002f0c4]->node[39] = (Node) {0x50000026,0x40000028};
  book->defs[0x0002f0c4]->node[40] = (Node) {0x50000027,0x40000029};
  book->defs[0x0002f0c4]->node[41] = (Node) {0x50000028,0x4000002a};
  book->defs[0x0002f0c4]->node[42] = (Node) {0x50000029,0x4000002b};
  book->defs[0x0002f0c4]->node[43] = (Node) {0x5000002a,0x4000002c};
  book->defs[0x0002f0c4]->node[44] = (Node) {0x5000002b,0x4000002d};
  book->defs[0x0002f0c4]->node[45] = (Node) {0x5000002c,0x5000002e};
  book->defs[0x0002f0c4]->node[46] = (Node) {0x40000017,0x5000002d};
  // k24
  book->defs[0x0002f0c5]           = (Term*) malloc(sizeof(Term));
  book->defs[0x0002f0c5]->root     = 0xa0000000;
  book->defs[0x0002f0c5]->alen     = 0;
  book->defs[0x0002f0c5]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x0002f0c5]->nlen     = 49;
  book->defs[0x0002f0c5]->node     = (Node*) malloc(49 * sizeof(Node));
  book->defs[0x0002f0c5]->node[ 0] = (Node) {0xc0000001,0xa0000030};
  book->defs[0x0002f0c5]->node[ 1] = (Node) {0xc0000002,0xa000002f};
  book->defs[0x0002f0c5]->node[ 2] = (Node) {0xc0000003,0xa000002e};
  book->defs[0x0002f0c5]->node[ 3] = (Node) {0xc0000004,0xa000002d};
  book->defs[0x0002f0c5]->node[ 4] = (Node) {0xc0000005,0xa000002c};
  book->defs[0x0002f0c5]->node[ 5] = (Node) {0xc0000006,0xa000002b};
  book->defs[0x0002f0c5]->node[ 6] = (Node) {0xc0000007,0xa000002a};
  book->defs[0x0002f0c5]->node[ 7] = (Node) {0xc0000008,0xa0000029};
  book->defs[0x0002f0c5]->node[ 8] = (Node) {0xc0000009,0xa0000028};
  book->defs[0x0002f0c5]->node[ 9] = (Node) {0xc000000a,0xa0000027};
  book->defs[0x0002f0c5]->node[10] = (Node) {0xc000000b,0xa0000026};
  book->defs[0x0002f0c5]->node[11] = (Node) {0xc000000c,0xa0000025};
  book->defs[0x0002f0c5]->node[12] = (Node) {0xc000000d,0xa0000024};
  book->defs[0x0002f0c5]->node[13] = (Node) {0xc000000e,0xa0000023};
  book->defs[0x0002f0c5]->node[14] = (Node) {0xc000000f,0xa0000022};
  book->defs[0x0002f0c5]->node[15] = (Node) {0xc0000010,0xa0000021};
  book->defs[0x0002f0c5]->node[16] = (Node) {0xc0000011,0xa0000020};
  book->defs[0x0002f0c5]->node[17] = (Node) {0xc0000012,0xa000001f};
  book->defs[0x0002f0c5]->node[18] = (Node) {0xc0000013,0xa000001e};
  book->defs[0x0002f0c5]->node[19] = (Node) {0xc0000014,0xa000001d};
  book->defs[0x0002f0c5]->node[20] = (Node) {0xc0000015,0xa000001c};
  book->defs[0x0002f0c5]->node[21] = (Node) {0xc0000016,0xa000001b};
  book->defs[0x0002f0c5]->node[22] = (Node) {0xc0000017,0xa000001a};
  book->defs[0x0002f0c5]->node[23] = (Node) {0xa0000018,0xa0000019};
  book->defs[0x0002f0c5]->node[24] = (Node) {0x40000030,0x40000019};
  book->defs[0x0002f0c5]->node[25] = (Node) {0x50000018,0x4000001a};
  book->defs[0x0002f0c5]->node[26] = (Node) {0x50000019,0x4000001b};
  book->defs[0x0002f0c5]->node[27] = (Node) {0x5000001a,0x4000001c};
  book->defs[0x0002f0c5]->node[28] = (Node) {0x5000001b,0x4000001d};
  book->defs[0x0002f0c5]->node[29] = (Node) {0x5000001c,0x4000001e};
  book->defs[0x0002f0c5]->node[30] = (Node) {0x5000001d,0x4000001f};
  book->defs[0x0002f0c5]->node[31] = (Node) {0x5000001e,0x40000020};
  book->defs[0x0002f0c5]->node[32] = (Node) {0x5000001f,0x40000021};
  book->defs[0x0002f0c5]->node[33] = (Node) {0x50000020,0x40000022};
  book->defs[0x0002f0c5]->node[34] = (Node) {0x50000021,0x40000023};
  book->defs[0x0002f0c5]->node[35] = (Node) {0x50000022,0x40000024};
  book->defs[0x0002f0c5]->node[36] = (Node) {0x50000023,0x40000025};
  book->defs[0x0002f0c5]->node[37] = (Node) {0x50000024,0x40000026};
  book->defs[0x0002f0c5]->node[38] = (Node) {0x50000025,0x40000027};
  book->defs[0x0002f0c5]->node[39] = (Node) {0x50000026,0x40000028};
  book->defs[0x0002f0c5]->node[40] = (Node) {0x50000027,0x40000029};
  book->defs[0x0002f0c5]->node[41] = (Node) {0x50000028,0x4000002a};
  book->defs[0x0002f0c5]->node[42] = (Node) {0x50000029,0x4000002b};
  book->defs[0x0002f0c5]->node[43] = (Node) {0x5000002a,0x4000002c};
  book->defs[0x0002f0c5]->node[44] = (Node) {0x5000002b,0x4000002d};
  book->defs[0x0002f0c5]->node[45] = (Node) {0x5000002c,0x4000002e};
  book->defs[0x0002f0c5]->node[46] = (Node) {0x5000002d,0x4000002f};
  book->defs[0x0002f0c5]->node[47] = (Node) {0x5000002e,0x50000030};
  book->defs[0x0002f0c5]->node[48] = (Node) {0x40000018,0x5000002f};
  // low
  book->defs[0x00030cfb]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00030cfb]->root     = 0xa0000000;
  book->defs[0x00030cfb]->alen     = 0;
  book->defs[0x00030cfb]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00030cfb]->nlen     = 4;
  book->defs[0x00030cfb]->node     = (Node*) malloc(4 * sizeof(Node));
  book->defs[0x00030cfb]->node[ 0] = (Node) {0xa0000001,0x50000003};
  book->defs[0x00030cfb]->node[ 1] = (Node) {0x10c33ed9,0xa0000002};
  book->defs[0x00030cfb]->node[ 2] = (Node) {0x10c33ed3,0xa0000003};
  book->defs[0x00030cfb]->node[ 3] = (Node) {0x1000000f,0x50000000};
  // mul
  book->defs[0x00031e70]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00031e70]->root     = 0xa0000000;
  book->defs[0x00031e70]->alen     = 0;
  book->defs[0x00031e70]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00031e70]->nlen     = 5;
  book->defs[0x00031e70]->node     = (Node*) malloc(5 * sizeof(Node));
  book->defs[0x00031e70]->node[ 0] = (Node) {0xa0000001,0xa0000002};
  book->defs[0x00031e70]->node[ 1] = (Node) {0x50000003,0x50000004};
  book->defs[0x00031e70]->node[ 2] = (Node) {0xa0000003,0xa0000004};
  book->defs[0x00031e70]->node[ 3] = (Node) {0x40000004,0x40000001};
  book->defs[0x00031e70]->node[ 4] = (Node) {0x40000003,0x50000001};
  // nid
  book->defs[0x00032b68]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00032b68]->root     = 0xa0000000;
  book->defs[0x00032b68]->alen     = 0;
  book->defs[0x00032b68]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00032b68]->nlen     = 3;
  book->defs[0x00032b68]->node     = (Node*) malloc(3 * sizeof(Node));
  book->defs[0x00032b68]->node[ 0] = (Node) {0xa0000001,0x50000002};
  book->defs[0x00032b68]->node[ 1] = (Node) {0x10cada1d,0xa0000002};
  book->defs[0x00032b68]->node[ 2] = (Node) {0x10000024,0x50000000};
  // not
  book->defs[0x00032cf8]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00032cf8]->root     = 0xa0000000;
  book->defs[0x00032cf8]->alen     = 0;
  book->defs[0x00032cf8]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00032cf8]->nlen     = 5;
  book->defs[0x00032cf8]->node     = (Node*) malloc(5 * sizeof(Node));
  book->defs[0x00032cf8]->node[ 0] = (Node) {0xa0000001,0xa0000003};
  book->defs[0x00032cf8]->node[ 1] = (Node) {0x40000004,0xa0000002};
  book->defs[0x00032cf8]->node[ 2] = (Node) {0x40000003,0x50000004};
  book->defs[0x00032cf8]->node[ 3] = (Node) {0x40000002,0xa0000004};
  book->defs[0x00032cf8]->node[ 4] = (Node) {0x40000001,0x50000002};
  // run
  book->defs[0x00036e72]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00036e72]->root     = 0xa0000000;
  book->defs[0x00036e72]->alen     = 0;
  book->defs[0x00036e72]->acts     = (Wire*) malloc(0 * sizeof(Wire));
  book->defs[0x00036e72]->nlen     = 4;
  book->defs[0x00036e72]->node     = (Node*) malloc(4 * sizeof(Node));
  book->defs[0x00036e72]->node[ 0] = (Node) {0xa0000001,0x50000003};
  book->defs[0x00036e72]->node[ 1] = (Node) {0x10db9c99,0xa0000002};
  book->defs[0x00036e72]->node[ 2] = (Node) {0x10db9c93,0xa0000003};
  book->defs[0x00036e72]->node[ 3] = (Node) {0x1000000f,0x50000000};
  // brnS
  book->defs[0x009b6c9d]           = (Term*) malloc(sizeof(Term));
  book->defs[0x009b6c9d]->root     = 0xa0000000;
  book->defs[0x009b6c9d]->alen     = 2;
  book->defs[0x009b6c9d]->acts     = (Wire*) malloc(2 * sizeof(Wire));
  book->defs[0x009b6c9d]->acts[ 0] = mkwire(0x10026db2,0xa0000003);
  book->defs[0x009b6c9d]->acts[ 1] = mkwire(0x10026db2,0xa0000004);
  book->defs[0x009b6c9d]->nlen     = 5;
  book->defs[0x009b6c9d]->node     = (Node*) malloc(5 * sizeof(Node));
  book->defs[0x009b6c9d]->node[ 0] = (Node) {0xb0000001,0xa0000002};
  book->defs[0x009b6c9d]->node[ 1] = (Node) {0x40000003,0x40000004};
  book->defs[0x009b6c9d]->node[ 2] = (Node) {0x50000003,0x50000004};
  book->defs[0x009b6c9d]->node[ 3] = (Node) {0x40000001,0x40000002};
  book->defs[0x009b6c9d]->node[ 4] = (Node) {0x50000001,0x50000002};
  // brnZ
  book->defs[0x009b6ca4]           = (Term*) malloc(sizeof(Term));
  book->defs[0x009b6ca4]->root     = 0x50000000;
  book->defs[0x009b6ca4]->alen     = 2;
  book->defs[0x009b6ca4]->acts     = (Wire*) malloc(2 * sizeof(Wire));
  book->defs[0x009b6ca4]->acts[ 0] = mkwire(0x10036e72,0xa0000000);
  book->defs[0x009b6ca4]->acts[ 1] = mkwire(0x10031e70,0xa0000001);
  book->defs[0x009b6ca4]->nlen     = 5;
  book->defs[0x009b6ca4]->node     = (Node*) malloc(5 * sizeof(Node));
  book->defs[0x009b6ca4]->node[ 0] = (Node) {0x50000004,0x30000000};
  book->defs[0x009b6ca4]->node[ 1] = (Node) {0x100009c3,0xa0000002};
  book->defs[0x009b6ca4]->node[ 2] = (Node) {0x100009c6,0xa0000003};
  book->defs[0x009b6ca4]->node[ 3] = (Node) {0x10000013,0xa0000004};
  book->defs[0x009b6ca4]->node[ 4] = (Node) {0x1000000f,0x40000000};
  // decI
  book->defs[0x00a299d3]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00a299d3]->root     = 0xa0000000;
  book->defs[0x00a299d3]->alen     = 1;
  book->defs[0x00a299d3]->acts     = (Wire*) malloc(1 * sizeof(Wire));
  book->defs[0x00a299d3]->acts[ 0] = mkwire(0x10030cfb,0xa0000001);
  book->defs[0x00a299d3]->nlen     = 2;
  book->defs[0x00a299d3]->node     = (Node*) malloc(2 * sizeof(Node));
  book->defs[0x00a299d3]->node[ 0] = (Node) {0x40000001,0x50000001};
  book->defs[0x00a299d3]->node[ 1] = (Node) {0x40000000,0x50000000};
  // decO
  book->defs[0x00a299d9]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00a299d9]->root     = 0xa0000000;
  book->defs[0x00a299d9]->alen     = 2;
  book->defs[0x00a299d9]->acts     = (Wire*) malloc(2 * sizeof(Wire));
  book->defs[0x00a299d9]->acts[ 0] = mkwire(0x10000013,0xa0000001);
  book->defs[0x00a299d9]->acts[ 1] = mkwire(0x10028a67,0xa0000002);
  book->defs[0x00a299d9]->nlen     = 3;
  book->defs[0x00a299d9]->node     = (Node*) malloc(3 * sizeof(Node));
  book->defs[0x00a299d9]->node[ 0] = (Node) {0x40000002,0x50000001};
  book->defs[0x00a299d9]->node[ 1] = (Node) {0x50000002,0x50000000};
  book->defs[0x00a299d9]->node[ 2] = (Node) {0x40000000,0x40000001};
  // lowI
  book->defs[0x00c33ed3]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00c33ed3]->root     = 0xa0000000;
  book->defs[0x00c33ed3]->alen     = 2;
  book->defs[0x00c33ed3]->acts     = (Wire*) malloc(2 * sizeof(Wire));
  book->defs[0x00c33ed3]->acts[ 0] = mkwire(0x10000013,0xa0000001);
  book->defs[0x00c33ed3]->acts[ 1] = mkwire(0x10000019,0xa0000002);
  book->defs[0x00c33ed3]->nlen     = 3;
  book->defs[0x00c33ed3]->node     = (Node*) malloc(3 * sizeof(Node));
  book->defs[0x00c33ed3]->node[ 0] = (Node) {0x40000001,0x50000002};
  book->defs[0x00c33ed3]->node[ 1] = (Node) {0x40000000,0x40000002};
  book->defs[0x00c33ed3]->node[ 2] = (Node) {0x50000001,0x50000000};
  // lowO
  book->defs[0x00c33ed9]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00c33ed9]->root     = 0xa0000000;
  book->defs[0x00c33ed9]->alen     = 2;
  book->defs[0x00c33ed9]->acts     = (Wire*) malloc(2 * sizeof(Wire));
  book->defs[0x00c33ed9]->acts[ 0] = mkwire(0x10000019,0xa0000001);
  book->defs[0x00c33ed9]->acts[ 1] = mkwire(0x10000019,0xa0000002);
  book->defs[0x00c33ed9]->nlen     = 3;
  book->defs[0x00c33ed9]->node     = (Node*) malloc(3 * sizeof(Node));
  book->defs[0x00c33ed9]->node[ 0] = (Node) {0x40000001,0x50000002};
  book->defs[0x00c33ed9]->node[ 1] = (Node) {0x40000000,0x40000002};
  book->defs[0x00c33ed9]->node[ 2] = (Node) {0x50000001,0x50000000};
  // nidS
  book->defs[0x00cada1d]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00cada1d]->root     = 0xa0000000;
  book->defs[0x00cada1d]->alen     = 2;
  book->defs[0x00cada1d]->acts     = (Wire*) malloc(2 * sizeof(Wire));
  book->defs[0x00cada1d]->acts[ 0] = mkwire(0x1000001d,0xa0000001);
  book->defs[0x00cada1d]->acts[ 1] = mkwire(0x10032b68,0xa0000002);
  book->defs[0x00cada1d]->nlen     = 3;
  book->defs[0x00cada1d]->node     = (Node*) malloc(3 * sizeof(Node));
  book->defs[0x00cada1d]->node[ 0] = (Node) {0x40000002,0x50000001};
  book->defs[0x00cada1d]->node[ 1] = (Node) {0x50000002,0x50000000};
  book->defs[0x00cada1d]->node[ 2] = (Node) {0x40000000,0x40000001};
  // runI
  book->defs[0x00db9c93]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00db9c93]->root     = 0xa0000000;
  book->defs[0x00db9c93]->alen     = 3;
  book->defs[0x00db9c93]->acts     = (Wire*) malloc(3 * sizeof(Wire));
  book->defs[0x00db9c93]->acts[ 0] = mkwire(0x10036e72,0xa0000001);
  book->defs[0x00db9c93]->acts[ 1] = mkwire(0x10028a67,0xa0000002);
  book->defs[0x00db9c93]->acts[ 2] = mkwire(0x10000013,0xa0000003);
  book->defs[0x00db9c93]->nlen     = 4;
  book->defs[0x00db9c93]->node     = (Node*) malloc(4 * sizeof(Node));
  book->defs[0x00db9c93]->node[ 0] = (Node) {0x40000003,0x50000001};
  book->defs[0x00db9c93]->node[ 1] = (Node) {0x50000002,0x50000000};
  book->defs[0x00db9c93]->node[ 2] = (Node) {0x50000003,0x40000001};
  book->defs[0x00db9c93]->node[ 3] = (Node) {0x40000000,0x40000002};
  // runO
  book->defs[0x00db9c99]           = (Term*) malloc(sizeof(Term));
  book->defs[0x00db9c99]->root     = 0xa0000000;
  book->defs[0x00db9c99]->alen     = 3;
  book->defs[0x00db9c99]->acts     = (Wire*) malloc(3 * sizeof(Wire));
  book->defs[0x00db9c99]->acts[ 0] = mkwire(0x10036e72,0xa0000001);
  book->defs[0x00db9c99]->acts[ 1] = mkwire(0x10028a67,0xa0000002);
  book->defs[0x00db9c99]->acts[ 2] = mkwire(0x10000019,0xa0000003);
  book->defs[0x00db9c99]->nlen     = 4;
  book->defs[0x00db9c99]->node     = (Node*) malloc(4 * sizeof(Node));
  book->defs[0x00db9c99]->node[ 0] = (Node) {0x40000003,0x50000001};
  book->defs[0x00db9c99]->node[ 1] = (Node) {0x50000002,0x50000000};
  book->defs[0x00db9c99]->node[ 2] = (Node) {0x50000003,0x40000001};
  book->defs[0x00db9c99]->node[ 3] = (Node) {0x40000000,0x40000002};
}

__host__ void boot(Net* net, u32 ref_id) {
  net->root = mkptr(REF, ref_id);
}

// Main
// ----

int main() {
  // Prints device info
  int device;
  hipDeviceProp_t prop;
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);
  printf("CUDA Device: %s, Compute Capability: %d.%d\n\n", prop.name, prop.major, prop.minor);

  // Allocates net and book on CPU
  Net* cpu_net = mknet();
  Book* cpu_book = mkbook();
  populate(cpu_book);
  boot(cpu_net, 0x00029f04); // initial term
  //boot(cpu_net, 0x009b6ca4); // initial term

  // Prints the input net
  printf("\nINPUT\n=====\n\n");
  print_net(cpu_net);

  // Uploads net and book to GPU
  Net* gpu_net = net_to_gpu(cpu_net);
  Book* gpu_book = book_to_gpu(cpu_book);

  // Marks init time
  struct timespec start, end;
  clock_gettime(CLOCK_MONOTONIC_RAW, &start);

  // Normalizes
  do_global_expand(gpu_net, gpu_book);
  for (u32 tick = 0; tick < 128; ++tick) {
    do_global_rewrite(gpu_net, gpu_book, 16, tick, (tick / GROUP_LOG2) % 2);
  }
  do_global_expand(gpu_net, gpu_book);
  do_global_rewrite(gpu_net, gpu_book, 200000, 0, 0);
  hipDeviceSynchronize();

  // Gets end time
  clock_gettime(CLOCK_MONOTONIC_RAW, &end);
  uint32_t delta_time = (end.tv_sec - start.tv_sec) * 1000 + (end.tv_nsec - start.tv_nsec) / 1000000;

  // Reads result back to cpu
  Net* norm = net_to_cpu(gpu_net);

  // Prints the output
  //print_tree(norm, norm->root);
  printf("\nNORMAL ~ rewrites=%llu\n======\n\n", norm->rwts);
  print_net(norm);
  printf("Time: %.3f s\n", ((double)delta_time) / 1000.0);
  printf("RPS : %.3f million\n", ((double)norm->rwts) / ((double)delta_time) / 1000.0);

  // Clears CPU memory
  net_free_on_gpu(gpu_net);
  book_free_on_gpu(gpu_book);

  // Clears GPU memory
  net_free_on_cpu(cpu_net);
  book_free_on_cpu(cpu_book);
  net_free_on_cpu(norm);

  return 0;
}
