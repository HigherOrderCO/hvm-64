#include "hip/hip_runtime.h"
// NOTE: This file is adapted from hvm2.cu and must be kept in sync (for now)
// The only changes are that only kernel functions and related types are kept,
// host functions are removed, they are moved to the Rust host code.

//// Prelude

typedef unsigned char      uint8_t;
typedef unsigned short     uint16_t;
typedef unsigned int       uint32_t;
typedef unsigned long long uint64_t;

//// Runtime

typedef uint8_t u8;
typedef uint16_t u16;
typedef uint32_t u32;
typedef uint64_t u64;

// Configuration
// -------------

// This code is initially optimized for RTX 4090

// Bags dimensions (128x128 redex bags)
const u32 BAGS_WIDTH_L2  = 7;
const u32 BAGS_WIDTH     = 1 << BAGS_WIDTH_L2;
const u32 BAGS_HEIGHT_L2 = 7;
const u32 BAGS_HEIGHT    = 1 << BAGS_HEIGHT_L2;
const u32 BAGS_TOTAL_L2  = BAGS_WIDTH_L2 + BAGS_HEIGHT_L2;
const u32 BAGS_TOTAL     = 1 << BAGS_TOTAL_L2;

// Threads per Squad (4)
const u32 SQUAD_SIZE_L2 = 2;
const u32 SQUAD_SIZE    = 1 << SQUAD_SIZE_L2;

// Squads per Block (128)
const u32 GROUP_SIZE_L2 = BAGS_WIDTH_L2;
const u32 GROUP_SIZE    = 1 << GROUP_SIZE_L2;

// Threads per Block (512)
const u32 BLOCK_SIZE_L2 = GROUP_SIZE_L2 + SQUAD_SIZE_L2;
const u32 BLOCK_SIZE    = 1 << BLOCK_SIZE_L2;

// Heap Size (max total nodes = 256m = 2GB)
const u32 HEAP_SIZE_L2 = 28;
const u32 HEAP_SIZE    = 1 << HEAP_SIZE_L2;

// Jump Table (max book entries = 16m definitions)
const u32 JUMP_SIZE_L2 = 24;
const u32 JUMP_SIZE    = 1 << JUMP_SIZE_L2;

// Max Redexes per Interaction
const u32 MAX_NEW_REDEX = 16; // FIXME: use to check full rbags

// Local Attributes per Squad
const u32 SMEM_SIZE = 4; // local attributes

// Total Number of Squads
const u32 SQUAD_TOTAL_L2 = BAGS_TOTAL_L2;
const u32 SQUAD_TOTAL    = 1 << SQUAD_TOTAL_L2;

// Total Allocation Nodes per Squad
const u32 AREA_SIZE = HEAP_SIZE / SQUAD_TOTAL;

// Redexes per Redex Bag
const u32 RBAG_SIZE = 256;

// Total Redexes on All Bags
const u32 BAGS_SIZE = BAGS_TOTAL * RBAG_SIZE;

// Max Global Expansion Ptrs (1 per squad)
const u32 HEAD_SIZE_L2 = SQUAD_TOTAL_L2;
const u32 HEAD_SIZE    = 1 << HEAD_SIZE_L2;

// Max Local Expansion Ptrs per Squad
const u32 EXPANSIONS_PER_SQUAD = 16;

// Types
// -----

typedef u8  Tag; // pointer tag: 4-bit
typedef u32 Val; // pointer val: 28-bit

// Core terms
const Tag VR1 = 0x0; // variable to aux port 1
const Tag VR2 = 0x1; // variable to aux port 2
const Tag RD1 = 0x2; // redirect to aux port 1
const Tag RD2 = 0x3; // redirect to aux port 2
const Tag REF = 0x4; // lazy closed net
const Tag ERA = 0x5; // unboxed eraser
const Tag NUM = 0x6; // unboxed number
const Tag OP2 = 0x7; // numeric operation binary
const Tag OP1 = 0x8; // numeric operation unary
const Tag ITE = 0x9; // numeric if-then-else
const Tag CT0 = 0xA; // main port of con node 0
const Tag CT1 = 0xB; // main port of con node 1
const Tag CT2 = 0xC; // main port of con node 2
const Tag CT3 = 0xD; // main port of con node 3
const Tag CT4 = 0xE; // main port of con node 4
const Tag CT5 = 0xF; // main port of con node 5

// Special values
const u32 ROOT = 0x0 | VR2;  // pointer to root port
const u32 NONE = 0x00000000; // empty value, not allocated
const u32 GONE = 0xFFFFFFFE; // node has been moved to redex bag by paired thread
const u32 LOCK = 0xFFFFFFFF; // value taken by another thread, will be replaced soon
const u32 FAIL = 0xFFFFFFFF; // signals failure to allocate

// Unit types
const u32 A1 = 0; // focuses on the A node, P1 port
const u32 A2 = 1; // focuses on the A node, P2 port
const u32 B1 = 2; // focuses on the B node, P1 port
const u32 B2 = 3; // focuses on the B node, P2 port

// Ports (P1 or P2)
typedef u8 Port;
const u32 P1 = 0;
const u32 P2 = 1;

// Pointers = 4-bit tag + 28-bit val
typedef u32 Ptr;

// Nodes are pairs of pointers
typedef struct {
  Ptr ports[2];
} Node;

// Wires are pairs of pointers
typedef u64 Wire;

// An interaction net
typedef struct {
  Wire* bags; // redex bags (active pairs)
  Node* heap; // memory buffer with all nodes
  Wire* head; // head expansion buffer
  u32*  jump; // book jump table
  u64   rwts; // number of rewrites performed
} Net;

// A unit local data
typedef struct {
  u32   tid;  // thread id (local)
  u32   gid;  // global id (global)
  u32   sid;  // squad id (local)
  u32   uid;  // squad id (global)
  u32   qid;  // quarter id (A1|A2|B1|B2)
  u32   port; // unit port (P1|P2)
  u64   rwts; // local rewrites performed
  u32   mask; // squad warp mask
  u32*  aloc; // where to alloc next node
  u32*  sm32; // shared 32-bit buffer
  u64*  sm64; // shared 64-bit buffer
  u64*  RBAG; // init of my redex bag
  u32*  rlen; // local redex bag length
  Wire* rbag; // local redex bag
} Unit;

// TermBook
// --------

__constant__ u32* BOOK;

typedef u32 Book; // stored in a flat buffer
/*
Book* init_book_on_gpu(u32* data, u32 size) {
  u32* gpu_book;
  hipMalloc(&gpu_book, size * sizeof(u32));
  hipMemcpy(gpu_book, data, size * sizeof(u32), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(BOOK), &gpu_book, sizeof(u32*));
  return gpu_book;
}

void book_free_on_gpu(Book* gpu_book) {
  hipFree(gpu_book);
}
 */
// Runtime
// -------

// Integer ceil division
__host__ __device__ inline u32 div(u32 a, u32 b) {
  return (a + b - 1) / b;
}

// Creates a new pointer
__host__ __device__ inline Ptr mkptr(Tag tag, Val val) {
  return (val << 4) | ((Val)tag);
}

// Gets the tag of a pointer
__host__ __device__ inline Tag tag(Ptr ptr) {
  return (Tag)(ptr & 0xF);
}

// Gets the value of a pointer
__host__ __device__ inline Val val(Ptr ptr) {
  return (Val)(ptr >> 4);
}

// Is this pointer a variable?
__host__ __device__ inline bool is_var(Ptr ptr) {
  return ptr != 0 && tag(ptr) >= VR1 && tag(ptr) <= VR2;
}

// Is this pointer a redirection?
__host__ __device__ inline bool is_red(Ptr ptr) {
  return tag(ptr) >= RD1 && tag(ptr) <= RD2;
}

// Is this pointer a constructor?
__host__ __device__ inline bool is_ctr(Ptr ptr) {
  return tag(ptr) >= CT0 && tag(ptr) < CT5; // FIXME: CT5 excluded
}

// Is this pointer an eraser?
__host__ __device__ inline bool is_era(Ptr ptr) {
  return tag(ptr) == ERA;
}

// Is this pointer a reference?
__host__ __device__ inline bool is_ref(Ptr ptr) {
  return tag(ptr) == REF;
}

// Is this pointer a main port?
__host__ __device__ inline bool is_pri(Ptr ptr) {
  return is_ctr(ptr) || is_era(ptr) || is_ref(ptr);
}

// Is this pointer carrying a location (that needs adjustment)?
__host__ __device__ inline bool has_loc(Ptr ptr) {
  return is_ctr(ptr) || is_var(ptr);
}

// Gets the target ref of a var or redirection pointer
__host__ __device__ inline Ptr* target(Net* net, Ptr ptr) {
  return &net->heap[val(ptr)].ports[ptr & 1];
}

// Traverses to the other side of a wire
__host__ __device__ inline Ptr enter(Net* net, Ptr ptr) {
  Ptr* ref = target(net, ptr);
  while (is_red(*ref)) {
    ptr = *ref;
    ref = target(net, ptr);
  }
  return ptr;
}

// Transforms a variable into a redirection
__host__ __device__ inline Ptr redir(Ptr ptr) {
  return mkptr(tag(ptr) + (is_var(ptr) ? 2 : 0), val(ptr));
}

// Transforms a redirection into a variable
__host__ __device__ inline Ptr undir(Ptr ptr) {
  return mkptr(tag(ptr) - (is_red(ptr) ? 2 : 0), val(ptr));
}

// Creates a new wire
__host__ __device__ inline Wire mkwire(Ptr p1, Ptr p2) {
  return (((u64)p1) << 32) | ((u64)p2);
}

// Gets the left element of a wire
__host__ __device__ inline Ptr wire_lft(Wire wire) {
  return wire >> 32;
}

// Gets the right element of a wire
__host__ __device__ inline Ptr wire_rgt(Wire wire) {
  return wire & 0xFFFFFFFF;
}

// Creates a new node
__host__ __device__ inline Node mknode(Ptr p1, Ptr p2) {
  Node node;
  node.ports[P1] = p1;
  node.ports[P2] = p2;
  return node;
}

// Creates a nil node
__host__ __device__ inline Node Node_nil() {
  return mknode(NONE, NONE);
}

// Checks if a node is nil
__host__ __device__ inline bool Node_is_nil(Node* node) {
  return node->ports[P1] == NONE && node->ports[P2] == NONE;
}

// Gets a reference to the index/port Ptr on the net
__device__ inline Ptr* at(Net* net, Val idx, Port port) {
  return &net->heap[idx].ports[port];
}

// Allocates one node in memory
__device__ u32 alloc(Unit *unit, Net *net, u32 size) {
  u32 size4 = div(size, (u32)4) * 4;
  u32 begin = unit->uid * AREA_SIZE;
  u32 space = 0;
  u32 index = *unit->aloc - (*unit->aloc % 4);
  for (u32 i = 0; i < 256; ++i) {
    Node node = net->heap[begin + index + unit->qid];
    bool null = Node_is_nil(&node);
    bool succ = __all_sync(unit->mask, null);
    index = (index + 4) % AREA_SIZE;
    space = succ && index > 0 ? space + 4 : 0;
    if (space == size4) {
      *unit->aloc = index;
      return (begin + index - space) % HEAP_SIZE;
    }
  }
  return FAIL;
}

// Gets the value of a ref; waits if taken.
__device__ inline Ptr take(Ptr* ref) {
  Ptr got = atomicExch((u32*)ref, LOCK);
  while (got == LOCK) {
    got = atomicExch((u32*)ref, LOCK);
  }
  return got;
}

// Attempts to replace 'exp' by 'neo', until it succeeds
__device__ inline bool replace(Ptr* ref, Ptr exp, Ptr neo) {
  Ptr got = atomicCAS((u32*)ref, exp, neo);
  while (got != exp) {
    got = atomicCAS((u32*)ref, exp, neo);
  }
  return true;
}

// Splits elements of two arrays evenly between each-other
// FIXME: it is desirable to split when size=1, to rotate out of starving squads
__device__ __noinline__ void split(u32 tid, u64* a_len, u64* a_arr, u64* b_len, u64* b_arr, u64 max_len) {
  __syncthreads();
  u64* A_len = *a_len < *b_len ? a_len : b_len;
  u64* B_len = *a_len < *b_len ? b_len : a_len;
  u64* A_arr = *a_len < *b_len ? a_arr : b_arr;
  u64* B_arr = *a_len < *b_len ? b_arr : a_arr;
  bool move  = *A_len + 1 < *B_len;
  u64  min   = *A_len;
  u64  max   = *B_len;
  __syncthreads();
  for (u64 t = 0; t < max_len / (SQUAD_SIZE * 2); ++t) {
    u64 i = min + t * (SQUAD_SIZE * 2) + tid;
    u64 value;
    if (move && i < max) {
      value = B_arr[i];
      B_arr[i] = 0;
    }
    __syncthreads();
    if (move && i < max) {
      if ((i - min) % 2 == 0) {
        A_arr[min + (t * (SQUAD_SIZE * 2) + tid) / 2] = value;
      } else {
        B_arr[min + (t * (SQUAD_SIZE * 2) + tid) / 2] = value;
      }
    }
  }
  __syncthreads();
  u64 old_A_len = *A_len;
  u64 old_B_len = *B_len;
  if (move && tid == 0) {
    u64 new_A_len = (*A_len + *B_len) / 2 + (*A_len + *B_len) % 2;
    u64 new_B_len = (*A_len + *B_len) / 2;
    *A_len = new_A_len;
    *B_len = new_B_len;
  }
  __syncthreads();
}

// Pops a redex
__device__ Wire pop_redex(Unit* unit) {
  Wire redex = mkwire(0, 0);

  u32 rlen = *unit->rlen;
  if (rlen > 0 && rlen <= RBAG_SIZE - MAX_NEW_REDEX) {
    redex = unit->rbag[rlen-1];
  }
  __syncwarp(unit->mask);
  if (rlen > 0 && rlen <= RBAG_SIZE - MAX_NEW_REDEX) {
    unit->rbag[rlen-1] = mkwire(0, 0);
    *unit->rlen = rlen-1;
  }
  __syncwarp(unit->mask);

  if (unit->qid <= A2) {
    return mkwire(wire_lft(redex), wire_rgt(redex));
  } else {
    return mkwire(wire_rgt(redex), wire_lft(redex));
  }
}

// Puts a redex
__device__ void put_redex(Unit* unit, Ptr a_ptr, Ptr b_ptr) {
  // optimization: avoids pushing non-reactive redexes
  bool a_era = is_era(a_ptr);
  bool b_era = is_era(b_ptr);
  bool a_ref = is_ref(a_ptr);
  bool b_ref = is_ref(b_ptr);
  if ( a_era && b_era
    || a_ref && b_era
    || a_era && b_ref
    || a_ref && b_ref) {
    unit->rwts += 1;
    return;
  }

  // pushes redex to end of bag
  u32 index = atomicAdd(unit->rlen, 1);
  if (index < RBAG_SIZE - 1) {
    unit->rbag[index] = mkwire(a_ptr, b_ptr);
  } else {
    printf("ERROR: PUSHED TO FULL TBAG (NOT IMPLEMENTED YET)\n");
  }
}

// Adjusts a dereferenced pointer
__device__ inline Ptr adjust(Unit* unit, Ptr ptr, u32 delta) {
  return mkptr(tag(ptr), has_loc(ptr) ? val(ptr) + delta - 1 : val(ptr));
}

// Expands a reference
__device__ bool deref(Unit* unit, Net* net, Book* book, Ptr* ref, Ptr up) {
  // Assert ref is either a REF or NULL
  ref = ref != NULL && is_ref(*ref) ? ref : NULL;

  // Load definition
  const u32  jump = ref != NULL ? net->jump[val(*ref) & 0xFFFFFF] : 0;
  const u32  nlen = book[jump + 0];
  const u32  rlen = book[jump + 1];
  const u32* node = &book[jump + 2];
  const u32* acts = &book[jump + 2 + nlen * 2];

  // Allocate needed space
  u32 loc = FAIL;
  if (ref != NULL) {
    loc = alloc(unit, net, nlen - 1);
  }

  if (ref != NULL && loc != FAIL) {
    // Increment rewrite count.
    if (unit->qid == A1) {
      unit->rwts += 1;
    }

    // Load nodes, adjusted.
    for (u32 i = 0; i < div(nlen - 1, SQUAD_SIZE); ++i) {
      u32 idx = i * SQUAD_SIZE + unit->qid;
      if (idx < nlen - 1) {
        Ptr p1 = adjust(unit, node[2+idx*2+0], loc);
        Ptr p2 = adjust(unit, node[2+idx*2+1], loc);
        *at(net, loc + idx, P1) = p1;
        *at(net, loc + idx, P2) = p2;
      }
    }

    // Load redexes, adjusted.
    for (u32 i = 0; i < div(rlen, SQUAD_SIZE); ++i) {
      u32 idx = i * SQUAD_SIZE + unit->qid;
      if (idx < rlen) {
        Ptr p1 = adjust(unit, acts[idx*2+0], loc);
        Ptr p2 = adjust(unit, acts[idx*2+1], loc);
        put_redex(unit, p1, p2);
      }
    }

    // Load root, adjusted.
    *ref = adjust(unit, node[1], loc);

    // Link root.
    if (unit->qid == A1 && is_var(*ref)) {
      *target(net, *ref) = up;
    }
  }

  return ref == NULL || loc != FAIL;
}

// Rewrite
// -------

__device__ u32 interleave(u32 idx, u32 width, u32 height) {
  u32 old_row = idx / width;
  u32 old_col = idx % width;
  u32 new_row = old_col % height;
  u32 new_col = old_col / height + old_row * (width / height);
  return new_row * width + new_col;
}

// Local Squad Id (sid) to Global Squad Id (uid)
__device__ u32 sid_to_uid(u32 sid, bool flip) {
  return flip ? interleave(sid, BAGS_WIDTH, BAGS_HEIGHT) : sid;
}

__device__ Unit init_unit(Net* net, bool flip) {
  __shared__ u32 SMEM[GROUP_SIZE * SMEM_SIZE];
  __shared__ u32 ALOC[GROUP_SIZE];

  for (u32 i = 0; i < GROUP_SIZE * SMEM_SIZE / BLOCK_SIZE; ++i) {
    SMEM[i * BLOCK_SIZE + threadIdx.x] = 0;
  }
  __syncthreads();

  for (u32 i = 0; i < GROUP_SIZE / BLOCK_SIZE; ++i) {
    ALOC[i * BLOCK_SIZE + threadIdx.x] = 0;
  }
  __syncthreads();

  Unit unit;
  unit.tid  = threadIdx.x;
  unit.gid  = blockIdx.x * blockDim.x + unit.tid;
  unit.sid  = unit.gid / SQUAD_SIZE;
  unit.uid  = sid_to_uid(unit.sid, flip);
  unit.qid  = unit.tid % 4;
  unit.rwts = 0;
  unit.mask = ((1 << SQUAD_SIZE) - 1) << (unit.tid % 32 / SQUAD_SIZE * SQUAD_SIZE);
  unit.port = unit.tid % 2;
  unit.aloc = (u32*)(ALOC + unit.tid / SQUAD_SIZE); // locally cached
  unit.sm32 = (u32*)(SMEM + unit.tid / SQUAD_SIZE * SMEM_SIZE);
  unit.sm64 = (u64*)(SMEM + unit.tid / SQUAD_SIZE * SMEM_SIZE);
  unit.RBAG = net->bags + unit.uid * RBAG_SIZE;
  unit.rlen = (u32*)(unit.RBAG + 0); // TODO: cache locally
  unit.rbag = unit.RBAG + 1;
  *unit.aloc = 0; // TODO: randomize or persist

  return unit;
}

__device__ void save_unit(Unit* unit, Net* net) {
  if (unit->rwts > 0) {
    atomicAdd(&net->rwts, unit->rwts);
  }
}

__device__ void share_redexes(Unit* unit, Net* net, Book* book, u32 tick, bool flip) {
  u32  side  = ((unit->tid / SQUAD_SIZE) >> (BAGS_WIDTH_L2 - 1 - (tick % BAGS_WIDTH_L2))) & 1;
  u32  shift = (1 << (BAGS_WIDTH_L2 - 1)) >> (tick % BAGS_WIDTH_L2);
  u32  a_sid = unit->sid;
  u32  b_sid = side ? a_sid - shift : a_sid + shift;
  u32  a_uid = sid_to_uid(a_sid, flip);
  u32  b_uid = sid_to_uid(b_sid, flip);
  u64* a_len = net->bags + a_uid * RBAG_SIZE;
  u64* b_len = net->bags + b_uid * RBAG_SIZE;
  u32  sp_id = unit->tid % SQUAD_SIZE + side * SQUAD_SIZE;
  split(sp_id, a_len, a_len+1, b_len, b_len+1, RBAG_SIZE);
}

__device__ void atomic_join(Unit* unit, Net* net, Book* book, Ptr a_ptr, Ptr* a_ref, Ptr b_ptr) {
  while (true) {
    Ptr* ste_ref = target(net, b_ptr);
    Ptr  ste_ptr = *ste_ref;
    if (is_var(ste_ptr)) {
      Ptr* trg_ref = target(net, ste_ptr);
      Ptr  trg_ptr = atomicAdd(trg_ref, 0);
      if (is_red(trg_ptr)) {
        Ptr neo_ptr = undir(trg_ptr);
        Ptr updated = atomicCAS(ste_ref, ste_ptr, neo_ptr);
        if (updated == ste_ptr) {
          *trg_ref = 0;
          continue;
        }
      }
    }
    break;
  }
}

__device__ void atomic_link(Unit* unit, Net* net, Book* book, Ptr a_ptr, Ptr* a_ref, Ptr b_ptr) {
  while (true) {
    // Peek the target, which may not be owned by us.
    Ptr* t_ref = target(net, a_ptr);
    Ptr  t_ptr = atomicAdd(t_ref, 0);

    // If target is a redirection, clear and move forward.
    if (is_red(t_ptr)) {
      // We own the redirection, so we can mutate it.
      *t_ref = 0;
      a_ptr = t_ptr;
      continue;
    }

    // If target is a variable, try replacing it by the node.
    else if (is_var(t_ptr)) {
      // We don't own the var, so we must try replacing with a CAS.
      if (atomicCAS(t_ref, t_ptr, b_ptr) == t_ptr) {
        // Clear source location.
        *a_ref = 0;
        // Collect the orphaned backward path.
        t_ref = target(net, t_ptr);
        t_ptr = *t_ref;
        while (is_red(t_ptr)) {
          *t_ref = 0;
          t_ref = target(net, t_ptr);
          t_ptr = *t_ref;
        }
        return;
      }

      // If the CAS failed, the var changed, so we try again.
      continue;
    }

    // If it is a node, two threads will reach this branch.
    else if (is_pri(t_ptr) || is_ref(t_ptr) || t_ptr == GONE) {
      // Sort references, to avoid deadlocks.
      Ptr *x_ref = a_ref < t_ref ? a_ref : t_ref;
      Ptr *y_ref = a_ref < t_ref ? t_ref : a_ref;

      // Swap first reference by GONE placeholder.
      Ptr x_ptr = atomicExch(x_ref, GONE);

      // First to arrive creates a redex.
      if (x_ptr != GONE) {
        Ptr y_ptr = atomicExch(y_ref, GONE);
        put_redex(unit, x_ptr, y_ptr);
        return;

      // Second to arrive clears up the memory.
      } else {
        *x_ref = 0;
        replace(y_ref, GONE, 0);
        return;
      }
    }

    // If it is taken, we wait.
    else if (t_ptr == LOCK) {
      continue;
    }

    // Shouldn't be reached.
    else {
      return;
    }
  }
}

__device__ void atomic_subst(Unit* unit, Net* net, Book* book, Ptr a_ptr, Ptr a_dir, Ptr b_ptr, bool put) {
  Ptr* a_ref = target(net, a_dir);
  if (is_var(a_ptr)) {
    Ptr got = atomicCAS(target(net, a_ptr), a_dir, b_ptr);
    if (got == a_dir) {
      atomicExch(a_ref, NONE);
    } else if (is_var(b_ptr)) {
      atomicExch(a_ref, redir(b_ptr));
      atomic_join(unit, net, book, a_ptr, a_ref, redir(b_ptr));
    } else if (is_pri(b_ptr)) {
      atomicExch(a_ref, b_ptr);
      atomic_link(unit, net, book, a_ptr, a_ref, b_ptr);
    }
  } else if (is_pri(a_ptr) && is_pri(b_ptr)) {
    if (a_ptr < b_ptr || put) {
      put_redex(unit, b_ptr, a_ptr); // FIXME: swapping bloats rbag; why?
    }
    atomicExch(a_ref, NONE);
  } else {
    atomicExch(a_ref, NONE);
  }
}

__device__ void interact(Unit* unit, Net* net, Book* book) {
  // Pops a redex from local bag
  Wire redex = pop_redex(unit);
  Ptr  a_ptr = wire_lft(redex);
  Ptr  b_ptr = wire_rgt(redex);

  // Flag to abort in case of failure
  bool abort = false;

  // Dereferences
  Ptr* deref_ptr = NULL;
  if (is_ref(a_ptr) && is_ctr(b_ptr)) {
    deref_ptr = &a_ptr;
  }
  if (is_ref(b_ptr) && is_ctr(a_ptr)) {
    deref_ptr = &b_ptr;
  }
  if (!deref(unit, net, book, deref_ptr, NONE)) {
    abort = true;
  }

  // Defines type of interaction
  bool rewrite = a_ptr != 0 && b_ptr != 0;
  bool var_pri = rewrite && is_var(a_ptr) && is_pri(b_ptr) && unit->port == P1;
  bool era_ctr = rewrite && is_era(a_ptr) && is_ctr(b_ptr);
  bool ctr_era = rewrite && is_ctr(a_ptr) && is_era(b_ptr);
  bool con_con = rewrite && is_ctr(a_ptr) && is_ctr(b_ptr) && tag(a_ptr) == tag(b_ptr);
  bool con_dup = rewrite && is_ctr(a_ptr) && is_ctr(b_ptr) && tag(a_ptr) != tag(b_ptr);

  // Local rewrite variables
  Ptr  ak_dir; // dir to our aux port
  Ptr  bk_dir; // dir to other aux port
  Ptr *ak_ref; // ref to our aux port
  Ptr *bk_ref; // ref to other aux port
  Ptr  ak_ptr; // val of our aux port
  Ptr  bk_ptr; // val to other aux port
  Ptr  mv_ptr; // val of ptr to send to other side
  u32  dp_loc; // duplication allocation index

  // If con_dup, alloc clones base index
  if (rewrite && con_dup) {
    dp_loc = alloc(unit, net, 4);
  }

  // Aborts if allocation failed
  if (rewrite && con_dup && dp_loc == FAIL) {
    abort = true;
  }

  // Reverts when abort=true
  if (rewrite && abort) {
    rewrite = false;
    put_redex(unit, a_ptr, b_ptr);
  }
  __syncwarp(unit->mask);

  // Inc rewrite count
  if (rewrite && unit->qid == A1) {
    unit->rwts += 1;
  }

  // Gets port here
  if (rewrite && (ctr_era || con_con || con_dup)) {
    ak_dir = mkptr(VR1 + unit->port, val(a_ptr));
    ak_ref = target(net, ak_dir);
    ak_ptr = take(ak_ref);
  }

  // Gets port there
  if (rewrite && (era_ctr || con_con || con_dup)) {
    bk_dir = mkptr(VR1 + unit->port, val(b_ptr));
    bk_ref = target(net, bk_dir);
  }

  // If era_ctr, send an erasure
  if (rewrite && era_ctr) {
    mv_ptr = mkptr(ERA, 0);
  }

  // If con_con, send a redirection
  if (rewrite && con_con) {
    mv_ptr = ak_ptr;
  }

  // If con_dup, create inner wires between clones
  if (rewrite && con_dup) {
    u32 cx_loc = dp_loc + unit->qid;
    u32 c1_loc = dp_loc + (unit->qid <= A2 ? 2 : 0);
    u32 c2_loc = dp_loc + (unit->qid <= A2 ? 3 : 1);
    atomicExch(target(net, mkptr(VR1, cx_loc)), mkptr(unit->port == P1 ? VR1 : VR2, c1_loc));
    atomicExch(target(net, mkptr(VR2, cx_loc)), mkptr(unit->port == P1 ? VR1 : VR2, c2_loc));
    mv_ptr = mkptr(tag(a_ptr), cx_loc);
  }
  __syncwarp(unit->mask);

  // Send ptr to other side
  if (rewrite && (era_ctr || con_con || con_dup)) {
    unit->sm32[unit->qid + (unit->qid <= A2 ? 2 : -2)] = mv_ptr;
  }
  __syncwarp(unit->mask);

  // Receive ptr from other side
  if (rewrite && (con_con || ctr_era || con_dup)) {
    bk_ptr = unit->sm32[unit->qid];
  }
  __syncwarp(unit->mask);

  // If var_pri, the var must be a deref root, so we just subst
  if (rewrite && var_pri && unit->port == P1) {
    atomicExch(target(net, a_ptr), b_ptr);
  }
  __syncwarp(unit->mask);

  // Substitutes
  if (rewrite && (con_con || ctr_era || con_dup)) {
    atomic_subst(unit, net, book, ak_ptr, ak_dir, bk_ptr, ctr_era || con_dup);
  }
  __syncwarp(unit->mask);
}

// An active wire is reduced by 4 parallel threads, each one performing "1/4" of
// the work. Each thread will be pointing to a node of the active pair, and an
// aux port of that node. So, when nodes A-B interact, we have 4 thread quads:
// - Thread A1: points to node A and its aux1
// - Thread A2: points to node A and its aux2
// - Thread B1: points to node B and its aux1
// - Thread B2: points to node B and its aux2
// This is organized so that local threads can perform the same instructions
// whenever possible. So, for example, in a commutation rule, all the 4 clones
// would be allocated at the same time.
// __launch_bounds__(BLOCK_SIZE, 1)
extern "C" __global__ void global_rewrite(Net* net, Book* book, u32 repeat, u32 tick, bool flip) {
  // Initializes local vars
  Unit unit = init_unit(net, flip);

  // Performs interactions
  for (u32 turn = 0; turn < repeat; ++turn) {
    interact(&unit, net, book);
  }

  // Shares redexes with paired neighbor
  share_redexes(&unit, net, book, tick, flip);

  // When the work ends, sum stats
  save_unit(&unit, net);
}
/*
void do_global_rewrite(Net* net, Book* book, u32 repeat, u32 tick, bool flip) {
  global_rewrite<<<BAGS_HEIGHT, BLOCK_SIZE>>>(net, book, repeat, tick, flip);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(err));
  }
}
 */
// Expand
// ------

// Collects local expansion heads recursively
__device__ void expand(Unit* unit, Net* net, Book* book, Ptr dir, u32* len, u32* lhds) {
  Ptr ptr = *target(net, dir);
  if (is_ctr(ptr)) {
    expand(unit, net, book, mkptr(VR1, val(ptr)), len, lhds);
    expand(unit, net, book, mkptr(VR2, val(ptr)), len, lhds);
  } else if (is_red(ptr)) {
    expand(unit, net, book, ptr, len, lhds);
  } else if (is_ref(ptr) && *len < EXPANSIONS_PER_SQUAD) {
    lhds[(*len)++] = dir;
  }
}

// Takes an initial head location for each squad
extern "C" __global__ void global_expand_prepare(Net* net) {
  u32 uid = blockIdx.x * blockDim.x + threadIdx.x;

  // Traverses down
  u32 key = uid;
  Ptr dir = ROOT;
  Ptr ptr, *ref;
  for (u32 depth = 0; depth < BAGS_TOTAL_L2; ++depth) {
    dir = enter(net, dir);
    ref = target(net, dir);
    if (is_var(dir)) {
      ptr = *ref;
      if (is_ctr(ptr)) {
        dir = mkptr(key & 1 ? VR1 : VR2, val(ptr));
        key = key >> 1;
      }
    }
  }

  // Takes ptr
  dir = enter(net, dir);
  ref = target(net, dir);
  if (is_var(dir)) {
    ptr = atomicExch(ref, LOCK);
  }

  // Stores ptr
  if (ptr != LOCK) {
    net->head[uid] = mkwire(dir, ptr);
  } else {
    net->head[uid] = mkwire(NONE, NONE);
  }

}

// Performs global expansion of heads
extern "C" __global__ void global_expand(Net* net, Book* book) {
  __shared__ u32 HEAD[GROUP_SIZE * EXPANSIONS_PER_SQUAD];

  for (u32 i = 0; i < GROUP_SIZE * EXPANSIONS_PER_SQUAD / BLOCK_SIZE; ++i) {
    HEAD[i * BLOCK_SIZE + threadIdx.x] = 0;
  }
  __syncthreads();

  Unit unit = init_unit(net, 0);

  u32* head = HEAD + unit.tid / SQUAD_SIZE * EXPANSIONS_PER_SQUAD;

  Wire got = net->head[unit.uid];
  Ptr  dir = wire_lft(got);
  Ptr* ref = target(net, dir);
  Ptr  ptr = wire_rgt(got);

  if (unit.qid == A1 && ptr != NONE) {
    *ref = ptr;
  }
  __syncthreads();

  u32 len = 0;
  if (unit.qid == A1 && ptr != NONE) {
    expand(&unit, net, book, dir, &len, head);
  }
  __syncthreads();

  for (u32 i = 0; i < EXPANSIONS_PER_SQUAD; ++i) {
    Ptr  dir = head[i];
    Ptr* ref = target(net, dir);
    if (!deref(&unit, net, book, ref, dir)) {
      printf("ERROR: DEREF FAILED ON EXPAND (NOT IMPLEMENTED YET)\n");
    }
  }
  __syncthreads();

  save_unit(&unit, net);
}
/*
// Performs a global head expansion (1 deref per bag)
void do_global_expand(Net* net, Book* book) {
  global_expand_prepare<<<BAGS_HEIGHT, GROUP_SIZE>>>(net);
  global_expand<<<BAGS_HEIGHT, BLOCK_SIZE>>>(net, book);
}

// Host<->Device
// -------------

__host__ Net* mknet(u32 root_fn, u32* jump_data, u32 jump_data_size) {
  Net* net  = (Net*)malloc(sizeof(Net));
  net->rwts = 0;
  net->bags = (Wire*)malloc(BAGS_SIZE * sizeof(Wire));
  net->heap = (Node*)malloc(HEAP_SIZE * sizeof(Node));
  net->head = (Wire*)malloc(HEAD_SIZE * sizeof(Wire));
  net->jump = (u32*) malloc(JUMP_SIZE * sizeof(u32));
  memset(net->bags, 0, BAGS_SIZE * sizeof(Wire));
  memset(net->heap, 0, HEAP_SIZE * sizeof(Node));
  memset(net->head, 0, HEAD_SIZE * sizeof(Wire));
  memset(net->jump, 0, JUMP_SIZE * sizeof(u32));
  *target(net, ROOT) = mkptr(REF, root_fn);
  for (u32 i = 0; i < jump_data_size / 2; ++i) {
    net->jump[jump_data[i*2+0]] = jump_data[i*2+1];
  }
  return net;
}

__host__ Net* net_to_gpu(Net* host_net) {
  // Allocate memory on the device for the Net object, and its data
  Net*  device_net;
  Wire* device_bags;
  Node* device_heap;
  Wire* device_head;
  u32*  device_jump;

  hipMalloc((void**)&device_net, sizeof(Net));
  hipMalloc((void**)&device_bags, BAGS_SIZE * sizeof(Wire));
  hipMalloc((void**)&device_heap, HEAP_SIZE * sizeof(Node));
  hipMalloc((void**)&device_head, HEAD_SIZE * sizeof(Wire));
  hipMalloc((void**)&device_jump, JUMP_SIZE * sizeof(u32));

  // Copy the host data to the device memory
  hipMemcpy(device_bags, host_net->bags, BAGS_SIZE * sizeof(Wire), hipMemcpyHostToDevice);
  hipMemcpy(device_heap, host_net->heap, HEAP_SIZE * sizeof(Node), hipMemcpyHostToDevice);
  hipMemcpy(device_head, host_net->head, HEAD_SIZE * sizeof(Wire), hipMemcpyHostToDevice);
  hipMemcpy(device_jump, host_net->jump, JUMP_SIZE * sizeof(u32), hipMemcpyHostToDevice);

  // Create a temporary host Net object with device pointers
  Net temp_net  = *host_net;
  temp_net.bags = device_bags;
  temp_net.heap = device_heap;
  temp_net.head = device_head;
  temp_net.jump = device_jump;

  // Copy the temporary host Net object to the device memory
  hipMemcpy(device_net, &temp_net, sizeof(Net), hipMemcpyHostToDevice);

  // Return the device pointer to the created Net object
  return device_net;
}

__host__ Net* net_to_cpu(Net* device_net) {
  // Create a new host Net object
  Net* host_net = (Net*)malloc(sizeof(Net));

  // Copy the device Net object to the host memory
  hipMemcpy(host_net, device_net, sizeof(Net), hipMemcpyDeviceToHost);

  // Allocate host memory for data
  host_net->bags = (Wire*)malloc(BAGS_SIZE * sizeof(Wire));
  host_net->heap = (Node*)malloc(HEAP_SIZE * sizeof(Node));
  host_net->head = (Wire*)malloc(HEAD_SIZE * sizeof(Wire));
  host_net->jump = (u32*) malloc(JUMP_SIZE * sizeof(u32));

  // Retrieve the device pointers for data
  Wire* device_bags;
  Node* device_heap;
  Wire* device_head;
  u32*  device_jump;
  hipMemcpy(&device_bags, &(device_net->bags), sizeof(Wire*), hipMemcpyDeviceToHost);
  hipMemcpy(&device_heap, &(device_net->heap), sizeof(Node*), hipMemcpyDeviceToHost);
  hipMemcpy(&device_head, &(device_net->head), sizeof(Wire*), hipMemcpyDeviceToHost);
  hipMemcpy(&device_jump, &(device_net->jump), sizeof(u32*), hipMemcpyDeviceToHost);

  // Copy the device data to the host memory
  hipMemcpy(host_net->bags, device_bags, BAGS_SIZE * sizeof(Wire), hipMemcpyDeviceToHost);
  hipMemcpy(host_net->heap, device_heap, HEAP_SIZE * sizeof(Node), hipMemcpyDeviceToHost);
  hipMemcpy(host_net->head, device_head, HEAD_SIZE * sizeof(Wire), hipMemcpyDeviceToHost);
  hipMemcpy(host_net->jump, device_jump, JUMP_SIZE * sizeof(u32),  hipMemcpyDeviceToHost);

  return host_net;
}

__host__ void net_free_on_gpu(Net* device_net) {
  // Retrieve the device pointers for data
  Wire* device_bags;
  Node* device_heap;
  Wire* device_head;
  u32*  device_jump;
  hipMemcpy(&device_bags, &(device_net->bags), sizeof(Wire*), hipMemcpyDeviceToHost);
  hipMemcpy(&device_heap, &(device_net->heap), sizeof(Node*), hipMemcpyDeviceToHost);
  hipMemcpy(&device_head, &(device_net->head), sizeof(Wire*), hipMemcpyDeviceToHost);
  hipMemcpy(&device_jump, &(device_net->jump), sizeof(u32*),  hipMemcpyDeviceToHost);

  // Free the device memory
  hipFree(device_bags);
  hipFree(device_heap);
  hipFree(device_head);
  hipFree(device_jump);
  hipFree(device_net);
}

__host__ void net_free_on_cpu(Net* host_net) {
  free(host_net->bags);
  free(host_net->heap);
  free(host_net->head);
  free(host_net->jump);
  free(host_net);
}

// Debugging
// ---------

__host__ const char* show_ptr(Ptr ptr, u32 slot) {
  static char buffer[8][20];
  if (ptr == NONE) {
    strcpy(buffer[slot], "           ");
    return buffer[slot];
  } else if (ptr == LOCK) {
    strcpy(buffer[slot], "[LOCK.....]");
    return buffer[slot];
  } else {
    const char* tag_str = NULL;
    switch (tag(ptr)) {
      case VR1: tag_str = "VR1"; break;
      case VR2: tag_str = "VR2"; break;
      case RD1: tag_str = "RD1"; break;
      case RD2: tag_str = "RD2"; break;
      case REF: tag_str = "REF"; break;
      case ERA: tag_str = "ERA"; break;
      case NUM: tag_str = "NUM"; break;
      case OP2: tag_str = "OP2"; break;
      case OP1: tag_str = "OP1"; break;
      case ITE: tag_str = "ITE"; break;
      case CT0: tag_str = "CT0"; break;
      case CT1: tag_str = "CT1"; break;
      case CT2: tag_str = "CT2"; break;
      case CT3: tag_str = "CT3"; break;
      case CT4: tag_str = "CT4"; break;
      case CT5: tag_str = "CT5"; break;
      default : tag_str = "???"; break;
    }
    snprintf(buffer[slot], sizeof(buffer[slot]), "%s:%07X", tag_str, val(ptr));
    return buffer[slot];
  }
}

// Prints a net in hexadecimal, limited to a given size
void print_net(Net* net) {
  printf("Bags:\n");
  for (u32 i = 0; i < BAGS_SIZE; ++i) {
    if (i % RBAG_SIZE == 0 && net->bags[i] > 0) {
      printf("- [%07X] LEN=%llu\n", i, net->bags[i]);
    } else if (i % RBAG_SIZE >= 1) {
      //Ptr a = wire_lft(net->bags[i]);
      //Ptr b = wire_rgt(net->bags[i]);
      //if (a != 0 || b != 0) {
        //printf("- [%07X] %s %s\n", i, show_ptr(a,0), show_ptr(b,1));
      //}
    }
  }
  //printf("Heap:\n");
  //for (u32 i = 0; i < HEAP_SIZE; ++i) {
    //Ptr a = net->heap[i].ports[P1];
    //Ptr b = net->heap[i].ports[P2];
    //if (a != 0 || b != 0) {
      //printf("- [%07X] %s %s\n", i, show_ptr(a,0), show_ptr(b,1));
    //}
  //}
  printf("Rwts: %llu\n", net->rwts);
}

// Struct to represent a Map of entries using a simple array of (key,id) pairs
typedef struct {
  u32 keys[65536];
  u32 vals[65536];
  u32 size;
} Map;

// Function to insert a new entry into the map
__host__ void map_insert(Map* map, u32 key, u32 val) {
  map->keys[map->size] = key;
  map->vals[map->size] = val;
  map->size++;
}

// Function to lookup an id in the map by key
__host__ u32 map_lookup(Map* map, u32 key) {
  for (u32 i = 0; i < map->size; ++i) {
    if (map->keys[i] == key) {
      return map->vals[i];
    }
  }
  return map->size;
}

// Recursive function to print a term as a tree with unique variable IDs
__host__ void print_tree_go(Net* net, Ptr ptr, Map* var_ids) {
  if (is_var(ptr)) {
    u32 got = map_lookup(var_ids, ptr);
    if (got == var_ids->size) {
      u32 name = var_ids->size;
      Ptr targ = *target(net, enter(net, ptr));
      map_insert(var_ids, targ, name);
      printf("x%d", name);
    } else {
      printf("x%d", got);
    }
  } else if (is_ref(ptr)) {
    printf("{%x}", val(ptr));
  } else if (tag(ptr) == ERA) {
    printf("*");
  } else {
    switch (tag(ptr)) {
      case RD1: case RD2:
        print_tree_go(net, *target(net, ptr), var_ids);
        break;
      default:
        printf("(%d ", tag(ptr) - CT0);
        print_tree_go(net, net->heap[val(ptr)].ports[P1], var_ids);
        printf(" ");
        print_tree_go(net, net->heap[val(ptr)].ports[P2], var_ids);
        printf(")");
    }
  }
}

__host__ void print_tree(Net* net, Ptr ptr) {
  var_ids.size = 0;
  print_tree_go(net, ptr, &var_ids);
  printf("\n");
}

// Book
// ----

const u32 F_E = 0xe;
const u32 F_F = 0xf;
const u32 F_I = 0x12;
const u32 F_O = 0x18;
const u32 F_S = 0x1c;
const u32 F_T = 0x1d;
const u32 F_Z = 0x23;
const u32 F_af = 0x929;
const u32 F_c0 = 0x980;
const u32 F_c1 = 0x981;
const u32 F_c2 = 0x982;
const u32 F_c3 = 0x983;
const u32 F_c4 = 0x984;
const u32 F_c5 = 0x985;
const u32 F_c6 = 0x986;
const u32 F_c7 = 0x987;
const u32 F_c8 = 0x988;
const u32 F_c9 = 0x989;
const u32 F_id = 0xb27;
const u32 F_k0 = 0xb80;
const u32 F_k1 = 0xb81;
const u32 F_k2 = 0xb82;
const u32 F_k3 = 0xb83;
const u32 F_k4 = 0xb84;
const u32 F_k5 = 0xb85;
const u32 F_k6 = 0xb86;
const u32 F_k7 = 0xb87;
const u32 F_k8 = 0xb88;
const u32 F_k9 = 0xb89;
const u32 F_afS = 0x24a5c;
const u32 F_afZ = 0x24a63;
const u32 F_and = 0x24c67;
const u32 F_brn = 0x25d71;
const u32 F_c10 = 0x26040;
const u32 F_c11 = 0x26041;
const u32 F_c12 = 0x26042;
const u32 F_c13 = 0x26043;
const u32 F_c14 = 0x26044;
const u32 F_c15 = 0x26045;
const u32 F_c16 = 0x26046;
const u32 F_c17 = 0x26047;
const u32 F_c18 = 0x26048;
const u32 F_c19 = 0x26049;
const u32 F_c20 = 0x26080;
const u32 F_c21 = 0x26081;
const u32 F_c22 = 0x26082;
const u32 F_c23 = 0x26083;
const u32 F_c24 = 0x26084;
const u32 F_c25 = 0x26085;
const u32 F_c26 = 0x26086;
const u32 F_c_s = 0x26fb6;
const u32 F_c_z = 0x26fbd;
const u32 F_dec = 0x27a26;
const u32 F_ex0 = 0x28ec0;
const u32 F_ex1 = 0x28ec1;
const u32 F_ex2 = 0x28ec2;
const u32 F_ex3 = 0x28ec3;
const u32 F_ex4 = 0x28ec4;
const u32 F_ex5 = 0x28ec5;
const u32 F_g_s = 0x2afb6;
const u32 F_g_z = 0x2afbd;
const u32 F_k10 = 0x2e040;
const u32 F_k11 = 0x2e041;
const u32 F_k12 = 0x2e042;
const u32 F_k13 = 0x2e043;
const u32 F_k14 = 0x2e044;
const u32 F_k15 = 0x2e045;
const u32 F_k16 = 0x2e046;
const u32 F_k17 = 0x2e047;
const u32 F_k18 = 0x2e048;
const u32 F_k19 = 0x2e049;
const u32 F_k20 = 0x2e080;
const u32 F_k21 = 0x2e081;
const u32 F_k22 = 0x2e082;
const u32 F_k23 = 0x2e083;
const u32 F_k24 = 0x2e084;
const u32 F_low = 0x2fcba;
const u32 F_mul = 0x30e2f;
const u32 F_nid = 0x31b27;
const u32 F_not = 0x31cb7;
const u32 F_run = 0x35e31;
const u32 F_brnS = 0x975c5c;
const u32 F_brnZ = 0x975c63;
const u32 F_decI = 0x9e8992;
const u32 F_decO = 0x9e8998;
const u32 F_lowI = 0xbf2e92;
const u32 F_lowO = 0xbf2e98;
const u32 F_nidS = 0xc6c9dc;
const u32 F_runI = 0xd78c52;
const u32 F_runO = 0xd78c58;

u32 BOOK_DATA[] = {
  // @E
  // .nlen
  0x00000004,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x00000005, 0x0000002A,  0x00000005, 0x0000003A,  0x00000031, 0x00000030,
  // .rdex
  // @F
  // .nlen
  0x00000003,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x00000005, 0x0000002A,  0x00000021, 0x00000020,
  // .rdex
  // @I
  // .nlen
  0x00000006,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x00000040, 0x0000002A,  0x00000005, 0x0000003A,  0x0000004A, 0x0000005A,
  0x00000010, 0x00000051,  0x00000005, 0x00000041,
  // .rdex
  // @O
  // .nlen
  0x00000006,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x00000030, 0x0000002A,  0x0000003A, 0x0000004A,  0x00000010, 0x00000051,
  0x00000005, 0x0000005A,  0x00000005, 0x00000031,
  // .rdex
  // @S
  // .nlen
  0x00000005,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x00000030, 0x0000002A,  0x0000003A, 0x0000004A,  0x00000010, 0x00000041,
  0x00000005, 0x00000031,
  // .rdex
  // @T
  // .nlen
  0x00000003,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x00000021, 0x0000002A,  0x00000005, 0x00000010,
  // .rdex
  // @Z
  // .nlen
  0x00000003,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x00000005, 0x0000002A,  0x00000021, 0x00000020,
  // .rdex
  // @af
  // .nlen
  0x00000004,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002A, 0x00000031,  0x0024A5C4, 0x0000003A,  0x0024A634, 0x00000011,
  // .rdex
  // @c0
  // .nlen
  0x00000003,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x00000005, 0x0000002A,  0x00000021, 0x00000020,
  // .rdex
  // @c1
  // .nlen
  0x00000004,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002A, 0x0000003A,  0x00000030, 0x00000031,  0x00000020, 0x00000021,
  // .rdex
  // @c2
  // .nlen
  0x00000006,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x0000005A,  0x0000003A, 0x0000004A,  0x00000050, 0x00000040,
  0x00000031, 0x00000051,  0x00000030, 0x00000041,
  // .rdex
  // @c3
  // .nlen
  0x00000008,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x0000007A,  0x0000003B, 0x0000006A,  0x0000004A, 0x0000005A,
  0x00000070, 0x00000050,  0x00000041, 0x00000060,  0x00000051, 0x00000071,  0x00000040, 0x00000061,
  // .rdex
  // @c4
  // .nlen
  0x0000000A,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x0000009A,  0x0000003B, 0x0000008A,  0x0000004B, 0x0000007A,
  0x0000005A, 0x0000006A,  0x00000090, 0x00000060,  0x00000051, 0x00000070,  0x00000061, 0x00000080,
  0x00000071, 0x00000091,  0x00000050, 0x00000081,
  // .rdex
  // @c5
  // .nlen
  0x0000000C,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x000000BA,  0x0000003B, 0x000000AA,  0x0000004B, 0x0000009A,
  0x0000005B, 0x0000008A,  0x0000006A, 0x0000007A,  0x000000B0, 0x00000070,  0x00000061, 0x00000080,
  0x00000071, 0x00000090,  0x00000081, 0x000000A0,  0x00000091, 0x000000B1,  0x00000060, 0x000000A1,
  // .rdex
  // @c6
  // .nlen
  0x0000000E,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x000000DA,  0x0000003B, 0x000000CA,  0x0000004B, 0x000000BA,
  0x0000005B, 0x000000AA,  0x0000006B, 0x0000009A,  0x0000007A, 0x0000008A,  0x000000D0, 0x00000080,
  0x00000071, 0x00000090,  0x00000081, 0x000000A0,  0x00000091, 0x000000B0,  0x000000A1, 0x000000C0,
  0x000000B1, 0x000000D1,  0x00000070, 0x000000C1,
  // .rdex
  // @c7
  // .nlen
  0x00000010,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x000000FA,  0x0000003B, 0x000000EA,  0x0000004B, 0x000000DA,
  0x0000005B, 0x000000CA,  0x0000006B, 0x000000BA,  0x0000007B, 0x000000AA,  0x0000008A, 0x0000009A,
  0x000000F0, 0x00000090,  0x00000081, 0x000000A0,  0x00000091, 0x000000B0,  0x000000A1, 0x000000C0,
  0x000000B1, 0x000000D0,  0x000000C1, 0x000000E0,  0x000000D1, 0x000000F1,  0x00000080, 0x000000E1,
  // .rdex
  // @c8
  // .nlen
  0x00000012,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x0000011A,  0x0000003B, 0x0000010A,  0x0000004B, 0x000000FA,
  0x0000005B, 0x000000EA,  0x0000006B, 0x000000DA,  0x0000007B, 0x000000CA,  0x0000008B, 0x000000BA,
  0x0000009A, 0x000000AA,  0x00000110, 0x000000A0,  0x00000091, 0x000000B0,  0x000000A1, 0x000000C0,
  0x000000B1, 0x000000D0,  0x000000C1, 0x000000E0,  0x000000D1, 0x000000F0,  0x000000E1, 0x00000100,
  0x000000F1, 0x00000111,  0x00000090, 0x00000101,
  // .rdex
  // @c9
  // .nlen
  0x00000014,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x0000013A,  0x0000003B, 0x0000012A,  0x0000004B, 0x0000011A,
  0x0000005B, 0x0000010A,  0x0000006B, 0x000000FA,  0x0000007B, 0x000000EA,  0x0000008B, 0x000000DA,
  0x0000009B, 0x000000CA,  0x000000AA, 0x000000BA,  0x00000130, 0x000000B0,  0x000000A1, 0x000000C0,
  0x000000B1, 0x000000D0,  0x000000C1, 0x000000E0,  0x000000D1, 0x000000F0,  0x000000E1, 0x00000100,
  0x000000F1, 0x00000110,  0x00000101, 0x00000120,  0x00000111, 0x00000131,  0x000000A0, 0x00000121,
  // .rdex
  // @id
  // .nlen
  0x00000002,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x00000011, 0x00000010,
  // .rdex
  // @k0
  // .nlen
  0x00000003,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x00000005, 0x0000002A,  0x00000021, 0x00000020,
  // .rdex
  // @k1
  // .nlen
  0x00000004,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002A, 0x0000003A,  0x00000030, 0x00000031,  0x00000020, 0x00000021,
  // .rdex
  // @k2
  // .nlen
  0x00000006,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002C, 0x0000005A,  0x0000003A, 0x0000004A,  0x00000050, 0x00000040,
  0x00000031, 0x00000051,  0x00000030, 0x00000041,
  // .rdex
  // @k3
  // .nlen
  0x00000008,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002C, 0x0000007A,  0x0000003C, 0x0000006A,  0x0000004A, 0x0000005A,
  0x00000070, 0x00000050,  0x00000041, 0x00000060,  0x00000051, 0x00000071,  0x00000040, 0x00000061,
  // .rdex
  // @k4
  // .nlen
  0x0000000A,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002C, 0x0000009A,  0x0000003C, 0x0000008A,  0x0000004C, 0x0000007A,
  0x0000005A, 0x0000006A,  0x00000090, 0x00000060,  0x00000051, 0x00000070,  0x00000061, 0x00000080,
  0x00000071, 0x00000091,  0x00000050, 0x00000081,
  // .rdex
  // @k5
  // .nlen
  0x0000000C,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002C, 0x000000BA,  0x0000003C, 0x000000AA,  0x0000004C, 0x0000009A,
  0x0000005C, 0x0000008A,  0x0000006A, 0x0000007A,  0x000000B0, 0x00000070,  0x00000061, 0x00000080,
  0x00000071, 0x00000090,  0x00000081, 0x000000A0,  0x00000091, 0x000000B1,  0x00000060, 0x000000A1,
  // .rdex
  // @k6
  // .nlen
  0x0000000E,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002C, 0x000000DA,  0x0000003C, 0x000000CA,  0x0000004C, 0x000000BA,
  0x0000005C, 0x000000AA,  0x0000006C, 0x0000009A,  0x0000007A, 0x0000008A,  0x000000D0, 0x00000080,
  0x00000071, 0x00000090,  0x00000081, 0x000000A0,  0x00000091, 0x000000B0,  0x000000A1, 0x000000C0,
  0x000000B1, 0x000000D1,  0x00000070, 0x000000C1,
  // .rdex
  // @k7
  // .nlen
  0x00000010,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002C, 0x000000FA,  0x0000003C, 0x000000EA,  0x0000004C, 0x000000DA,
  0x0000005C, 0x000000CA,  0x0000006C, 0x000000BA,  0x0000007C, 0x000000AA,  0x0000008A, 0x0000009A,
  0x000000F0, 0x00000090,  0x00000081, 0x000000A0,  0x00000091, 0x000000B0,  0x000000A1, 0x000000C0,
  0x000000B1, 0x000000D0,  0x000000C1, 0x000000E0,  0x000000D1, 0x000000F1,  0x00000080, 0x000000E1,
  // .rdex
  // @k8
  // .nlen
  0x00000012,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002C, 0x0000011A,  0x0000003C, 0x0000010A,  0x0000004C, 0x000000FA,
  0x0000005C, 0x000000EA,  0x0000006C, 0x000000DA,  0x0000007C, 0x000000CA,  0x0000008C, 0x000000BA,
  0x0000009A, 0x000000AA,  0x00000110, 0x000000A0,  0x00000091, 0x000000B0,  0x000000A1, 0x000000C0,
  0x000000B1, 0x000000D0,  0x000000C1, 0x000000E0,  0x000000D1, 0x000000F0,  0x000000E1, 0x00000100,
  0x000000F1, 0x00000111,  0x00000090, 0x00000101,
  // .rdex
  // @k9
  // .nlen
  0x00000014,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002C, 0x0000013A,  0x0000003C, 0x0000012A,  0x0000004C, 0x0000011A,
  0x0000005C, 0x0000010A,  0x0000006C, 0x000000FA,  0x0000007C, 0x000000EA,  0x0000008C, 0x000000DA,
  0x0000009C, 0x000000CA,  0x000000AA, 0x000000BA,  0x00000130, 0x000000B0,  0x000000A1, 0x000000C0,
  0x000000B1, 0x000000D0,  0x000000C1, 0x000000E0,  0x000000D1, 0x000000F0,  0x000000E1, 0x00000100,
  0x000000F1, 0x00000110,  0x00000101, 0x00000120,  0x00000111, 0x00000131,  0x000000A0, 0x00000121,
  // .rdex
  // @afS
  // .nlen
  0x00000007,
  // .rlen
  0x00000003,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x00000051,  0x00000060, 0x00000030,  0x00000021, 0x00000050,
  0x00000061, 0x0000005A,  0x00000031, 0x00000011,  0x00000020, 0x00000040,
  // .rdex
  0x0000003A, 0x00009294,  0x0000004A, 0x0024C674,  0x0000006A, 0x00009294,
  // @afZ
  // .nlen
  0x00000001,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x000001D4,
  // .rdex
  // @and
  // .nlen
  0x0000000A,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002A, 0x00000061,  0x0000003A, 0x0000006A,  0x0000004A, 0x00000051,
  0x000001D4, 0x0000005A,  0x000000F4, 0x00000031,  0x0000007A, 0x00000011,  0x0000008A, 0x00000091,
  0x000000F4, 0x0000009A,  0x000000F4, 0x00000071,
  // .rdex
  // @brn
  // .nlen
  0x00000004,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002A, 0x00000031,  0x0975C5C4, 0x0000003A,  0x0975C634, 0x00000011,
  // .rdex
  // @c10
  // .nlen
  0x00000016,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x0000015A,  0x0000003B, 0x0000014A,  0x0000004B, 0x0000013A,
  0x0000005B, 0x0000012A,  0x0000006B, 0x0000011A,  0x0000007B, 0x0000010A,  0x0000008B, 0x000000FA,
  0x0000009B, 0x000000EA,  0x000000AB, 0x000000DA,  0x000000BA, 0x000000CA,  0x00000150, 0x000000C0,
  0x000000B1, 0x000000D0,  0x000000C1, 0x000000E0,  0x000000D1, 0x000000F0,  0x000000E1, 0x00000100,
  0x000000F1, 0x00000110,  0x00000101, 0x00000120,  0x00000111, 0x00000130,  0x00000121, 0x00000140,
  0x00000131, 0x00000151,  0x000000B0, 0x00000141,
  // .rdex
  // @c11
  // .nlen
  0x00000018,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x0000017A,  0x0000003B, 0x0000016A,  0x0000004B, 0x0000015A,
  0x0000005B, 0x0000014A,  0x0000006B, 0x0000013A,  0x0000007B, 0x0000012A,  0x0000008B, 0x0000011A,
  0x0000009B, 0x0000010A,  0x000000AB, 0x000000FA,  0x000000BB, 0x000000EA,  0x000000CA, 0x000000DA,
  0x00000170, 0x000000D0,  0x000000C1, 0x000000E0,  0x000000D1, 0x000000F0,  0x000000E1, 0x00000100,
  0x000000F1, 0x00000110,  0x00000101, 0x00000120,  0x00000111, 0x00000130,  0x00000121, 0x00000140,
  0x00000131, 0x00000150,  0x00000141, 0x00000160,  0x00000151, 0x00000171,  0x000000C0, 0x00000161,
  // .rdex
  // @c12
  // .nlen
  0x0000001A,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x0000019A,  0x0000003B, 0x0000018A,  0x0000004B, 0x0000017A,
  0x0000005B, 0x0000016A,  0x0000006B, 0x0000015A,  0x0000007B, 0x0000014A,  0x0000008B, 0x0000013A,
  0x0000009B, 0x0000012A,  0x000000AB, 0x0000011A,  0x000000BB, 0x0000010A,  0x000000CB, 0x000000FA,
  0x000000DA, 0x000000EA,  0x00000190, 0x000000E0,  0x000000D1, 0x000000F0,  0x000000E1, 0x00000100,
  0x000000F1, 0x00000110,  0x00000101, 0x00000120,  0x00000111, 0x00000130,  0x00000121, 0x00000140,
  0x00000131, 0x00000150,  0x00000141, 0x00000160,  0x00000151, 0x00000170,  0x00000161, 0x00000180,
  0x00000171, 0x00000191,  0x000000D0, 0x00000181,
  // .rdex
  // @c13
  // .nlen
  0x0000001C,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x000001BA,  0x0000003B, 0x000001AA,  0x0000004B, 0x0000019A,
  0x0000005B, 0x0000018A,  0x0000006B, 0x0000017A,  0x0000007B, 0x0000016A,  0x0000008B, 0x0000015A,
  0x0000009B, 0x0000014A,  0x000000AB, 0x0000013A,  0x000000BB, 0x0000012A,  0x000000CB, 0x0000011A,
  0x000000DB, 0x0000010A,  0x000000EA, 0x000000FA,  0x000001B0, 0x000000F0,  0x000000E1, 0x00000100,
  0x000000F1, 0x00000110,  0x00000101, 0x00000120,  0x00000111, 0x00000130,  0x00000121, 0x00000140,
  0x00000131, 0x00000150,  0x00000141, 0x00000160,  0x00000151, 0x00000170,  0x00000161, 0x00000180,
  0x00000171, 0x00000190,  0x00000181, 0x000001A0,  0x00000191, 0x000001B1,  0x000000E0, 0x000001A1,
  // .rdex
  // @c14
  // .nlen
  0x0000001E,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x000001DA,  0x0000003B, 0x000001CA,  0x0000004B, 0x000001BA,
  0x0000005B, 0x000001AA,  0x0000006B, 0x0000019A,  0x0000007B, 0x0000018A,  0x0000008B, 0x0000017A,
  0x0000009B, 0x0000016A,  0x000000AB, 0x0000015A,  0x000000BB, 0x0000014A,  0x000000CB, 0x0000013A,
  0x000000DB, 0x0000012A,  0x000000EB, 0x0000011A,  0x000000FA, 0x0000010A,  0x000001D0, 0x00000100,
  0x000000F1, 0x00000110,  0x00000101, 0x00000120,  0x00000111, 0x00000130,  0x00000121, 0x00000140,
  0x00000131, 0x00000150,  0x00000141, 0x00000160,  0x00000151, 0x00000170,  0x00000161, 0x00000180,
  0x00000171, 0x00000190,  0x00000181, 0x000001A0,  0x00000191, 0x000001B0,  0x000001A1, 0x000001C0,
  0x000001B1, 0x000001D1,  0x000000F0, 0x000001C1,
  // .rdex
  // @c15
  // .nlen
  0x00000020,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x000001FA,  0x0000003B, 0x000001EA,  0x0000004B, 0x000001DA,
  0x0000005B, 0x000001CA,  0x0000006B, 0x000001BA,  0x0000007B, 0x000001AA,  0x0000008B, 0x0000019A,
  0x0000009B, 0x0000018A,  0x000000AB, 0x0000017A,  0x000000BB, 0x0000016A,  0x000000CB, 0x0000015A,
  0x000000DB, 0x0000014A,  0x000000EB, 0x0000013A,  0x000000FB, 0x0000012A,  0x0000010A, 0x0000011A,
  0x000001F0, 0x00000110,  0x00000101, 0x00000120,  0x00000111, 0x00000130,  0x00000121, 0x00000140,
  0x00000131, 0x00000150,  0x00000141, 0x00000160,  0x00000151, 0x00000170,  0x00000161, 0x00000180,
  0x00000171, 0x00000190,  0x00000181, 0x000001A0,  0x00000191, 0x000001B0,  0x000001A1, 0x000001C0,
  0x000001B1, 0x000001D0,  0x000001C1, 0x000001E0,  0x000001D1, 0x000001F1,  0x00000100, 0x000001E1,
  // .rdex
  // @c16
  // .nlen
  0x00000022,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x0000021A,  0x0000003B, 0x0000020A,  0x0000004B, 0x000001FA,
  0x0000005B, 0x000001EA,  0x0000006B, 0x000001DA,  0x0000007B, 0x000001CA,  0x0000008B, 0x000001BA,
  0x0000009B, 0x000001AA,  0x000000AB, 0x0000019A,  0x000000BB, 0x0000018A,  0x000000CB, 0x0000017A,
  0x000000DB, 0x0000016A,  0x000000EB, 0x0000015A,  0x000000FB, 0x0000014A,  0x0000010B, 0x0000013A,
  0x0000011A, 0x0000012A,  0x00000210, 0x00000120,  0x00000111, 0x00000130,  0x00000121, 0x00000140,
  0x00000131, 0x00000150,  0x00000141, 0x00000160,  0x00000151, 0x00000170,  0x00000161, 0x00000180,
  0x00000171, 0x00000190,  0x00000181, 0x000001A0,  0x00000191, 0x000001B0,  0x000001A1, 0x000001C0,
  0x000001B1, 0x000001D0,  0x000001C1, 0x000001E0,  0x000001D1, 0x000001F0,  0x000001E1, 0x00000200,
  0x000001F1, 0x00000211,  0x00000110, 0x00000201,
  // .rdex
  // @c17
  // .nlen
  0x00000024,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x0000023A,  0x0000003B, 0x0000022A,  0x0000004B, 0x0000021A,
  0x0000005B, 0x0000020A,  0x0000006B, 0x000001FA,  0x0000007B, 0x000001EA,  0x0000008B, 0x000001DA,
  0x0000009B, 0x000001CA,  0x000000AB, 0x000001BA,  0x000000BB, 0x000001AA,  0x000000CB, 0x0000019A,
  0x000000DB, 0x0000018A,  0x000000EB, 0x0000017A,  0x000000FB, 0x0000016A,  0x0000010B, 0x0000015A,
  0x0000011B, 0x0000014A,  0x0000012A, 0x0000013A,  0x00000230, 0x00000130,  0x00000121, 0x00000140,
  0x00000131, 0x00000150,  0x00000141, 0x00000160,  0x00000151, 0x00000170,  0x00000161, 0x00000180,
  0x00000171, 0x00000190,  0x00000181, 0x000001A0,  0x00000191, 0x000001B0,  0x000001A1, 0x000001C0,
  0x000001B1, 0x000001D0,  0x000001C1, 0x000001E0,  0x000001D1, 0x000001F0,  0x000001E1, 0x00000200,
  0x000001F1, 0x00000210,  0x00000201, 0x00000220,  0x00000211, 0x00000231,  0x00000120, 0x00000221,
  // .rdex
  // @c18
  // .nlen
  0x00000026,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x0000025A,  0x0000003B, 0x0000024A,  0x0000004B, 0x0000023A,
  0x0000005B, 0x0000022A,  0x0000006B, 0x0000021A,  0x0000007B, 0x0000020A,  0x0000008B, 0x000001FA,
  0x0000009B, 0x000001EA,  0x000000AB, 0x000001DA,  0x000000BB, 0x000001CA,  0x000000CB, 0x000001BA,
  0x000000DB, 0x000001AA,  0x000000EB, 0x0000019A,  0x000000FB, 0x0000018A,  0x0000010B, 0x0000017A,
  0x0000011B, 0x0000016A,  0x0000012B, 0x0000015A,  0x0000013A, 0x0000014A,  0x00000250, 0x00000140,
  0x00000131, 0x00000150,  0x00000141, 0x00000160,  0x00000151, 0x00000170,  0x00000161, 0x00000180,
  0x00000171, 0x00000190,  0x00000181, 0x000001A0,  0x00000191, 0x000001B0,  0x000001A1, 0x000001C0,
  0x000001B1, 0x000001D0,  0x000001C1, 0x000001E0,  0x000001D1, 0x000001F0,  0x000001E1, 0x00000200,
  0x000001F1, 0x00000210,  0x00000201, 0x00000220,  0x00000211, 0x00000230,  0x00000221, 0x00000240,
  0x00000231, 0x00000251,  0x00000130, 0x00000241,
  // .rdex
  // @c19
  // .nlen
  0x00000028,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x0000027A,  0x0000003B, 0x0000026A,  0x0000004B, 0x0000025A,
  0x0000005B, 0x0000024A,  0x0000006B, 0x0000023A,  0x0000007B, 0x0000022A,  0x0000008B, 0x0000021A,
  0x0000009B, 0x0000020A,  0x000000AB, 0x000001FA,  0x000000BB, 0x000001EA,  0x000000CB, 0x000001DA,
  0x000000DB, 0x000001CA,  0x000000EB, 0x000001BA,  0x000000FB, 0x000001AA,  0x0000010B, 0x0000019A,
  0x0000011B, 0x0000018A,  0x0000012B, 0x0000017A,  0x0000013B, 0x0000016A,  0x0000014A, 0x0000015A,
  0x00000270, 0x00000150,  0x00000141, 0x00000160,  0x00000151, 0x00000170,  0x00000161, 0x00000180,
  0x00000171, 0x00000190,  0x00000181, 0x000001A0,  0x00000191, 0x000001B0,  0x000001A1, 0x000001C0,
  0x000001B1, 0x000001D0,  0x000001C1, 0x000001E0,  0x000001D1, 0x000001F0,  0x000001E1, 0x00000200,
  0x000001F1, 0x00000210,  0x00000201, 0x00000220,  0x00000211, 0x00000230,  0x00000221, 0x00000240,
  0x00000231, 0x00000250,  0x00000241, 0x00000260,  0x00000251, 0x00000271,  0x00000140, 0x00000261,
  // .rdex
  // @c20
  // .nlen
  0x0000002A,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x0000029A,  0x0000003B, 0x0000028A,  0x0000004B, 0x0000027A,
  0x0000005B, 0x0000026A,  0x0000006B, 0x0000025A,  0x0000007B, 0x0000024A,  0x0000008B, 0x0000023A,
  0x0000009B, 0x0000022A,  0x000000AB, 0x0000021A,  0x000000BB, 0x0000020A,  0x000000CB, 0x000001FA,
  0x000000DB, 0x000001EA,  0x000000EB, 0x000001DA,  0x000000FB, 0x000001CA,  0x0000010B, 0x000001BA,
  0x0000011B, 0x000001AA,  0x0000012B, 0x0000019A,  0x0000013B, 0x0000018A,  0x0000014B, 0x0000017A,
  0x0000015A, 0x0000016A,  0x00000290, 0x00000160,  0x00000151, 0x00000170,  0x00000161, 0x00000180,
  0x00000171, 0x00000190,  0x00000181, 0x000001A0,  0x00000191, 0x000001B0,  0x000001A1, 0x000001C0,
  0x000001B1, 0x000001D0,  0x000001C1, 0x000001E0,  0x000001D1, 0x000001F0,  0x000001E1, 0x00000200,
  0x000001F1, 0x00000210,  0x00000201, 0x00000220,  0x00000211, 0x00000230,  0x00000221, 0x00000240,
  0x00000231, 0x00000250,  0x00000241, 0x00000260,  0x00000251, 0x00000270,  0x00000261, 0x00000280,
  0x00000271, 0x00000291,  0x00000150, 0x00000281,
  // .rdex
  // @c21
  // .nlen
  0x0000002C,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x000002BA,  0x0000003B, 0x000002AA,  0x0000004B, 0x0000029A,
  0x0000005B, 0x0000028A,  0x0000006B, 0x0000027A,  0x0000007B, 0x0000026A,  0x0000008B, 0x0000025A,
  0x0000009B, 0x0000024A,  0x000000AB, 0x0000023A,  0x000000BB, 0x0000022A,  0x000000CB, 0x0000021A,
  0x000000DB, 0x0000020A,  0x000000EB, 0x000001FA,  0x000000FB, 0x000001EA,  0x0000010B, 0x000001DA,
  0x0000011B, 0x000001CA,  0x0000012B, 0x000001BA,  0x0000013B, 0x000001AA,  0x0000014B, 0x0000019A,
  0x0000015B, 0x0000018A,  0x0000016A, 0x0000017A,  0x000002B0, 0x00000170,  0x00000161, 0x00000180,
  0x00000171, 0x00000190,  0x00000181, 0x000001A0,  0x00000191, 0x000001B0,  0x000001A1, 0x000001C0,
  0x000001B1, 0x000001D0,  0x000001C1, 0x000001E0,  0x000001D1, 0x000001F0,  0x000001E1, 0x00000200,
  0x000001F1, 0x00000210,  0x00000201, 0x00000220,  0x00000211, 0x00000230,  0x00000221, 0x00000240,
  0x00000231, 0x00000250,  0x00000241, 0x00000260,  0x00000251, 0x00000270,  0x00000261, 0x00000280,
  0x00000271, 0x00000290,  0x00000281, 0x000002A0,  0x00000291, 0x000002B1,  0x00000160, 0x000002A1,
  // .rdex
  // @c22
  // .nlen
  0x0000002E,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x000002DA,  0x0000003B, 0x000002CA,  0x0000004B, 0x000002BA,
  0x0000005B, 0x000002AA,  0x0000006B, 0x0000029A,  0x0000007B, 0x0000028A,  0x0000008B, 0x0000027A,
  0x0000009B, 0x0000026A,  0x000000AB, 0x0000025A,  0x000000BB, 0x0000024A,  0x000000CB, 0x0000023A,
  0x000000DB, 0x0000022A,  0x000000EB, 0x0000021A,  0x000000FB, 0x0000020A,  0x0000010B, 0x000001FA,
  0x0000011B, 0x000001EA,  0x0000012B, 0x000001DA,  0x0000013B, 0x000001CA,  0x0000014B, 0x000001BA,
  0x0000015B, 0x000001AA,  0x0000016B, 0x0000019A,  0x0000017A, 0x0000018A,  0x000002D0, 0x00000180,
  0x00000171, 0x00000190,  0x00000181, 0x000001A0,  0x00000191, 0x000001B0,  0x000001A1, 0x000001C0,
  0x000001B1, 0x000001D0,  0x000001C1, 0x000001E0,  0x000001D1, 0x000001F0,  0x000001E1, 0x00000200,
  0x000001F1, 0x00000210,  0x00000201, 0x00000220,  0x00000211, 0x00000230,  0x00000221, 0x00000240,
  0x00000231, 0x00000250,  0x00000241, 0x00000260,  0x00000251, 0x00000270,  0x00000261, 0x00000280,
  0x00000271, 0x00000290,  0x00000281, 0x000002A0,  0x00000291, 0x000002B0,  0x000002A1, 0x000002C0,
  0x000002B1, 0x000002D1,  0x00000170, 0x000002C1,
  // .rdex
  // @c23
  // .nlen
  0x00000030,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x000002FA,  0x0000003B, 0x000002EA,  0x0000004B, 0x000002DA,
  0x0000005B, 0x000002CA,  0x0000006B, 0x000002BA,  0x0000007B, 0x000002AA,  0x0000008B, 0x0000029A,
  0x0000009B, 0x0000028A,  0x000000AB, 0x0000027A,  0x000000BB, 0x0000026A,  0x000000CB, 0x0000025A,
  0x000000DB, 0x0000024A,  0x000000EB, 0x0000023A,  0x000000FB, 0x0000022A,  0x0000010B, 0x0000021A,
  0x0000011B, 0x0000020A,  0x0000012B, 0x000001FA,  0x0000013B, 0x000001EA,  0x0000014B, 0x000001DA,
  0x0000015B, 0x000001CA,  0x0000016B, 0x000001BA,  0x0000017B, 0x000001AA,  0x0000018A, 0x0000019A,
  0x000002F0, 0x00000190,  0x00000181, 0x000001A0,  0x00000191, 0x000001B0,  0x000001A1, 0x000001C0,
  0x000001B1, 0x000001D0,  0x000001C1, 0x000001E0,  0x000001D1, 0x000001F0,  0x000001E1, 0x00000200,
  0x000001F1, 0x00000210,  0x00000201, 0x00000220,  0x00000211, 0x00000230,  0x00000221, 0x00000240,
  0x00000231, 0x00000250,  0x00000241, 0x00000260,  0x00000251, 0x00000270,  0x00000261, 0x00000280,
  0x00000271, 0x00000290,  0x00000281, 0x000002A0,  0x00000291, 0x000002B0,  0x000002A1, 0x000002C0,
  0x000002B1, 0x000002D0,  0x000002C1, 0x000002E0,  0x000002D1, 0x000002F1,  0x00000180, 0x000002E1,
  // .rdex
  // @c24
  // .nlen
  0x00000032,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x0000031A,  0x0000003B, 0x0000030A,  0x0000004B, 0x000002FA,
  0x0000005B, 0x000002EA,  0x0000006B, 0x000002DA,  0x0000007B, 0x000002CA,  0x0000008B, 0x000002BA,
  0x0000009B, 0x000002AA,  0x000000AB, 0x0000029A,  0x000000BB, 0x0000028A,  0x000000CB, 0x0000027A,
  0x000000DB, 0x0000026A,  0x000000EB, 0x0000025A,  0x000000FB, 0x0000024A,  0x0000010B, 0x0000023A,
  0x0000011B, 0x0000022A,  0x0000012B, 0x0000021A,  0x0000013B, 0x0000020A,  0x0000014B, 0x000001FA,
  0x0000015B, 0x000001EA,  0x0000016B, 0x000001DA,  0x0000017B, 0x000001CA,  0x0000018B, 0x000001BA,
  0x0000019A, 0x000001AA,  0x00000310, 0x000001A0,  0x00000191, 0x000001B0,  0x000001A1, 0x000001C0,
  0x000001B1, 0x000001D0,  0x000001C1, 0x000001E0,  0x000001D1, 0x000001F0,  0x000001E1, 0x00000200,
  0x000001F1, 0x00000210,  0x00000201, 0x00000220,  0x00000211, 0x00000230,  0x00000221, 0x00000240,
  0x00000231, 0x00000250,  0x00000241, 0x00000260,  0x00000251, 0x00000270,  0x00000261, 0x00000280,
  0x00000271, 0x00000290,  0x00000281, 0x000002A0,  0x00000291, 0x000002B0,  0x000002A1, 0x000002C0,
  0x000002B1, 0x000002D0,  0x000002C1, 0x000002E0,  0x000002D1, 0x000002F0,  0x000002E1, 0x00000300,
  0x000002F1, 0x00000311,  0x00000190, 0x00000301,
  // .rdex
  // @c25
  // .nlen
  0x00000034,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x0000033A,  0x0000003B, 0x0000032A,  0x0000004B, 0x0000031A,
  0x0000005B, 0x0000030A,  0x0000006B, 0x000002FA,  0x0000007B, 0x000002EA,  0x0000008B, 0x000002DA,
  0x0000009B, 0x000002CA,  0x000000AB, 0x000002BA,  0x000000BB, 0x000002AA,  0x000000CB, 0x0000029A,
  0x000000DB, 0x0000028A,  0x000000EB, 0x0000027A,  0x000000FB, 0x0000026A,  0x0000010B, 0x0000025A,
  0x0000011B, 0x0000024A,  0x0000012B, 0x0000023A,  0x0000013B, 0x0000022A,  0x0000014B, 0x0000021A,
  0x0000015B, 0x0000020A,  0x0000016B, 0x000001FA,  0x0000017B, 0x000001EA,  0x0000018B, 0x000001DA,
  0x0000019B, 0x000001CA,  0x000001AA, 0x000001BA,  0x00000000, 0x00000330,  0x000001A1, 0x000001C0,
  0x000001B1, 0x000001D0,  0x000001C1, 0x000001E0,  0x000001D1, 0x000001F0,  0x000001E1, 0x00000200,
  0x000001F1, 0x00000210,  0x00000201, 0x00000220,  0x00000211, 0x00000230,  0x00000221, 0x00000240,
  0x00000231, 0x00000250,  0x00000241, 0x00000260,  0x00000251, 0x00000270,  0x00000261, 0x00000280,
  0x00000271, 0x00000290,  0x00000281, 0x000002A0,  0x00000291, 0x000002B0,  0x000002A1, 0x000002C0,
  0x000002B1, 0x000002D0,  0x000002C1, 0x000002E0,  0x000002D1, 0x000002F0,  0x000002E1, 0x00000300,
  0x000002F1, 0x00000310,  0x00000301, 0x00000320,  0x00000311, 0x00000331,  0x000001A1, 0x00000321,
  // .rdex
  // @c26
  // .nlen
  0x00000036,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x0000035A,  0x0000003B, 0x0000034A,  0x0000004B, 0x0000033A,
  0x0000005B, 0x0000032A,  0x0000006B, 0x0000031A,  0x0000007B, 0x0000030A,  0x0000008B, 0x000002FA,
  0x0000009B, 0x000002EA,  0x000000AB, 0x000002DA,  0x000000BB, 0x000002CA,  0x000000CB, 0x000002BA,
  0x000000DB, 0x000002AA,  0x000000EB, 0x0000029A,  0x000000FB, 0x0000028A,  0x0000010B, 0x0000027A,
  0x0000011B, 0x0000026A,  0x0000012B, 0x0000025A,  0x0000013B, 0x0000024A,  0x0000014B, 0x0000023A,
  0x0000015B, 0x0000022A,  0x0000016B, 0x0000021A,  0x0000017B, 0x0000020A,  0x0000018B, 0x000001FA,
  0x0000019B, 0x000001EA,  0x000001AB, 0x000001DA,  0x000001BA, 0x000001CA,  0x00000000, 0x000001C0,
  0x000001B1, 0x00000350,  0x000001C1, 0x000001E0,  0x000001D1, 0x000001F0,  0x000001E1, 0x00000200,
  0x000001F1, 0x00000210,  0x00000201, 0x00000220,  0x00000211, 0x00000230,  0x00000221, 0x00000240,
  0x00000231, 0x00000250,  0x00000241, 0x00000260,  0x00000251, 0x00000270,  0x00000261, 0x00000280,
  0x00000271, 0x00000290,  0x00000281, 0x000002A0,  0x00000291, 0x000002B0,  0x000002A1, 0x000002C0,
  0x000002B1, 0x000002D0,  0x000002C1, 0x000002E0,  0x000002D1, 0x000002F0,  0x000002E1, 0x00000300,
  0x000002F1, 0x00000310,  0x00000301, 0x00000320,  0x00000311, 0x00000330,  0x00000321, 0x00000340,
  0x00000331, 0x00000351,  0x000001C1, 0x00000341,
  // .rdex
  // @c_s
  // .nlen
  0x00000008,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002A, 0x0000004A,  0x00000051, 0x0000003A,  0x00000070, 0x00000060,
  0x0000005B, 0x0000007A,  0x0000006A, 0x00000020,  0x00000031, 0x00000071,  0x00000030, 0x00000061,
  // .rdex
  // @c_z
  // .nlen
  0x00000003,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x00000005, 0x0000002A,  0x00000021, 0x00000020,
  // .rdex
  // @dec
  // .nlen
  0x00000005,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002A, 0x00000041,  0x09E89984, 0x0000003A,  0x09E89924, 0x0000004A,
  0x000000E4, 0x00000011,
  // .rdex
  // @ex0
  // .nlen
  0x00000002,
  // .rlen
  0x00000001,
  // .node
  0x00000000, 0x00000011,  0x0000B824, 0x00000001,
  // .rdex
  0x00009824, 0x0000001A,
  // @ex1
  // .nlen
  0x00000003,
  // .rlen
  0x00000001,
  // .node
  0x00000000, 0x00000021,  0x002AFB64, 0x0000002A,  0x002AFBD4, 0x00000001,
  // .rdex
  0x00260844, 0x0000001A,
  // @ex2
  // .nlen
  0x00000004,
  // .rlen
  0x00000002,
  // .node
  0x00000000, 0x00000031,  0x00000124, 0x0000002A,  0x000000E4, 0x00000030,  0x00000021, 0x00000001,
  // .rdex
  0x00260864, 0x0000001A,  0x0035E314, 0x0000003A,
  // @ex3
  // .nlen
  0x00000004,
  // .rlen
  0x00000002,
  // .node
  0x00000000, 0x00000031,  0x000001C4, 0x0000002A,  0x00000234, 0x00000030,  0x00000021, 0x00000001,
  // .rdex
  0x00260464, 0x0000001A,  0x0025D714, 0x0000003A,
  // @ex4
  // .nlen
  0x00000003,
  // .rlen
  0x00000001,
  // .node
  0x00000000, 0x00000021,  0x000007B6, 0x00000027,  0x00001416, 0x00000001,
  // .rdex
  0x00000036, 0x00000017,
  // @ex5
  // .nlen
  0x00000003,
  // .rlen
  0x00000001,
  // .node
  0x00000000, 0x00000011,  0x0000002A, 0x00000001,  0x000007B6, 0x00001416,
  // .rdex
  0x00000016, 0x00000019,
  // @g_s
  // .nlen
  0x00000006,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002C, 0x0000003A,  0x00000040, 0x00000050,  0x0000004A, 0x00000051,
  0x00000020, 0x0000005A,  0x00000021, 0x00000031,
  // .rdex
  // @g_z
  // .nlen
  0x00000002,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x00000011, 0x00000010,
  // .rdex
  // @k10
  // .nlen
  0x00000016,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002C, 0x0000015A,  0x0000003C, 0x0000014A,  0x0000004C, 0x0000013A,
  0x0000005C, 0x0000012A,  0x0000006C, 0x0000011A,  0x0000007C, 0x0000010A,  0x0000008C, 0x000000FA,
  0x0000009C, 0x000000EA,  0x000000AC, 0x000000DA,  0x000000BA, 0x000000CA,  0x00000150, 0x000000C0,
  0x000000B1, 0x000000D0,  0x000000C1, 0x000000E0,  0x000000D1, 0x000000F0,  0x000000E1, 0x00000100,
  0x000000F1, 0x00000110,  0x00000101, 0x00000120,  0x00000111, 0x00000130,  0x00000121, 0x00000140,
  0x00000131, 0x00000151,  0x000000B0, 0x00000141,
  // .rdex
  // @k11
  // .nlen
  0x00000018,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002C, 0x0000017A,  0x0000003C, 0x0000016A,  0x0000004C, 0x0000015A,
  0x0000005C, 0x0000014A,  0x0000006C, 0x0000013A,  0x0000007C, 0x0000012A,  0x0000008C, 0x0000011A,
  0x0000009C, 0x0000010A,  0x000000AC, 0x000000FA,  0x000000BC, 0x000000EA,  0x000000CA, 0x000000DA,
  0x00000170, 0x000000D0,  0x000000C1, 0x000000E0,  0x000000D1, 0x000000F0,  0x000000E1, 0x00000100,
  0x000000F1, 0x00000110,  0x00000101, 0x00000120,  0x00000111, 0x00000130,  0x00000121, 0x00000140,
  0x00000131, 0x00000150,  0x00000141, 0x00000160,  0x00000151, 0x00000171,  0x000000C0, 0x00000161,
  // .rdex
  // @k12
  // .nlen
  0x0000001A,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002C, 0x0000019A,  0x0000003C, 0x0000018A,  0x0000004C, 0x0000017A,
  0x0000005C, 0x0000016A,  0x0000006C, 0x0000015A,  0x0000007C, 0x0000014A,  0x0000008C, 0x0000013A,
  0x0000009C, 0x0000012A,  0x000000AC, 0x0000011A,  0x000000BC, 0x0000010A,  0x000000CC, 0x000000FA,
  0x000000DA, 0x000000EA,  0x00000190, 0x000000E0,  0x000000D1, 0x000000F0,  0x000000E1, 0x00000100,
  0x000000F1, 0x00000110,  0x00000101, 0x00000120,  0x00000111, 0x00000130,  0x00000121, 0x00000140,
  0x00000131, 0x00000150,  0x00000141, 0x00000160,  0x00000151, 0x00000170,  0x00000161, 0x00000180,
  0x00000171, 0x00000191,  0x000000D0, 0x00000181,
  // .rdex
  // @k13
  // .nlen
  0x0000001C,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002C, 0x000001BA,  0x0000003C, 0x000001AA,  0x0000004C, 0x0000019A,
  0x0000005C, 0x0000018A,  0x0000006C, 0x0000017A,  0x0000007C, 0x0000016A,  0x0000008C, 0x0000015A,
  0x0000009C, 0x0000014A,  0x000000AC, 0x0000013A,  0x000000BC, 0x0000012A,  0x000000CC, 0x0000011A,
  0x000000DC, 0x0000010A,  0x000000EA, 0x000000FA,  0x000001B0, 0x000000F0,  0x000000E1, 0x00000100,
  0x000000F1, 0x00000110,  0x00000101, 0x00000120,  0x00000111, 0x00000130,  0x00000121, 0x00000140,
  0x00000131, 0x00000150,  0x00000141, 0x00000160,  0x00000151, 0x00000170,  0x00000161, 0x00000180,
  0x00000171, 0x00000190,  0x00000181, 0x000001A0,  0x00000191, 0x000001B1,  0x000000E0, 0x000001A1,
  // .rdex
  // @k14
  // .nlen
  0x0000001E,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002C, 0x000001DA,  0x0000003C, 0x000001CA,  0x0000004C, 0x000001BA,
  0x0000005C, 0x000001AA,  0x0000006C, 0x0000019A,  0x0000007C, 0x0000018A,  0x0000008C, 0x0000017A,
  0x0000009C, 0x0000016A,  0x000000AC, 0x0000015A,  0x000000BC, 0x0000014A,  0x000000CC, 0x0000013A,
  0x000000DC, 0x0000012A,  0x000000EC, 0x0000011A,  0x000000FA, 0x0000010A,  0x000001D0, 0x00000100,
  0x000000F1, 0x00000110,  0x00000101, 0x00000120,  0x00000111, 0x00000130,  0x00000121, 0x00000140,
  0x00000131, 0x00000150,  0x00000141, 0x00000160,  0x00000151, 0x00000170,  0x00000161, 0x00000180,
  0x00000171, 0x00000190,  0x00000181, 0x000001A0,  0x00000191, 0x000001B0,  0x000001A1, 0x000001C0,
  0x000001B1, 0x000001D1,  0x000000F0, 0x000001C1,
  // .rdex
  // @k15
  // .nlen
  0x00000020,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002C, 0x000001FA,  0x0000003C, 0x000001EA,  0x0000004C, 0x000001DA,
  0x0000005C, 0x000001CA,  0x0000006C, 0x000001BA,  0x0000007C, 0x000001AA,  0x0000008C, 0x0000019A,
  0x0000009C, 0x0000018A,  0x000000AC, 0x0000017A,  0x000000BC, 0x0000016A,  0x000000CC, 0x0000015A,
  0x000000DC, 0x0000014A,  0x000000EC, 0x0000013A,  0x000000FC, 0x0000012A,  0x0000010A, 0x0000011A,
  0x000001F0, 0x00000110,  0x00000101, 0x00000120,  0x00000111, 0x00000130,  0x00000121, 0x00000140,
  0x00000131, 0x00000150,  0x00000141, 0x00000160,  0x00000151, 0x00000170,  0x00000161, 0x00000180,
  0x00000171, 0x00000190,  0x00000181, 0x000001A0,  0x00000191, 0x000001B0,  0x000001A1, 0x000001C0,
  0x000001B1, 0x000001D0,  0x000001C1, 0x000001E0,  0x000001D1, 0x000001F1,  0x00000100, 0x000001E1,
  // .rdex
  // @k16
  // .nlen
  0x00000022,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002C, 0x0000021A,  0x0000003C, 0x0000020A,  0x0000004C, 0x000001FA,
  0x0000005C, 0x000001EA,  0x0000006C, 0x000001DA,  0x0000007C, 0x000001CA,  0x0000008C, 0x000001BA,
  0x0000009C, 0x000001AA,  0x000000AC, 0x0000019A,  0x000000BC, 0x0000018A,  0x000000CC, 0x0000017A,
  0x000000DC, 0x0000016A,  0x000000EC, 0x0000015A,  0x000000FC, 0x0000014A,  0x0000010C, 0x0000013A,
  0x0000011A, 0x0000012A,  0x00000210, 0x00000120,  0x00000111, 0x00000130,  0x00000121, 0x00000140,
  0x00000131, 0x00000150,  0x00000141, 0x00000160,  0x00000151, 0x00000170,  0x00000161, 0x00000180,
  0x00000171, 0x00000190,  0x00000181, 0x000001A0,  0x00000191, 0x000001B0,  0x000001A1, 0x000001C0,
  0x000001B1, 0x000001D0,  0x000001C1, 0x000001E0,  0x000001D1, 0x000001F0,  0x000001E1, 0x00000200,
  0x000001F1, 0x00000211,  0x00000110, 0x00000201,
  // .rdex
  // @k17
  // .nlen
  0x00000024,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002C, 0x0000023A,  0x0000003C, 0x0000022A,  0x0000004C, 0x0000021A,
  0x0000005C, 0x0000020A,  0x0000006C, 0x000001FA,  0x0000007C, 0x000001EA,  0x0000008C, 0x000001DA,
  0x0000009C, 0x000001CA,  0x000000AC, 0x000001BA,  0x000000BC, 0x000001AA,  0x000000CC, 0x0000019A,
  0x000000DC, 0x0000018A,  0x000000EC, 0x0000017A,  0x000000FC, 0x0000016A,  0x0000010C, 0x0000015A,
  0x0000011C, 0x0000014A,  0x0000012A, 0x0000013A,  0x00000230, 0x00000130,  0x00000121, 0x00000140,
  0x00000131, 0x00000150,  0x00000141, 0x00000160,  0x00000151, 0x00000170,  0x00000161, 0x00000180,
  0x00000171, 0x00000190,  0x00000181, 0x000001A0,  0x00000191, 0x000001B0,  0x000001A1, 0x000001C0,
  0x000001B1, 0x000001D0,  0x000001C1, 0x000001E0,  0x000001D1, 0x000001F0,  0x000001E1, 0x00000200,
  0x000001F1, 0x00000210,  0x00000201, 0x00000220,  0x00000211, 0x00000231,  0x00000120, 0x00000221,
  // .rdex
  // @k18
  // .nlen
  0x00000026,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002C, 0x0000025A,  0x0000003C, 0x0000024A,  0x0000004C, 0x0000023A,
  0x0000005C, 0x0000022A,  0x0000006C, 0x0000021A,  0x0000007C, 0x0000020A,  0x0000008C, 0x000001FA,
  0x0000009C, 0x000001EA,  0x000000AC, 0x000001DA,  0x000000BC, 0x000001CA,  0x000000CC, 0x000001BA,
  0x000000DC, 0x000001AA,  0x000000EC, 0x0000019A,  0x000000FC, 0x0000018A,  0x0000010C, 0x0000017A,
  0x0000011C, 0x0000016A,  0x0000012C, 0x0000015A,  0x0000013A, 0x0000014A,  0x00000250, 0x00000140,
  0x00000131, 0x00000150,  0x00000141, 0x00000160,  0x00000151, 0x00000170,  0x00000161, 0x00000180,
  0x00000171, 0x00000190,  0x00000181, 0x000001A0,  0x00000191, 0x000001B0,  0x000001A1, 0x000001C0,
  0x000001B1, 0x000001D0,  0x000001C1, 0x000001E0,  0x000001D1, 0x000001F0,  0x000001E1, 0x00000200,
  0x000001F1, 0x00000210,  0x00000201, 0x00000220,  0x00000211, 0x00000230,  0x00000221, 0x00000240,
  0x00000231, 0x00000251,  0x00000130, 0x00000241,
  // .rdex
  // @k19
  // .nlen
  0x00000028,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002C, 0x0000027A,  0x0000003C, 0x0000026A,  0x0000004C, 0x0000025A,
  0x0000005C, 0x0000024A,  0x0000006C, 0x0000023A,  0x0000007C, 0x0000022A,  0x0000008C, 0x0000021A,
  0x0000009C, 0x0000020A,  0x000000AC, 0x000001FA,  0x000000BC, 0x000001EA,  0x000000CC, 0x000001DA,
  0x000000DC, 0x000001CA,  0x000000EC, 0x000001BA,  0x000000FC, 0x000001AA,  0x0000010C, 0x0000019A,
  0x0000011C, 0x0000018A,  0x0000012C, 0x0000017A,  0x0000013C, 0x0000016A,  0x0000014A, 0x0000015A,
  0x00000270, 0x00000150,  0x00000141, 0x00000160,  0x00000151, 0x00000170,  0x00000161, 0x00000180,
  0x00000171, 0x00000190,  0x00000181, 0x000001A0,  0x00000191, 0x000001B0,  0x000001A1, 0x000001C0,
  0x000001B1, 0x000001D0,  0x000001C1, 0x000001E0,  0x000001D1, 0x000001F0,  0x000001E1, 0x00000200,
  0x000001F1, 0x00000210,  0x00000201, 0x00000220,  0x00000211, 0x00000230,  0x00000221, 0x00000240,
  0x00000231, 0x00000250,  0x00000241, 0x00000260,  0x00000251, 0x00000271,  0x00000140, 0x00000261,
  // .rdex
  // @k20
  // .nlen
  0x0000002A,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002C, 0x0000029A,  0x0000003C, 0x0000028A,  0x0000004C, 0x0000027A,
  0x0000005C, 0x0000026A,  0x0000006C, 0x0000025A,  0x0000007C, 0x0000024A,  0x0000008C, 0x0000023A,
  0x0000009C, 0x0000022A,  0x000000AC, 0x0000021A,  0x000000BC, 0x0000020A,  0x000000CC, 0x000001FA,
  0x000000DC, 0x000001EA,  0x000000EC, 0x000001DA,  0x000000FC, 0x000001CA,  0x0000010C, 0x000001BA,
  0x0000011C, 0x000001AA,  0x0000012C, 0x0000019A,  0x0000013C, 0x0000018A,  0x0000014C, 0x0000017A,
  0x0000015A, 0x0000016A,  0x00000290, 0x00000160,  0x00000151, 0x00000170,  0x00000161, 0x00000180,
  0x00000171, 0x00000190,  0x00000181, 0x000001A0,  0x00000191, 0x000001B0,  0x000001A1, 0x000001C0,
  0x000001B1, 0x000001D0,  0x000001C1, 0x000001E0,  0x000001D1, 0x000001F0,  0x000001E1, 0x00000200,
  0x000001F1, 0x00000210,  0x00000201, 0x00000220,  0x00000211, 0x00000230,  0x00000221, 0x00000240,
  0x00000231, 0x00000250,  0x00000241, 0x00000260,  0x00000251, 0x00000270,  0x00000261, 0x00000280,
  0x00000271, 0x00000291,  0x00000150, 0x00000281,
  // .rdex
  // @k21
  // .nlen
  0x0000002C,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002C, 0x000002BA,  0x0000003C, 0x000002AA,  0x0000004C, 0x0000029A,
  0x0000005C, 0x0000028A,  0x0000006C, 0x0000027A,  0x0000007C, 0x0000026A,  0x0000008C, 0x0000025A,
  0x0000009C, 0x0000024A,  0x000000AC, 0x0000023A,  0x000000BC, 0x0000022A,  0x000000CC, 0x0000021A,
  0x000000DC, 0x0000020A,  0x000000EC, 0x000001FA,  0x000000FC, 0x000001EA,  0x0000010C, 0x000001DA,
  0x0000011C, 0x000001CA,  0x0000012C, 0x000001BA,  0x0000013C, 0x000001AA,  0x0000014C, 0x0000019A,
  0x0000015C, 0x0000018A,  0x0000016A, 0x0000017A,  0x000002B0, 0x00000170,  0x00000161, 0x00000180,
  0x00000171, 0x00000190,  0x00000181, 0x000001A0,  0x00000191, 0x000001B0,  0x000001A1, 0x000001C0,
  0x000001B1, 0x000001D0,  0x000001C1, 0x000001E0,  0x000001D1, 0x000001F0,  0x000001E1, 0x00000200,
  0x000001F1, 0x00000210,  0x00000201, 0x00000220,  0x00000211, 0x00000230,  0x00000221, 0x00000240,
  0x00000231, 0x00000250,  0x00000241, 0x00000260,  0x00000251, 0x00000270,  0x00000261, 0x00000280,
  0x00000271, 0x00000290,  0x00000281, 0x000002A0,  0x00000291, 0x000002B1,  0x00000160, 0x000002A1,
  // .rdex
  // @k22
  // .nlen
  0x0000002E,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002C, 0x000002DA,  0x0000003C, 0x000002CA,  0x0000004C, 0x000002BA,
  0x0000005C, 0x000002AA,  0x0000006C, 0x0000029A,  0x0000007C, 0x0000028A,  0x0000008C, 0x0000027A,
  0x0000009C, 0x0000026A,  0x000000AC, 0x0000025A,  0x000000BC, 0x0000024A,  0x000000CC, 0x0000023A,
  0x000000DC, 0x0000022A,  0x000000EC, 0x0000021A,  0x000000FC, 0x0000020A,  0x0000010C, 0x000001FA,
  0x0000011C, 0x000001EA,  0x0000012C, 0x000001DA,  0x0000013C, 0x000001CA,  0x0000014C, 0x000001BA,
  0x0000015C, 0x000001AA,  0x0000016C, 0x0000019A,  0x0000017A, 0x0000018A,  0x000002D0, 0x00000180,
  0x00000171, 0x00000190,  0x00000181, 0x000001A0,  0x00000191, 0x000001B0,  0x000001A1, 0x000001C0,
  0x000001B1, 0x000001D0,  0x000001C1, 0x000001E0,  0x000001D1, 0x000001F0,  0x000001E1, 0x00000200,
  0x000001F1, 0x00000210,  0x00000201, 0x00000220,  0x00000211, 0x00000230,  0x00000221, 0x00000240,
  0x00000231, 0x00000250,  0x00000241, 0x00000260,  0x00000251, 0x00000270,  0x00000261, 0x00000280,
  0x00000271, 0x00000290,  0x00000281, 0x000002A0,  0x00000291, 0x000002B0,  0x000002A1, 0x000002C0,
  0x000002B1, 0x000002D1,  0x00000170, 0x000002C1,
  // .rdex
  // @k23
  // .nlen
  0x00000030,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002C, 0x000002FA,  0x0000003C, 0x000002EA,  0x0000004C, 0x000002DA,
  0x0000005C, 0x000002CA,  0x0000006C, 0x000002BA,  0x0000007C, 0x000002AA,  0x0000008C, 0x0000029A,
  0x0000009C, 0x0000028A,  0x000000AC, 0x0000027A,  0x000000BC, 0x0000026A,  0x000000CC, 0x0000025A,
  0x000000DC, 0x0000024A,  0x000000EC, 0x0000023A,  0x000000FC, 0x0000022A,  0x0000010C, 0x0000021A,
  0x0000011C, 0x0000020A,  0x0000012C, 0x000001FA,  0x0000013C, 0x000001EA,  0x0000014C, 0x000001DA,
  0x0000015C, 0x000001CA,  0x0000016C, 0x000001BA,  0x0000017C, 0x000001AA,  0x0000018A, 0x0000019A,
  0x000002F0, 0x00000190,  0x00000181, 0x000001A0,  0x00000191, 0x000001B0,  0x000001A1, 0x000001C0,
  0x000001B1, 0x000001D0,  0x000001C1, 0x000001E0,  0x000001D1, 0x000001F0,  0x000001E1, 0x00000200,
  0x000001F1, 0x00000210,  0x00000201, 0x00000220,  0x00000211, 0x00000230,  0x00000221, 0x00000240,
  0x00000231, 0x00000250,  0x00000241, 0x00000260,  0x00000251, 0x00000270,  0x00000261, 0x00000280,
  0x00000271, 0x00000290,  0x00000281, 0x000002A0,  0x00000291, 0x000002B0,  0x000002A1, 0x000002C0,
  0x000002B1, 0x000002D0,  0x000002C1, 0x000002E0,  0x000002D1, 0x000002F1,  0x00000180, 0x000002E1,
  // .rdex
  // @k24
  // .nlen
  0x00000032,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002C, 0x0000031A,  0x0000003C, 0x0000030A,  0x0000004C, 0x000002FA,
  0x0000005C, 0x000002EA,  0x0000006C, 0x000002DA,  0x0000007C, 0x000002CA,  0x0000008C, 0x000002BA,
  0x0000009C, 0x000002AA,  0x000000AC, 0x0000029A,  0x000000BC, 0x0000028A,  0x000000CC, 0x0000027A,
  0x000000DC, 0x0000026A,  0x000000EC, 0x0000025A,  0x000000FC, 0x0000024A,  0x0000010C, 0x0000023A,
  0x0000011C, 0x0000022A,  0x0000012C, 0x0000021A,  0x0000013C, 0x0000020A,  0x0000014C, 0x000001FA,
  0x0000015C, 0x000001EA,  0x0000016C, 0x000001DA,  0x0000017C, 0x000001CA,  0x0000018C, 0x000001BA,
  0x0000019A, 0x000001AA,  0x00000310, 0x000001A0,  0x00000191, 0x000001B0,  0x000001A1, 0x000001C0,
  0x000001B1, 0x000001D0,  0x000001C1, 0x000001E0,  0x000001D1, 0x000001F0,  0x000001E1, 0x00000200,
  0x000001F1, 0x00000210,  0x00000201, 0x00000220,  0x00000211, 0x00000230,  0x00000221, 0x00000240,
  0x00000231, 0x00000250,  0x00000241, 0x00000260,  0x00000251, 0x00000270,  0x00000261, 0x00000280,
  0x00000271, 0x00000290,  0x00000281, 0x000002A0,  0x00000291, 0x000002B0,  0x000002A1, 0x000002C0,
  0x000002B1, 0x000002D0,  0x000002C1, 0x000002E0,  0x000002D1, 0x000002F0,  0x000002E1, 0x00000300,
  0x000002F1, 0x00000311,  0x00000190, 0x00000301,
  // .rdex
  // @low
  // .nlen
  0x00000005,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002A, 0x00000041,  0x0BF2E984, 0x0000003A,  0x0BF2E924, 0x0000004A,
  0x000000E4, 0x00000011,
  // .rdex
  // @mul
  // .nlen
  0x00000006,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002A, 0x0000003A,  0x00000041, 0x00000051,  0x0000004A, 0x0000005A,
  0x00000050, 0x00000020,  0x00000040, 0x00000021,
  // .rdex
  // @nid
  // .nlen
  0x00000004,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002A, 0x00000031,  0x0C6C9DC4, 0x0000003A,  0x00000234, 0x00000011,
  // .rdex
  // @not
  // .nlen
  0x00000006,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002A, 0x0000004A,  0x00000050, 0x0000003A,  0x00000040, 0x00000051,
  0x00000030, 0x0000005A,  0x00000020, 0x00000031,
  // .rdex
  // @run
  // .nlen
  0x00000005,
  // .rlen
  0x00000000,
  // .node
  0x00000000, 0x0000001A,  0x0000002A, 0x00000041,  0x0D78C584, 0x0000003A,  0x0D78C524, 0x0000004A,
  0x000000E4, 0x00000011,
  // .rdex
  // @brnS
  // .nlen
  0x00000006,
  // .rlen
  0x00000002,
  // .node
  0x00000000, 0x0000001A,  0x0000002B, 0x0000003A,  0x00000040, 0x00000050,  0x00000041, 0x00000051,
  0x00000020, 0x00000030,  0x00000021, 0x00000031,
  // .rdex
  0x0025D714, 0x0000004A,  0x0025D714, 0x0000005A,
  // @brnZ
  // .nlen
  0x00000004,
  // .rlen
  0x00000002,
  // .node
  0x00000000, 0x00000011,  0x00000031, 0x00000001,  0x00000124, 0x0000003A,  0x000000E4, 0x00000010,
  // .rdex
  0x0035E314, 0x0000001A,  0x00260414, 0x0000002A,
  // @decI
  // .nlen
  0x00000003,
  // .rlen
  0x00000001,
  // .node
  0x00000000, 0x0000001A,  0x00000020, 0x00000021,  0x00000010, 0x00000011,
  // .rdex
  0x002FCBA4, 0x0000002A,
  // @decO
  // .nlen
  0x00000004,
  // .rlen
  0x00000002,
  // .node
  0x00000000, 0x0000001A,  0x00000030, 0x00000021,  0x00000031, 0x00000011,  0x00000010, 0x00000020,
  // .rdex
  0x00000124, 0x0000002A,  0x0027A264, 0x0000003A,
  // @lowI
  // .nlen
  0x00000004,
  // .rlen
  0x00000002,
  // .node
  0x00000000, 0x0000001A,  0x00000020, 0x00000031,  0x00000010, 0x00000030,  0x00000021, 0x00000011,
  // .rdex
  0x00000124, 0x0000002A,  0x00000184, 0x0000003A,
  // @lowO
  // .nlen
  0x00000004,
  // .rlen
  0x00000002,
  // .node
  0x00000000, 0x0000001A,  0x00000020, 0x00000031,  0x00000010, 0x00000030,  0x00000021, 0x00000011,
  // .rdex
  0x00000184, 0x0000002A,  0x00000184, 0x0000003A,
  // @nidS
  // .nlen
  0x00000004,
  // .rlen
  0x00000002,
  // .node
  0x00000000, 0x0000001A,  0x00000030, 0x00000021,  0x00000031, 0x00000011,  0x00000010, 0x00000020,
  // .rdex
  0x000001C4, 0x0000002A,  0x0031B274, 0x0000003A,
  // @runI
  // .nlen
  0x00000005,
  // .rlen
  0x00000003,
  // .node
  0x00000000, 0x0000001A,  0x00000040, 0x00000021,  0x00000031, 0x00000011,  0x00000041, 0x00000020,
  0x00000010, 0x00000030,
  // .rdex
  0x0035E314, 0x0000002A,  0x0027A264, 0x0000003A,  0x00000124, 0x0000004A,
  // @runO
  // .nlen
  0x00000005,
  // .rlen
  0x00000003,
  // .node
  0x00000000, 0x0000001A,  0x00000040, 0x00000021,  0x00000031, 0x00000011,  0x00000041, 0x00000020,
  0x00000010, 0x00000030,
  // .rdex
  0x0035E314, 0x0000002A,  0x0027A264, 0x0000003A,  0x00000184, 0x0000004A,
};
u32 JUMP_DATA[] = {
  0x0000000E, 0x00000000, // @E
  0x0000000F, 0x0000000A, // @F
  0x00000012, 0x00000012, // @I
  0x00000018, 0x00000020, // @O
  0x0000001C, 0x0000002E, // @S
  0x0000001D, 0x0000003A, // @T
  0x00000023, 0x00000042, // @Z
  0x00000929, 0x0000004A, // @af
  0x00000980, 0x00000054, // @c0
  0x00000981, 0x0000005C, // @c1
  0x00000982, 0x00000066, // @c2
  0x00000983, 0x00000074, // @c3
  0x00000984, 0x00000086, // @c4
  0x00000985, 0x0000009C, // @c5
  0x00000986, 0x000000B6, // @c6
  0x00000987, 0x000000D4, // @c7
  0x00000988, 0x000000F6, // @c8
  0x00000989, 0x0000011C, // @c9
  0x00000B27, 0x00000146, // @id
  0x00000B80, 0x0000014C, // @k0
  0x00000B81, 0x00000154, // @k1
  0x00000B82, 0x0000015E, // @k2
  0x00000B83, 0x0000016C, // @k3
  0x00000B84, 0x0000017E, // @k4
  0x00000B85, 0x00000194, // @k5
  0x00000B86, 0x000001AE, // @k6
  0x00000B87, 0x000001CC, // @k7
  0x00000B88, 0x000001EE, // @k8
  0x00000B89, 0x00000214, // @k9
  0x00024A5C, 0x0000023E, // @afS
  0x00024A63, 0x00000254, // @afZ
  0x00024C67, 0x00000258, // @and
  0x00025D71, 0x0000026E, // @brn
  0x00026040, 0x00000278, // @c10
  0x00026041, 0x000002A6, // @c11
  0x00026042, 0x000002D8, // @c12
  0x00026043, 0x0000030E, // @c13
  0x00026044, 0x00000348, // @c14
  0x00026045, 0x00000386, // @c15
  0x00026046, 0x000003C8, // @c16
  0x00026047, 0x0000040E, // @c17
  0x00026048, 0x00000458, // @c18
  0x00026049, 0x000004A6, // @c19
  0x00026080, 0x000004F8, // @c20
  0x00026081, 0x0000054E, // @c21
  0x00026082, 0x000005A8, // @c22
  0x00026083, 0x00000606, // @c23
  0x00026084, 0x00000668, // @c24
  0x00026085, 0x000006CE, // @c25
  0x00026086, 0x00000738, // @c26
  0x00026FB6, 0x000007A6, // @c_s
  0x00026FBD, 0x000007B8, // @c_z
  0x00027A26, 0x000007C0, // @dec
  0x00028EC0, 0x000007CC, // @ex0
  0x00028EC1, 0x000007D4, // @ex1
  0x00028EC2, 0x000007DE, // @ex2
  0x00028EC3, 0x000007EC, // @ex3
  0x00028EC4, 0x000007FA, // @ex4
  0x00028EC5, 0x00000804, // @ex5
  0x0002AFB6, 0x0000080E, // @g_s
  0x0002AFBD, 0x0000081C, // @g_z
  0x0002E040, 0x00000822, // @k10
  0x0002E041, 0x00000850, // @k11
  0x0002E042, 0x00000882, // @k12
  0x0002E043, 0x000008B8, // @k13
  0x0002E044, 0x000008F2, // @k14
  0x0002E045, 0x00000930, // @k15
  0x0002E046, 0x00000972, // @k16
  0x0002E047, 0x000009B8, // @k17
  0x0002E048, 0x00000A02, // @k18
  0x0002E049, 0x00000A50, // @k19
  0x0002E080, 0x00000AA2, // @k20
  0x0002E081, 0x00000AF8, // @k21
  0x0002E082, 0x00000B52, // @k22
  0x0002E083, 0x00000BB0, // @k23
  0x0002E084, 0x00000C12, // @k24
  0x0002FCBA, 0x00000C78, // @low
  0x00030E2F, 0x00000C84, // @mul
  0x00031B27, 0x00000C92, // @nid
  0x00031CB7, 0x00000C9C, // @not
  0x00035E31, 0x00000CAA, // @run
  0x00975C5C, 0x00000CB6, // @brnS
  0x00975C63, 0x00000CC8, // @brnZ
  0x009E8992, 0x00000CD6, // @decI
  0x009E8998, 0x00000CE0, // @decO
  0x00BF2E92, 0x00000CEE, // @lowI
  0x00BF2E98, 0x00000CFC, // @lowO
  0x00C6C9DC, 0x00000D0A, // @nidS
  0x00D78C52, 0x00000D18, // @runI
  0x00D78C58, 0x00000D2A, // @runO
};

const size_t BOOK_DATA_SIZE = sizeof(BOOK_DATA) / sizeof(u32);
const size_t JUMP_DATA_SIZE = sizeof(JUMP_DATA) / sizeof(u32);

// Main
// ----

int main() {
  // Prints device info
  int device;
  hipDeviceProp_t prop;
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);
  printf("CUDA Device: %s, Compute Capability: %d.%d\n\n", prop.name, prop.major, prop.minor);
  printf("Total global memory: %zu bytes\n", prop.totalGlobalMem);
  printf("Shared memory per block: %zu bytes\n", prop.sharedMemPerBlock);
  printf("Registers per block: %d\n", prop.regsPerBlock);
  printf("Warp size: %d\n", prop.warpSize);
  printf("Maximum threads per block: %d\n", prop.maxThreadsPerBlock);
  printf("Maximum thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
  printf("Maximum grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
  printf("Clock rate: %d kHz\n", prop.clockRate);
  printf("Total constant memory: %zu bytes\n", prop.totalConstMem);
  printf("Compute capability: %d.%d\n", prop.major, prop.minor);
  printf("Number of multiprocessors: %d\n", prop.multiProcessorCount);
  printf("Concurrent copy and execution: %s\n", (prop.deviceOverlap ? "Yes" : "No"));
  printf("Kernel execution timeout: %s\n", (prop.kernelExecTimeoutEnabled ? "Yes" : "No"));

  // Prints info about the do_global_rewrite kernel
  hipFuncAttributes attr;
  hipError_t err = hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(global_rewrite));
  if (err != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(err));
  } else {
    printf("\n");
    printf("Number of registers used: %d\n", attr.numRegs);
    printf("Shared memory used: %zu bytes\n", attr.sharedSizeBytes);
    printf("Constant memory used: %zu bytes\n", attr.constSizeBytes);
    printf("Size of local memory frame: %zu bytes\n", attr.localSizeBytes);
    printf("Maximum number of threads per block: %d\n", attr.maxThreadsPerBlock);
    printf("Number of PTX versions supported: %d\n", attr.ptxVersion);
    printf("Number of Binary versions supported: %d\n", attr.binaryVersion);
  }

  // Allocates net on CPU
  Net* cpu_net = mknet(F_ex3, JUMP_DATA, JUMP_DATA_SIZE);

  // Prints the input net
  printf("\nINPUT\n=====\n\n");
  print_net(cpu_net);

  // Uploads net and book to GPU
  Net* gpu_net = net_to_gpu(cpu_net);
  Book* gpu_book = init_book_on_gpu(BOOK_DATA, BOOK_DATA_SIZE);

  // Marks init time
  struct timespec start, end;
  // clock_gettime(CLOCK_MONOTONIC_RAW, &start);

  // Normalizes
  do_global_expand(gpu_net, gpu_book);
  for (u32 tick = 0; tick < 128; ++tick) {
    do_global_rewrite(gpu_net, gpu_book, 16, tick, (tick / BAGS_WIDTH_L2) % 2);
  }
  do_global_expand(gpu_net, gpu_book);
  do_global_rewrite(gpu_net, gpu_book, 200000, 0, 0);
  hipDeviceSynchronize();

  // Gets end time
  // clock_gettime(CLOCK_MONOTONIC_RAW, &end);
  uint32_t delta_time = (end.tv_sec - start.tv_sec) * 1000 + (end.tv_nsec - start.tv_nsec) / 1000000;

  // Reads result back to cpu
  Net* norm = net_to_cpu(gpu_net);

  // Prints the output
  printf("\nNORMAL ~ rewrites=%llu\n======\n\n", norm->rwts);
  //print_tree(norm, norm->root);
  print_net(norm);
  printf("Time: %.3f s\n", ((double)delta_time) / 1000.0);
  printf("RPS : %.3f million\n", ((double)norm->rwts) / ((double)delta_time) / 1000.0);

  // Clears CPU memory
  net_free_on_gpu(gpu_net);
  book_free_on_gpu(gpu_book);

  // Clears GPU memory
  net_free_on_cpu(cpu_net);
  net_free_on_cpu(norm);

  return 0;
}
 */